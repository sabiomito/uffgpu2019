
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <unistd.h>
#include <stdlib.h>
#include <math.h>
using namespace std;

__global__ void _copy_dr_to_de(int *d_e,int *d_r,int X,int Y){
    int x,y;
    x = threadIdx.x + (blockIdx.x*blockDim.x);
    y = threadIdx.y + (blockIdx.y*blockDim.y);
    int h_r_i = x + ( y * (X) );
    d_e[h_r_i] = d_r[h_r_i];
}
__global__ void _2Dstencil_global(int *d_e,int *d_r,float *c_coeff,int X,int Y,int k){

    int x,y;
    x = threadIdx.x + (blockIdx.x*blockDim.x);
    y = threadIdx.y + (blockIdx.y*blockDim.y);
    int h_r_i = x + ( y * (X) );
    int h_e_i = h_r_i;
    d_r[h_r_i] = d_e[h_e_i];
    for(int lk = 1;lk<(k/2)+1;lk++)
    {
        if(x+lk >= X)
            h_e_i = (x-lk) + ( (y) * (X) );
        else
            h_e_i = (x+lk) + ( (y) * (X) );
        d_r[h_r_i] += d_e[h_e_i]*c_coeff[lk-1];

        if(x-lk < 0)
            h_e_i = (x+lk) + ( (y) * (X) );
        else
            h_e_i = (x-lk) + ( (y) * (X) );
        d_r[h_r_i] += d_e[h_e_i]*c_coeff[lk-1];


        if(y+lk >= Y)
            h_e_i = (x) + ( (y-lk) * (X) );
        else
            h_e_i = (x) + ( (y+lk) * (X) );
        d_r[h_r_i] += d_e[h_e_i]*c_coeff[lk-1];

        if(y-lk < 0)
            h_e_i = (x) + ( (y+lk) * (X) );
        else
            h_e_i = (x) + ( (y-lk) * (X) );
        d_r[h_r_i] += d_e[h_e_i]*c_coeff[lk-1];

    }  


}


int main(int argc, char* argv[]) {

int *h_e,*h_r;
int *d_e, *d_r;
int size,tam;
int X=32;
int Y=32;
int k=4;
int times = 1;
int BX=32;
int BY=32;
int GX=1;
int GY=1;
float *c_coeff,*d_c_coeff;
if(argc > 1)
{
    X = atoi(argv[1]);
    Y = X;
}
if(argc > 2)
{
    k = atoi(argv[2]);
}

if(argc > 3)
{
    times = atoi(argv[3]);
}


if(X>32)
{
    GX = ceil((float)X/(float)32);
    BX = 32;
}
if(Y>32)
{
    GY = ceil((float)Y/(float)32);
    BY = 32;
}
    

dim3 block_dim(BX,BY,1);
dim3 grid_dim(GX,GY,1);

size = X * Y * sizeof(int);
tam = X * Y;


h_e = (int*) malloc(size);
h_r = (int*) malloc(size);
c_coeff = (float*)malloc((k/2+1)*sizeof(float));
hipMalloc(&d_e, size);
hipMalloc(&d_r, size);
hipMalloc(&d_c_coeff,(k/2+1)*sizeof(float));

for(int i=0;i<(k/2+1);i++)
    c_coeff[(k/2+1)-i-1]=(float)i/(float)(k/2+1);


FILE *arq;
arq = fopen("entrada.txt", "rt");
for(int i=0;i<X;i++)
    for(int j=0;j<Y;j++)
        fscanf(arq," %d",&h_e[i+j*X]);
fclose(arq);


/* Copy vectors from host memory to device memory */
hipMemcpy(d_e, h_e, size, hipMemcpyHostToDevice);
hipMemcpy(d_c_coeff, c_coeff, (k/2+1)*sizeof(float), hipMemcpyHostToDevice);


hipEvent_t start, stop;
hipEventCreate (&start);
hipEventCreate (&stop);
hipEventRecord (start, 0); 

/******************
*** Kernel Call ***
*******************/
//_3Dstencil_global<<<blks,th_p_blk>>>(d_e,d_r,X,Y,Z);
for(int t=0;t<times;t++)
{
_2Dstencil_global<<<grid_dim,block_dim>>>(d_e,d_r,d_c_coeff,X,Y,k);
_copy_dr_to_de<<<grid_dim,block_dim>>>(d_e,d_r,X,Y);
}
hipError_t err = hipSuccess;
err = hipGetLastError();
if (err != hipSuccess)
{
    fprintf(stderr, "Failed to launch _3Dstencil_global kernel (error code %s)!\n", hipGetErrorString(err));
}
/******************
*** Kernel Call ***
*******************/



    hipDeviceSynchronize();
    hipEventRecord (stop, 0);
    hipEventSynchronize (stop);
    float elapsedTime;
    hipEventElapsedTime (&elapsedTime, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);


    printf("X %d || Y %d \nBX %d || BY %d \n",X,Y,BX,BY);
    printf ("[%d,%.5f],\n", tam,elapsedTime);
 
    hipMemcpy(h_r, d_r, size, hipMemcpyDeviceToHost);

    
arq = fopen("resultado.txt", "wt");
for(int i=0;i<X;i++)
{
    for(int j=0;j<Y;j++)
    {
      fprintf(arq," %d",h_r[i+j*X]);
    }
    fprintf(arq,"\n");
}
fclose(arq);


    hipFree(d_e);
    hipFree(d_r);
    hipFree(d_c_coeff);
    std::free(h_e);
    std::free(h_r);
    std::free(c_coeff);

    return 0;
} /* main */

