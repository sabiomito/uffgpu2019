
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <unistd.h>
#include <stdlib.h>
#include <math.h>

using namespace std;
#define TYPE float

__global__ void fwd_3D_16x16_order8( TYPE *g_input, TYPE *g_output, TYPE *g_vsq, /* output initially contains (t-2) step*/const int dimx, const int dimy, const int dimz)
{
#define BDIMX 16 // tile (and threadblock) size in x
#define BDIMY 16 // tile (and threadblock) size in y
#define radius 4 // half of the order in space (k/2)
float c_coeff[5] = {1.0 , 0.8 , 0.6 , 0.4, 0.2};
__shared__ float s_data[BDIMY+2*radius][BDIMX+2*radius];
int ix = blockIdx.x*blockDim.x + threadIdx.x;
int iy = blockIdx.y*blockDim.y + threadIdx.y;
int in_idx = iy*dimx + ix; // index for reading input
int out_idx = 0; // index for writing output
int stride = dimx*dimy; // distance between 2D slices (in elements)
float infront1, infront2, infront3, infront4; // variables for input “in front of” the current slice
float behind1, behind2, behind3, behind4; // variables for input “behind” the current slice
float current; // input value in the current slice
int tx = threadIdx.x + radius; // thread’s x-index into corresponding shared memory tile (adjusted for halos)
int ty = threadIdx.y + radius; // thread’s y-index into corresponding shared memory tile (adjusted for halos)
// fill the "in-front" and "behind" data
behind3 = g_input[in_idx]; in_idx += stride;
behind2 = g_input[in_idx]; in_idx += stride;
behind1 = g_input[in_idx]; in_idx += stride;
current = g_input[in_idx]; out_idx = in_idx; in_idx += stride;
infront1 = g_input[in_idx]; in_idx += stride;
infront2 = g_input[in_idx]; in_idx += stride;
infront3 = g_input[in_idx]; in_idx += stride;
infront4 = g_input[in_idx]; in_idx += stride;
for(int i=radius; i<dimz-radius; i++)
{
//////////////////////////////////////////
// advance the slice (move the thread-front)
 behind4 = behind3;
 behind3 = behind2;
 behind2 = behind1;
 behind1 = current;
 current = infront1;
 infront1 = infront2;
 infront2 = infront3;
 infront3 = infront4;
 infront4 = g_input[in_idx];
 in_idx += stride;
 out_idx += stride;
 __syncthreads();
/////////////////////////////////////////
// update the data slice in smem
if(threadIdx.y<radius) // halo above/below
 {
 s_data[threadIdx.y][tx] = g_input[out_idx-radius*dimx];
 s_data[threadIdx.y+BDIMY+radius][tx] = g_input[out_idx+BDIMY*dimx];
 }
if(threadIdx.x<radius) // halo left/right
 {
 s_data[ty][threadIdx.x] = g_input[out_idx-radius];
 s_data[ty][threadIdx.x+BDIMX+radius] = g_input[out_idx+BDIMX];
 }
// update the slice in smem
 s_data[ty][tx] = current;
 __syncthreads();
/////////////////////////////////////////
// compute the output value
 float temp = 2.f*current - g_output[out_idx];
 float div = c_coeff[0] * current; //c_coefff deveria ser um array do tamanho do radius
 div += c_coeff[1]*( infront1 + behind1
 + s_data[ty-1][tx] + s_data[ty+1][tx] + s_data[ty][tx-1] + s_data[ty][tx+1] );
 div += c_coeff[2]*( infront2 + behind2 + s_data[ty-2][tx] + s_data[ty+2][tx] + s_data[ty][tx-2] + s_data[ty][tx+2] );
 div += c_coeff[3]*( infront3 + behind3 + s_data[ty-3][tx] + s_data[ty+3][tx] + s_data[ty][tx-3] + s_data[ty][tx+3] );
 div += c_coeff[4]*( infront4 + behind4 + s_data[ty-4][tx] + s_data[ty+4][tx] + s_data[ty][tx-4] + s_data[ty][tx+4] );
 g_output[out_idx] = temp + div*g_vsq[out_idx];
}
}
//Tamanho do radius não é variável
//A entrada de dados ja vem com as bordas aumentadas

/*
*argumentos
*1 - n_elementos
*2 - threads por bloco
*3 - n_blocos
*4 - print
*/
int main(int argc, char* argv[]) {

    float *h_e,*h_r,*h_r_test,*h_g_vsq;
    float *d_e, *d_r,*d_g_vsq;
    int size,tam,times;
    clock_t Ticks[2];

    

    

    times = 1;
    int X=8;
    int Y=8;
    int BX=8;
    int BY=8;
    int Z=4;
    int k=2;
    int GX=1;
    int GY=1;

    if(argc > 1)
    {
        X = atoi(argv[1]);
        BX=X;
    }
      
    if(argc > 2)
    {
        Y = atoi(argv[2]);
        BY = Y;
    }
      
    if(argc > 3)
      Z = atoi(argv[3]);
    if(argc > 4)
      k = atoi(argv[4]);

    if(X>32)
    {
        GX = ceil((float)X/(float)32);
        BX = 32;
    }
    if(Y>32)
    {
        GY = ceil((float)Y/(float)32);
        BY = 32;
    }
    
    
    dim3 block_dim(BX,BY,1);
    dim3 grid_dim(GX,GY,1);

    //sharedSize = block_dim.x*block_dim.y*sizeof(float);
    size = X * Y * Z * sizeof(float);
    tam = X * Y * Z;


    h_e = (float*) malloc(size);
    h_r = (float*) malloc(size);
    h_r_test = (float*) malloc(size);
    h_g_vsq = (float*) malloc(size);
    hipMalloc(&d_e, size);
    hipMalloc(&d_r, size);
    hipMalloc(&d_g_vsq, size);


    for (int i = 0; i < tam; i++) {
        h_g_vsq[i] = (float)(rand()%100)/100.0;
        h_e[i] = (float)(rand()%9000)/100.0;
        h_r[i] = 0;
    }

    /* Copy vectors from host memory to device memory */
    hipMemcpy(d_g_vsq, h_g_vsq, size, hipMemcpyHostToDevice);
    hipMemcpy(d_e, h_e, size, hipMemcpyHostToDevice);
    hipMemcpy(d_r, h_r, size, hipMemcpyHostToDevice);



    hipEvent_t start, stop;
    hipEventCreate (&start);
    hipEventCreate (&stop);
    hipEventRecord (start, 0); 

    /******************
    *** Kernel Call ***
    *******************/
    //_3Dstencil_global<<<blks,th_p_blk>>>(d_e,d_r,X,Y,Z);
    //_3Dstencil_sharedMemory<<<grid_dim,block_dim,sharedSize>>>(d_e,d_r,X,Y,Z,k);
    fwd_3D_16x16_order8<<<grid_dim,block_dim>>>(d_e,d_r,d_g_vsq, X,Y,Z);
    hipError_t err = hipSuccess;
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch _3Dstencil_global kernel (error code %s)!\n", hipGetErrorString(err));
    }
    /******************
    *** Kernel Call ***
    *******************/



    hipDeviceSynchronize();
    hipEventRecord (stop, 0);
    hipEventSynchronize (stop);
    float elapsedTime;
    hipEventElapsedTime (&elapsedTime, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    
    Ticks[1] = clock();
    double Tempo = (Ticks[1] - Ticks[0]) * 1000.0 / CLOCKS_PER_SEC;
    printf("X %d || Y %d \nBX %d || BY %d\nGX %d || GY %d\nZ %d \n",X,Y,BX,BY,GX,GY,Z);
    printf ("[%d,%.5f,%.5f],\n", tam,elapsedTime,Tempo/1000.0);
 
    hipMemcpy(h_r, d_r, size, hipMemcpyDeviceToHost);

    
    hipFree(d_e);
    hipFree(d_r);
    std::free(h_e);
    std::free(h_r);
    std::free(h_r_test);

    return 0;
} /* main */

