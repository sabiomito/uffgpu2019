#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <unistd.h>
#include <stdlib.h>
#include <math.h>
using namespace std;

__global__ void _3Dstencil_sharedMemory(float *d_e,float *d_r,int X,int Y,int Z,int k){

    //int thread_id = threadIdx.x + blockIdx.x * blockDim.x;
    //printf("sou id %d || threadIdx.x %d || blockIdx.x %d || blockDim.x %d \n",thread_id,threadIdx.x ,blockIdx.x,blockDim.x);
    //int thread_id = threadIdx.x + threadIdx.y*blockDim.x + (blockIdx.x + blockIdx.y*gridDim.x)*blockDim.x*blockDim.y;
    //printf("sou id %d || threadIdx.x %d || blockIdx.x %d|| blockIdx.y %d || blockDim.x %d|| blockDim.y %d \n",thread_id,threadIdx.x ,blockIdx.x,blockIdx.y,blockDim.x,blockDim.y);
    int x,y;
    x = threadIdx.x + (blockIdx.x*blockDim.x);
    y = threadIdx.y + (blockIdx.y*blockDim.y);

    extern __shared__ float fatia[];

    //printf("X = %d || Y = %d\n",x,y);
    float *Zdata = new float[k+1];
    int z=0;
    Zdata[k/2] = d_e[x + ( y * (X) ) + ( z* (X*Y) )];
    for(int lk =1;lk<(k/2)+1;lk++)
    {
        int h_e_i;
        if(z+lk >= Z)
            h_e_i = (x) + ( (y) * (X) ) + ( (z-lk) * (X*Y) );
        else
            h_e_i = (x) + ( (y) * (X) ) + ( (z+lk) * (X*Y) );
        Zdata[k+1-lk] = d_e[h_e_i];

        if(z-lk < 0)
            h_e_i = (x) + ( (y) * (X) ) + ( (z+lk) * (X*Y) );
        else
            h_e_i = (x) + ( (y) * (X) ) + ( (z-lk) * (X*Y) );
        Zdata[lk-1] = d_e[h_e_i];

    }
    for(int z=0;  z<Z;  z++)
    {
        
        int h_r_i = x + ( y * (X) ) + ( z* (X*Y) );
        //fatia[(k/2+threadIdx.x) + (k/2+threadIdx.y)*(blockDim.x+k)] = d_r[h_r_i];
        fatia[threadIdx.x + threadIdx.y*blockDim.x] = d_e[h_r_i];
        
        __syncthreads();
        
        /*if(blockIdx.x==0 && threadIdx.x==2 && threadIdx.y==2 && z==3)
        {
            printf("\n\n");
            printf("d_e[(x) + ( (y) * (X) ) + ( (z+lk) * (X*Y) )] = %f\n",d_e[(x) + ( (y) * (X) ) + ( (z+1) * (X*Y) )]);
            printf("d_e[h_r_i] = %f\n",d_e[h_r_i]);
            printf("d_e[(x) + ( (y) * (X) ) + ( (z-lk) * (X*Y) )] = %f\n",d_e[(x) + ( (y) * (X) ) + ( (z-1) * (X*Y) )]);
            for(int i=0;i<k+1;i++)
            printf("Zdata[%d] = %f\n",i,Zdata[i]);
            printf("\nFATIA\n");
            for(int i=0;i<blockDim.x;i++)
            {
                for(int j=0;j<blockDim.y;j++)
                {
                    printf(" %f",fatia[i+j*blockDim.x]);
                }
                printf("\n");
            }
        }*/
           
        
        
        
        
        int h_e_i = h_r_i;
        d_r[h_r_i] = d_e[h_e_i];
        for(int lk =1;lk<(k/2)+1;lk++)
            {
                if(x+lk >= X)
                    h_e_i = (x-lk) + ( (y) * (X) ) + ( (z) * (X*Y) );
                else
                    h_e_i = (x+lk) + ( (y) * (X) ) + ( (z) * (X*Y) );
                d_r[h_r_i] += d_e[h_e_i];

                if(x-lk < 0)
                    h_e_i = (x+lk) + ( (y) * (X) ) + ( (z) * (X*Y) );
                else
                    h_e_i = (x-lk) + ( (y) * (X) ) + ( (z) * (X*Y) );
                d_r[h_r_i] += d_e[h_e_i];


                if(y+lk >= Y)
                    h_e_i = (x) + ( (y-lk) * (X) ) + ( (z) * (X*Y) );
                else
                    h_e_i = (x) + ( (y+lk) * (X) ) + ( (z) * (X*Y) );
                d_r[h_r_i] += d_e[h_e_i];

                if(y-lk < 0)
                    h_e_i = (x) + ( (y+lk) * (X) ) + ( (z) * (X*Y) );
                else
                    h_e_i = (x) + ( (y-lk) * (X) ) + ( (z) * (X*Y) );
                d_r[h_r_i] += d_e[h_e_i];


                if(z+lk >= Z)
                    h_e_i = (x) + ( (y) * (X) ) + ( (z-lk) * (X*Y) );
                else
                    h_e_i = (x) + ( (y) * (X) ) + ( (z+lk) * (X*Y) );
                d_r[h_r_i] += d_e[h_e_i];

                if(z-lk < 0)
                    h_e_i = (x) + ( (y) * (X) ) + ( (z+lk) * (X*Y) );
                else
                    h_e_i = (x) + ( (y) * (X) ) + ( (z-lk) * (X*Y) );
                d_r[h_r_i] += d_e[h_e_i];

            }
         if(z==Z-1)
            break;   
        for(int i=0;i<k;i++)
            Zdata[i]=Zdata[i+1];
        int lk=k/2;
        if(z+1+lk >= Z)
            h_e_i = (x) + ( (y) * (X) ) + ( (z+1-lk) * (X*Y) );
        else
            h_e_i = (x) + ( (y) * (X) ) + ( (z+1+lk) * (X*Y) );
        Zdata[k] = d_e[h_e_i];

        
    }


}

/*
*argumentos
*1 - n_elementos
*2 - threads por bloco
*3 - n_blocos
*4 - print
*/
int main(int argc, char* argv[]) {

    float *h_e,*h_r,*h_r_test;
    float *d_e, *d_r;
    int size,tam,times,sharedSize;
    clock_t Ticks[2];

    

    

    times = 1;
    int X=8;
    int Y=8;
    int BX=8;
    int BY=8;
    int Z=4;
    int k=2;
    int GX=1;
    int GY=1;

    if(argc > 1)
    {
        X = atoi(argv[1]);
        BX=X;
    }
      
    if(argc > 2)
    {
        Y = atoi(argv[2]);
        BY = Y;
    }
      
    if(argc > 3)
      Z = atoi(argv[3]);
    if(argc > 4)
      k = atoi(argv[4]);

    if(X>32)
    {
        GX = ceil((float)X/(float)32);
        BX = 32;
    }
    if(Y>32)
    {
        GY = ceil((float)Y/(float)32);
        BY = 32;
    }
    
    
    dim3 block_dim(BX,BY,1);
    dim3 grid_dim(GX,GY,1);

    //sharedSize = (block_dim.x*block_dim.y + k*block_dim.x + k*block_dim.y)*sizeof(float);
    sharedSize = block_dim.x*block_dim.y*sizeof(float);
    size = X * Y * Z * sizeof(float);
    tam = X * Y * Z;


    h_e = (float*) malloc(size);
    h_r = (float*) malloc(size);
    h_r_test = (float*) malloc(size);
    hipMalloc(&d_e, size);
    hipMalloc(&d_r, size);


    for (int i = 0; i < tam; i++) {
        h_e[i] = (float)(rand()%9000)/100.0;
        h_r[i] = 0;
    }

    /* Copy vectors from host memory to device memory */
    hipMemcpy(d_e, h_e, size, hipMemcpyHostToDevice);
    hipMemcpy(d_r, h_r, size, hipMemcpyHostToDevice);

    for(int t =0; t<times; t++)
    {

        for(int z=0;  z<Z;  z++)
        {
        
            for(int y=0;  y<Y;  y++)
            {
                for(int x=0;  x<X;  x++)
                {
                    
                    
                    
                    int h_r_i = x + ( y * (X) ) + ( z* (X*Y) );
                        
                    int h_e_i = h_r_i;
                    //printf(" %f",h_e[h_e_i]);
                    h_r_test[h_r_i] = h_e[h_e_i];
                    for(int lk =1;lk<(k/2)+1;lk++)
                        {
                            

                            
                            if(x+lk >= X)
                                h_e_i = (x-lk) + ( (y) * (X) ) + ( (z) * (X*Y) );
                            else
                                h_e_i = (x+lk) + ( (y) * (X) ) + ( (z) * (X*Y) );
                            h_r_test[h_r_i] += h_e[h_e_i];

                            if(x-lk < 0)
                                h_e_i = (x+lk) + ( (y) * (X) ) + ( (z) * (X*Y) );
                            else
                                h_e_i = (x-lk) + ( (y) * (X) ) + ( (z) * (X*Y) );
                            h_r_test[h_r_i] += h_e[h_e_i];


                            if(y+lk >= Y)
                                h_e_i = (x) + ( (y-lk) * (X) ) + ( (z) * (X*Y) );
                            else
                                h_e_i = (x) + ( (y+lk) * (X) ) + ( (z) * (X*Y) );
                            h_r_test[h_r_i] += h_e[h_e_i];

                            if(y-lk < 0)
                                h_e_i = (x) + ( (y+lk) * (X) ) + ( (z) * (X*Y) );
                            else
                                h_e_i = (x) + ( (y-lk) * (X) ) + ( (z) * (X*Y) );
                            h_r_test[h_r_i] += h_e[h_e_i];


                            if(z+lk >= Z)
                                h_e_i = (x) + ( (y) * (X) ) + ( (z-lk) * (X*Y) );
                            else
                                h_e_i = (x) + ( (y) * (X) ) + ( (z+lk) * (X*Y) );
                            h_r_test[h_r_i] += h_e[h_e_i];

                            if(z-lk < 0)
                                h_e_i = (x) + ( (y) * (X) ) + ( (z+lk) * (X*Y) );
                            else
                                h_e_i = (x) + ( (y) * (X) ) + ( (z-lk) * (X*Y) );
                            h_r_test[h_r_i] += h_e[h_e_i];

                        }  
                }
                //printf("\n");
            }
            //printf("-----\n\n");
        }

        for (int i = 0; i < tam; i++) 
        {
            h_e[i] = h_r_test[i];
        }

    }


    hipEvent_t start, stop;
    hipEventCreate (&start);
    hipEventCreate (&stop);
    hipEventRecord (start, 0); 

    /******************
    *** Kernel Call ***
    *******************/
    //_3Dstencil_global<<<blks,th_p_blk>>>(d_e,d_r,X,Y,Z);
    _3Dstencil_sharedMemory<<<grid_dim,block_dim,sharedSize>>>(d_e,d_r,X,Y,Z,k);

    hipError_t err = hipSuccess;
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch _3Dstencil_global kernel (error code %s)!\n", hipGetErrorString(err));
    }
    /******************
    *** Kernel Call ***
    *******************/



    hipDeviceSynchronize();
    hipEventRecord (stop, 0);
    hipEventSynchronize (stop);
    float elapsedTime;
    hipEventElapsedTime (&elapsedTime, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    
    Ticks[1] = clock();
    double Tempo = (Ticks[1] - Ticks[0]) * 1000.0 / CLOCKS_PER_SEC;
    printf("X %d || Y %d \nBX %d || BY %d\nGX %d || GY %d\nZ %d \n",X,Y,BX,BY,GX,GY,Z);
    printf ("[%d,%.5f,%.5f],\n", tam,elapsedTime,Tempo);
 
    hipMemcpy(h_r, d_r, size, hipMemcpyDeviceToHost);

    bool certo=true;
    //printf("threads/blk %d -- blocks %d\n",th_p_blk,blks);
    for (int i = 0; i < 256; i++){
        //printf("%d - %d\n",h_z_res[i],h_z[i]);
        if(h_r_test[i] != h_r[i])
          certo=false;
    }
    if(!certo)
    printf("\n*****\n certo = %s\n*****\n", certo ? "true" : "false");

    hipFree(d_e);
    hipFree(d_r);
    std::free(h_e);
    std::free(h_r);
    std::free(h_r_test);

    return 0;
} /* main */

