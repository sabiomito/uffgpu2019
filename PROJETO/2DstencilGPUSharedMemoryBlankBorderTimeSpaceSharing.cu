
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <unistd.h>
#include <stdlib.h>
#include <math.h>
using namespace std;

__device__ void _2Dstencil_(int *d_e,int *d_r,float* c_coeff,int X,int Y,int k, int x, int y,int GX,int Gx,int Gy)
{     
    int h_e_i;
    int h_r_i = x + ( y * (X) );
    h_e_i = h_r_i;
    int temp = d_e[h_r_i];
    temp *= c_coeff[0];
    for(int lk =1;lk<(k/2)+1;lk++)
    {
        h_e_i = (x+lk) + ( (y) * (X) );
        temp += d_e[h_e_i]*c_coeff[lk];

        h_e_i = (x-lk) + ( (y) * (X) );
        temp += d_e[h_e_i]*c_coeff[lk];

        h_e_i = (x) + ( (y+lk) * (X) );
        temp += d_e[h_e_i]*c_coeff[lk];

        h_e_i = (x) + ( (y-lk) * (X) );
        temp += d_e[h_e_i]*c_coeff[lk];
    }
     h_r_i = Gx + ( (Gy) * (GX) );
    d_r[h_r_i] = temp;    
}
__global__ void _2Dstencil_global(int *d_e,int *d_r,float *c_coeff,int X,int Y,int k,int times){

    int x,y;//,h_e_i,h_r_i,Xs,Ys,Dx,Dy;
    x = threadIdx.x + (blockIdx.x*blockDim.x);
    y = threadIdx.y + (blockIdx.y*blockDim.y);
    int k2 = k/2*times;
    extern __shared__ int shared[];
    
    int blockThreadIndex = threadIdx.x + threadIdx.y*blockDim.x;
    // Xs = threadIdx.x;
    // Ys = threadIdx.y;
    int Dx = blockDim.x+(k*times);
    int Dy = blockDim.y+(k*times);
    int sharedTam = Dx*Dy;
    int * sharedRes = &shared[sharedTam];
    
    for(int stride=blockThreadIndex;stride<sharedTam;stride+=(blockDim.x*blockDim.y))
    {
        int globalIdx = (blockIdx.x*blockDim.x)-k2+stride%Dx + ((blockIdx.y*blockDim.y)-k2+stride/Dx)*X;
        if(globalIdx > 0 && (blockIdx.x*blockDim.x)-k2+stride%Dx < X && ((blockIdx.y*blockDim.y)-k2+stride/Dx)<Y)
            shared[stride] = d_e[globalIdx];
        else
            shared[stride] = 0;
       
    }
    __syncthreads();
    for(int t=times-1;t>0;t--)
    {  
        //_2Dstencil_(shared,sharedRes,c_coeff,Dx,Dy,k,threadIdx.x+k2,threadIdx.y+k2,Dx,threadIdx.x+k2,threadIdx.y+k2);
        int tDx = blockDim.x+(t*k);
        int tDy = blockDim.y+(t*k);
        int tk2 = (times-t)*k/2;
        // int tDx = blockDim.x+(1*k);
        // int tDy = blockDim.y+(1*k);
        // int tk2 = (1)*k/2;
        int tSharedTam = tDx * tDy;
        for(int stride=blockThreadIndex;stride<tSharedTam;stride+=(blockDim.x*blockDim.y))
        {
            _2Dstencil_(shared,sharedRes,c_coeff,Dx,Dy,k,(stride%tDx)+tk2,(stride/tDx)+tk2,Dx,(stride%tDx)+tk2,(stride/tDx)+tk2);
        }
        __syncthreads();
        for(int stride=blockThreadIndex;stride<sharedTam;stride+=(blockDim.x*blockDim.y))
        {
            shared[stride]=sharedRes[stride];
        }
        __syncthreads();
   }
   
    _2Dstencil_(shared,d_r,c_coeff,Dx,Dy,k,threadIdx.x+k2,threadIdx.y+k2,X,x,y);
   
    // for(int stride=blockThreadIndex;stride<sharedTam;stride+=(blockDim.x*blockDim.y))
    // {
    //     int globalIdx = (blockIdx.x*blockDim.x)-k2+stride%Dx + ((blockIdx.y*blockDim.y)-k2+stride/Dx)*X;
    //     if(globalIdx > 0 && (blockIdx.x*blockDim.x)-k2+stride%Dx < X && ((blockIdx.y*blockDim.y)-k2+stride/Dx)<Y)
    //     d_r[globalIdx] = sharedRes[stride];
       
    // }
        
}


int main(int argc, char* argv[]) {

int *h_e,*h_r;
int *d_e, *d_r;
int size,tam,sharedSize,sharedTam;
int X=32;
int Y=32;
int k=4;
int times = 1;
int BX=32;
int BY=32;
int GX=1;
int GY=1;
float *c_coeff,*d_c_coeff;
if(argc > 1)
{
    X = atoi(argv[1]);
    Y = X;
}
if(argc > 2)
{
    k = atoi(argv[2]);
}

if(argc > 3)
{
    times = atoi(argv[3]);
}


if(X>32)
{
    GX = ceil((float)X/(float)32);
    BX = 32;
}
if(Y>32)
{
    GY = ceil((float)Y/(float)32);
    BY = 32;
}
    

dim3 block_dim(BX,BY,1);
dim3 grid_dim(GX,GY,1);
//sharedSize = ((block_dim.x+k)*(block_dim.y+k))*sizeof(int);
sharedSize = ((block_dim.x+(k*times))*(block_dim.y+(k*times)))*sizeof(int)*2;
//sharedTam = ((block_dim.x+(k*2))*(block_dim.y+(k*2)));
size = X * Y * sizeof(int);
tam = X * Y;


h_e = (int*) malloc(size);
h_r = (int*) malloc(size);
c_coeff = (float*)malloc((k/2+1)*sizeof(float));
hipMalloc(&d_e, size);
hipMalloc(&d_r, size);
hipMalloc(&d_c_coeff,(k/2+1)*sizeof(float));

printf("\n coefs \n");
for(int i=0;i<(k/2+1);i++)
{
    c_coeff[i]=(float)((k/2+1)-i)/(float)(k/2+1);
   
}
for(int i=0;i<(k/2+1);i++)
{
    printf(" %f",c_coeff[i]);
}
printf("\n coefs \n");


FILE *arq;
arq = fopen("entrada.txt", "rt");
for(int i=0;i<X;i++)
    for(int j=0;j<Y;j++)
        fscanf(arq," %d",&h_e[i+j*X]);
fclose(arq);


/* Copy vectors from host memory to device memory */
hipMemcpy(d_e, h_e, size, hipMemcpyHostToDevice);
hipMemcpy(d_c_coeff, c_coeff, (k/2+1)*sizeof(float), hipMemcpyHostToDevice);


hipEvent_t start, stop;
hipEventCreate (&start);
hipEventCreate (&stop);
hipEventRecord (start, 0); 

/******************
*** Kernel Call ***
*******************/
//_3Dstencil_global<<<blks,th_p_blk>>>(d_e,d_r,X,Y,Z);
_2Dstencil_global<<<grid_dim,block_dim,sharedSize>>>(d_e,d_r,d_c_coeff,X,Y,k,times);

hipError_t err = hipSuccess;
err = hipGetLastError();
if (err != hipSuccess)
{
    fprintf(stderr, "Failed to launch _3Dstencil_global kernel (error code %s)!\n", hipGetErrorString(err));
}
/******************
*** Kernel Call ***
*******************/



    hipDeviceSynchronize();
    hipEventRecord (stop, 0);
    hipEventSynchronize (stop);
    float elapsedTime;
    hipEventElapsedTime (&elapsedTime, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);


    printf("X %d || Y %d \nBX %d || BY %d \n",X,Y,BX,BY);
    printf ("[%d,%.5f],\n", tam,elapsedTime);
 
    hipMemcpy(h_r, d_r, size, hipMemcpyDeviceToHost);

    
arq = fopen("resultado.txt", "wt");
for(int i=0;i<X;i++)
{
    for(int j=0;j<Y;j++)
    {
      fprintf(arq," %d",h_r[i+j*X]);
    }
    fprintf(arq,"\n");
}
fclose(arq);


    hipFree(d_e);
    hipFree(d_r);
    hipFree(d_c_coeff);
    std::free(h_e);
    std::free(h_r);
    std::free(c_coeff);

    return 0;
} /* main */



/*
for(int lk = 1;lk<(k/2)+1;lk++)
    {
        if(x+lk < X)
        {
            if((x+lk)/Dx == blockIdx.x)
            {
                h_e_i = ((x+lk)%Dx) + ( (Ys) * (Dx) );
                temp += shared[h_e_i]*c_coeff[lk];
            }else
            {
                h_e_i = (x+lk) + ( (y) * (X) );
                temp += d_e[h_e_i]*c_coeff[lk];
            }
            
        }
        if(x-lk >= 0)
        {
            if((x-lk)/Dx == blockIdx.x)
            {
                h_e_i = ((x-lk)%Dx) + ( (Ys) * (Dx) );
                temp += shared[h_e_i]*c_coeff[lk];
            }
            else
            {
                h_e_i = (x-lk) + ( (y) * (X) );
                temp += d_e[h_e_i]*c_coeff[lk];
            }
               
        }
        if(y+lk < Y)
        {
            if((y+lk)/Dy == blockIdx.y)
            {
                h_e_i = ((Xs) + ( ((y+lk)%Dy) * (Dx) ));
                temp += shared[h_e_i]*c_coeff[lk];
            }
            else
            {
                h_e_i = (x) + ( (y+lk) * (X) );
                temp += d_e[h_e_i]*c_coeff[lk];
            }
        }
        if(y-lk >= 0)
        {
            if((y-lk)/Dy == blockIdx.y)
            {
                h_e_i = ((Xs) + ( ((y-lk)%Dy) * (Dx) ));
                temp += shared[h_e_i]*c_coeff[lk];
            }
            else
            {
                h_e_i = (x) + ( (y-lk) * (X) );
                temp += d_e[h_e_i]*c_coeff[lk];
            }
        }
    }
    d_r[h_r_i] = temp;  
*/