
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <unistd.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>
#include <string>
/*
Instruções
COMPILAR -->  nvcc 2DstencilGPUSharedMemoryBlankBorderTimeSpaceSharingOpencvKarma.cu -o go `pkg-config --cflags --libs opencv` -w
EXECUTAR --> ./go DOMAIN_DIMS STENCIL_ORDER SPACE_TIME_BLOCK_TIMES BLOCK_DIM_X BLOCK_DIM_Y
*/



using namespace std;

//===> CONSTANTES karma model <===//
#ifndef MODEL_WIDTH
#define MODEL_WIDTH 0
#endif
#define Eh 3.0f
#define En 1.0f
#define Re 0.6f
#define tauE 5.0f
#define tauN 250.0f
#define gam 0.001f
#define East 1.5415f
#define DT 0.05f
#define DX (12.0f / MODEL_WIDTH)

/*
Função somente da GPU que recebe os parametros para o calculo de um stencil
d_e - dado de entrada
d_r - dado de saida
d_v - campo que deve ser atualizado
c_coeff - variável utilizada para armazenar o valores dos coeficcientes do stencil (utilizada apenas na versão com stencil simples usado anteriormente)
X - Y - Dimensões das estruturas de entrada
k - ordem do stencil
x -y - posição do centro do stencil na estrutura de entrada
GX - Dimensão horizontal da estrutura do dado de saída
Gx - Gy posição do centro do stencil na estrutura de saida
*/
__device__ void _2Dstencil_(float *d_e, float *d_r, float *d_v, int X, int x, int y, int GX, int Gx, int Gy)
{
    int h_e_i = x + (y * (X));
    float temp = d_e[h_e_i];
    
    float rv = d_v[h_e_i];


    float Rn = (1.0f / (1.0f - expf(-Re))) - rv;
    float p = (temp > En) * 1.0f;
    float dv = (Rn * p - (1.0f - p) * rv) / tauN;
    float Dn = rv * rv;
    float hE = (1.0f - tanh(temp - Eh)) * temp * temp / 2.0f;
    float du = (((East - Dn) * hE) - temp) / tauE;

    float xlapr = d_e[(x + 1) + ((y) * (X))] - temp;
    float xlapl = temp - d_e[(x - 1) + ((y) * (X))];
    float xlapf = d_e[(x) + ((y + 1) * (X))] - temp;
    float xlapb = temp - d_e[(x) + ((y - 1) * (X))];

    float lap = xlapr - xlapl + xlapf - xlapb;

   
    temp = (temp + (du * DT) + (lap * DT * gam / (DX * DX)));


    d_v[h_e_i] = rv + dv * DT;
    h_e_i = Gx + ((Gy) * (GX));
    d_r[h_e_i] = temp;
    

}
/*
função chamada pelo host que controla as cópias e a ordem do calculo dos stencils bem como a carga para cada thread
*/
__global__ void _2Dstencil_global(float *d_e, float *d_r, float *d_v, int X, int Y, int times)
{

    int x, y; //,h_e_i,h_r_i,Xs,Ys,Dx,Dy;
    x = threadIdx.x + (blockIdx.x * blockDim.x);
    y = threadIdx.y + (blockIdx.y * blockDim.y);
    extern __shared__ float sharedOrig[];

    int blockThreadIndex = threadIdx.x + threadIdx.y * blockDim.x;
    // Xs = threadIdx.x;
    // Ys = threadIdx.y;
    int Dx = blockDim.x + (2 * times);
    int Dy = blockDim.y + (2 * times);
    int sharedTam = Dx * Dy;

    float * shared = sharedOrig;
    float * sharedRes = shared + sharedTam;
    float * sharedV = sharedRes + sharedTam; 

    //float * sharedRes = &shared[sharedTam];
    //float *sharedV = &sharedRes[sharedTam];

    /*
    Copia o Tile de memória compartilhada necessária para a configuração de tempo desejada
    Stride é utilizado pois a quantidade de elementos a serem copiados é sempre maior que a quantidade de threads
    As bordas
    */
    for (int stride = blockThreadIndex; stride < sharedTam; stride += (blockDim.x * blockDim.y))
    {
         int sharedIdxX = stride % Dx;
         int sharedIdxY = int(stride / Dx);
         int globalIdxX =(blockIdx.x * blockDim.x) + sharedIdxX - times;
         int globalIdxY =(blockIdx.y * blockDim.y) + sharedIdxY - times;
         //int globalIdx = globalIdxX + (globalIdxX < 0) - (globalIdxX >= X)  +  (globalIdxY + (globalIdxY < 0) - (globalIdxY >= Y)) * X;
         int globalIdx = globalIdxX*(!(globalIdxX < 0 || globalIdxX >= X)) + (globalIdxX + (globalIdxX < 0) - (globalIdxX >= X))*((globalIdxX < 0 || globalIdxX >= X))  +   (globalIdxY*(!(globalIdxY < 0 || globalIdxY >= Y)) + (globalIdxY + (globalIdxY < 0) - (globalIdxY >= Y))*((globalIdxY < 0 || globalIdxY >= Y))) * X;
     
        shared[stride] = d_e[globalIdx];
        sharedV[stride] = d_v[globalIdx];
    }

    __syncthreads();

    /*
    Envia pra ser calculado todos os elementos além do ultimo instante de tempo
    */
    for (int t = 1; t < times; t++)
    {
        //_2Dstencil_(shared,sharedRes,c_coeff,Dx,Dy,k,threadIdx.x+k2,threadIdx.y+k2,Dx,threadIdx.x+k2,threadIdx.y+k2);
        int tDx = blockDim.x + ((times - t) * 2);
        int tDy = blockDim.y + ((times - t) * 2);
        int tk2 = (t);
        int tSharedTam = tDx * tDy;
        for (int stride = blockThreadIndex; stride < tSharedTam; stride += (blockDim.x * blockDim.y))
        {
            _2Dstencil_(shared, sharedRes, sharedV, Dx, (stride % tDx) + tk2, (int(stride / tDx)) + tk2, Dx, (stride % tDx) + tk2, (int(stride / tDx)) + tk2);
        }

        float * temp = shared;
        shared = sharedRes;
        sharedRes = temp;
        __syncthreads();
    }
    /*
    Envia pra ser calculado todos os elementos do ultimo instante de tempo
   */
   
    _2Dstencil_(shared, d_r, sharedV, Dx, ((x%(blockDim.x))+times), ((y%(blockDim.y))+times), X, x, y);

     __syncthreads();
    
    int sharedIdx = ((x%(blockDim.x))+times) + ((y%(blockDim.y))+times)*Dx;
    int globalIdx = x + y * X;
    d_v[globalIdx] = sharedV[sharedIdx];
}

int main(int argc, char *argv[])
{
    /*
    Declarações e valores padroes
    */
    float *h_e, *h_r, *h_v;
    float *d_e, *d_r, *d_v;
    int size, sharedSize;
    int X = 32;
    int Y = 32;
    int times = 1,globalTimes = 1;
    int BX = 32;
    int BY = 32;
    int GX = 1;
    int GY = 1;

    /*
    Obtenção dos parâmetros de entrada
    */
    if (argc > 1)
    {
        X = atoi(argv[1]);
        Y = X;
    }
    if (argc > 2)
    {
        times = atoi(argv[2]);
    }

    if (argc > 3)
    {
        globalTimes = atoi(argv[3]);
    }

    if (X > 32)
    {
        if (argc > 4)
            BX = atoi(argv[4]);
        GX = ceil((float)X / (float)BX);
        BX = 32;
    }
    if (Y > 32)
    {
        if (argc > 5)
            BY = atoi(argv[5]);
        GY = ceil((float)Y / (float)BY);
        BY = 32;
    }

    /*
    Allocações de memória e configuração dos blocos e grid
    */
    dim3 block_dim(BX, BY, 1);
    dim3 grid_dim(GX, GY, 1);
    //sharedSize = ((block_dim.x+k)*(block_dim.y+k))*sizeof(int);
    sharedSize = ((block_dim.x + (2 * times)) * (block_dim.y + (2 * times))) * sizeof(float) * 3;
    //sharedTam = ((block_dim.x+(k*2))*(block_dim.y+(k*2)));
    size = X * Y * sizeof(float);
    //tam = X * Y;

    h_e = (float *)malloc(size);
    h_r = (float *)malloc(size);
    h_v = (float *)malloc(size);
    hipMalloc(&d_e, size);
    hipMalloc(&d_r, size);
    hipMalloc(&d_v, size);

//Copia os dados do campo e envia para a GPU e inicializa o dominio de entrada

        


    FILE *arq;
    arq = fopen("entrada.txt", "rt");
    for (int i = 0; i < X; i++)
        for (int j = 0; j < Y; j++)
        {
            h_v[i + j * X] =0.5f;
            int temp;
            fscanf(arq," %d",&temp);
            h_e[i + j * X] = temp;
        }

    fclose(arq);
    hipMemcpy(d_v, h_v, size, hipMemcpyHostToDevice);
   
    /* 
    Copy vectors from host memory to device memory
    Copia os dados da entrada de volta a GPU
        */
    hipMemcpy(d_e, h_e, size, hipMemcpyHostToDevice);
    
    /*
    Começa o Timer
    */
    hipDeviceSynchronize();
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    /******************
    *** Kernel Call ***
    *******************/
    //_3Dstencil_global<<<blks,th_p_blk>>>(d_e,d_r,X,Y,Z);
    /*
    Executa o kernel
    */
    for(int i=0; i<globalTimes/times; i ++)
    {
        _2Dstencil_global<<<grid_dim, block_dim, sharedSize>>>(d_e, d_r, d_v, X, Y, times);
        float * temp = d_e;
        d_e = d_r;
        d_r = temp;
    }
    

    /*
    Identifica possíveis erros
    */
    hipError_t err = hipSuccess;
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch _3Dstencil_global kernel (error code %s)!\n", hipGetErrorString(err));
    }
    /******************
    *** Kernel Call ***
    *******************/

    hipDeviceSynchronize();
    /*
    Para o Timer
    */
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    printf ("[%d,%.5f]",times,elapsedTime);

    // arq = fopen("TempoExecucaoBlocking12000VariandoTimes.txt", "a");
    // //printf("X %d || Y %d \nBX %d || BY %d \n",X,Y,BX,BY);
    // // float sharedTime = 0.0;
    // //     if(MODEL_WIDTH == 64)
    // //         sharedTime = 108.41396;
    // //     if(MODEL_WIDTH == 96)
    // //         sharedTime = 89.01120;
    // //     if(MODEL_WIDTH == 128)
    // //         sharedTime = 95.11117;
    // //     if(MODEL_WIDTH == 160)
    // //         sharedTime = 113.37702;
    // //     if(MODEL_WIDTH == 192)
    // //         sharedTime = 101.13689;
    // //     if(MODEL_WIDTH == 224)
    // //         sharedTime = 154.31091;
    // //     if(MODEL_WIDTH == 256)
    // //         sharedTime = 186.73097;
    // //     if(MODEL_WIDTH == 288)
    // //         sharedTime = 218.92052;
    // //     if(MODEL_WIDTH == 320)
    // //         sharedTime = 232.28406;
    // //     if(MODEL_WIDTH == 352)
    // //         sharedTime = 295.31876;
    // //     if(MODEL_WIDTH == 384)
    // //         sharedTime = 304.94522;
    // //     if(MODEL_WIDTH == 416)
    // //         sharedTime = 385.76855;
    // //     if(MODEL_WIDTH == 448)
    // //         sharedTime = 570.88287;
    // //     if(MODEL_WIDTH == 480)
    // //         sharedTime = 701.02271;
    // //     if(MODEL_WIDTH == 512)
    // //         sharedTime = 768.65991;
    // //     if(MODEL_WIDTH == 544)
    // //         sharedTime = 881.91882;
    // //     if(MODEL_WIDTH == 576)
    // //         sharedTime = 979.11212;
    // //     if(MODEL_WIDTH == 608)
    // //         sharedTime = 1082.10193;
    // //     if(MODEL_WIDTH == 640)
    // //         sharedTime = 1188.77576;
    // //     if(MODEL_WIDTH == 672)
    // //         sharedTime = 1316.50024;
    // //     if(MODEL_WIDTH == 704)
    // //         sharedTime = 1436.11035;
    // //     if(MODEL_WIDTH == 736)
    // //         sharedTime = 1532.38489;
    // //     if(MODEL_WIDTH == 768)
    // //         sharedTime = 1576.36401;

    // fprintf (arq,"(%d,%.5f),\n",times,elapsedTime);//,sharedTime);
    // fclose(arq);
    /*
    Copia o resultado para a imagem de visualização
    */
    hipMemcpy(h_r, d_e, size, hipMemcpyDeviceToHost);
    arq = fopen("resultado.txt", "wt");
    for (int i = 0; i < X; i++)
    {
        for (int j = 0; j < Y; j++)
        {
            fprintf(arq," %6.4f",h_r[i+j*X]);
        }
        fprintf(arq,"\n");
    }
    fclose(arq);

    hipFree(d_e);
    hipFree(d_r);
    std::free(h_e);
    std::free(h_r);

    return 0;
} /* main */
