
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>
#include <iostream>
#include <string>



//===> FINITE DIFFERENCES PARAMETERS <===//
#define DT 0.05f                   //->Time in milliseconds
#define DX ( 12.0f / MODELSIZE_X ) //->Displacement in x
#define DY ( 12.0f / MODELSIZE_Y ) //->Displacement in y

//===> CONSTANTES <===//
#define Eh 3.0f
#define En 1.0f
#define Re 0.6f
#define tauE 5.0f
#define tauN 250.0f
#define gam 0.001f
#define East 1.5415f

//===> INITIAL CONDITIONS <===//
#define v0 0.5f
#define VOLT0 3.0f

//==> DISCRETE DOMAIN <==//
#ifndef MODEL_WIDTH
#define MODEL_WIDTH 0
#endif

#define MODELSIZE_X (MODEL_WIDTH)
#define MODELSIZE_Y (MODEL_WIDTH)
#define MODELSIZE_Z 1
#define MODELSIZE2D ( MODELSIZE_X*MODELSIZE_Y )

//==> CUDA THREAD BLOCK <==//
//#define TILESIZE   32
//#define BLOCKDIM_X ( TILESIZE )
//#define BLOCKDIM_Y ( TILESIZE )

#ifndef BLOCKDIM_X
#define BLOCKDIM_X 32
#endif

#ifndef BLOCKDIM_Y
#define BLOCKDIM_Y 32
#endif

#define BLOCKDIM_Z 1
#define BLOCKDIM2D ( BLOCKDIM_X*BLOCKDIM_Y )

//==> CUDA GRID <==//
#define GRIDDIM_X ( ( MODELSIZE_X / BLOCKDIM_X ) + ( ( MODELSIZE_X % BLOCKDIM_X ) > 0 ) )
#define GRIDDIM_Y ( ( MODELSIZE_Y / BLOCKDIM_Y ) + ( ( MODELSIZE_Y % BLOCKDIM_Y ) > 0 ) )
#define GRIDDIM_Z 1


//////////////////////////////////////////////////////////////////////////
static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))
//////////////////////////////////////////////////////////////////////////


__global__ void timeStep( const float *voltIN, float *v, float *voltOUT )
{
    int x = blockIdx.x*BLOCKDIM_X + threadIdx.x;
    int y = blockIdx.y*BLOCKDIM_Y + threadIdx.y;

    __shared__ float U[BLOCKDIM_X+2][BLOCKDIM_Y+2];

    if ( x < MODELSIZE_X && y < MODELSIZE_Y ) 
    {
        //
        int idx = y*MODELSIZE_X + x;

        int i = threadIdx.x+1;
        int j = threadIdx.y+1;

        U[i][j] = voltIN[idx];

        __syncthreads();

        float rv = v[idx];

        if ( threadIdx.y == 0 )
        U[i][0] = voltIN[(idx - ((y>0)-(y==0))*MODELSIZE_X)];
        else if ( threadIdx.y == (BLOCKDIM_Y-1) )
        U[i][(BLOCKDIM_Y+1)] = voltIN[(idx + ((y<MODELSIZE_Y-1)-(y==MODELSIZE_Y-1))*MODELSIZE_X)];

        if ( threadIdx.x == 0 )
        U[0][j] = voltIN[(idx - (x>0) + (x==0))];
        else if ( threadIdx.x == (BLOCKDIM_X-1) )
        U[(BLOCKDIM_X+1)][j] = voltIN[(idx + (x<MODELSIZE_X-1)-(x==MODELSIZE_X-1))];
        

        float Rn = ( 1.0f / ( 1.0f - expf(-Re) ) ) - rv;
        float p = ( U[i][j] > En ) * 1.0f;
        float dv = ( Rn * p - ( 1.0f - p ) * rv ) / tauN;
        float Dn = rv * rv;
        float hE = ( 1.0f - tanh(U[i][j] - Eh) ) * U[i][j] * U[i][j] / 2.0f;
        float du = ( ( ( East - Dn ) * hE ) - U[i][j] ) / tauE;

        float xlapr = U[i+1][j] - U[i][j];
        float xlapl = U[i][j]   - U[i-1][j];
        float xlapf = U[i][j+1] - U[i][j];
        float xlapb = U[i][j]   - U[i][j-1];

        float lap = xlapr - xlapl + xlapf - xlapb;

        voltOUT[idx] = ( U[i][j] + ( du * DT ) + ( lap * DT * gam / ( DX * DX ) ) );
        v[idx]       = rv + dv*DT;
    }
}

int main( int argc, char *argv[] )
{
    int nsteps = 3; //8000;
    // if ( argc > 1 ) 
    // {
    //     char *p;
    //     long conv = strtol(argv[1], &p, 10);
    //     //
    //     // Check for errors: e.g., the string does not represent an integer
    //     // or the integer is larger than int
    //     if (*p != '\0' || conv > INT_MAX) 
    //     {
    //         printf("Error with argument 1!");
    //         return 3;
    //     }
    //     else
    //     nsteps = int(conv/DT);
    // }
    if (argc > 1)
    {
        nsteps = atoi(argv[1]);
    }
    //
    hipEvent_t dstart,dstop;
    hipEventCreate( &dstart );
    hipEventCreate( &dstop );
    //
    long start, end;
    struct timeval timecheck;
    gettimeofday(&timecheck, NULL);
    start = (long)timecheck.tv_sec * 1000 + (long)timecheck.tv_usec / 1000;
    //
    float *hvolt, *hv;
    hvolt = (float*) malloc( MODELSIZE2D*sizeof(float) );
    hv    = (float*) malloc( MODELSIZE2D*sizeof(float) );

    // int x, y, idx;
    // for( y = 0; y < MODELSIZE_Y; y++ )
    // {
    //     for( x = 0; x < MODELSIZE_X; x++ )
    //     {
    //         idx = y*MODELSIZE_X + x;
    //         //
    //         hv[idx] = 0.5f;
    //         //
    //         if ( y < 10*(MODELSIZE_Y/20) && y > 8*(MODELSIZE_Y/20) && x < 10*(MODELSIZE_Y/20) &&  x > 8*(MODELSIZE_Y/20))
    //         hvolt[idx] = VOLT0;
    //         else
    //         hvolt[idx] = 0.0f;
    //         //
    //     }
    // }

     FILE *arq;
    arq = fopen("entrada.txt", "rt");
    for(int i=0;i<MODELSIZE_X;i++)
        for(int j=0;j<MODELSIZE_Y;j++)
        {
            hv[i+j*MODELSIZE_X] = 0.5f;
            int temp;
            fscanf(arq," %d",&temp);
            hvolt[i+j*MODELSIZE_X] = temp;
        }
            
    fclose(arq);

    // FILE *prof;
    // char fpname[100];
    // sprintf(fpname, "./profiles_%d_k2D_shared.csv",MODELSIZE_X);
    // prof = fopen(fpname,"w");
    // fprintf(prof,"index,timestep,P\n");
    // fprintf(prof,"0,%6.4f",0.0);
    // fclose(prof);


    dim3 point;
    //int pointIdx;
    point.x = MODELSIZE_X/2;
    point.y = MODELSIZE_Y/2;
    point.z = 0;
   // pointIdx = point.y*MODELSIZE_X + point.x;


    //fprintf(prof,",%6.4f\n",hvolt[pointIdx]);


    float *dvoltA, *dvoltB, *dv;
    HANDLE_ERROR( hipMalloc( (void**)&dvoltA, MODELSIZE2D*sizeof(float) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dvoltB, MODELSIZE2D*sizeof(float) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dv    , MODELSIZE2D*sizeof(float) ) );

    HANDLE_ERROR( hipMemcpy( dvoltA, hvolt, MODELSIZE2D*sizeof(float), hipMemcpyHostToDevice ) );
    HANDLE_ERROR( hipMemcpy( dvoltB, hvolt, MODELSIZE2D*sizeof(float), hipMemcpyHostToDevice ) );
    HANDLE_ERROR( hipMemcpy( dv    , hv   , MODELSIZE2D*sizeof(float), hipMemcpyHostToDevice ) );

    free( hv );

    dim3 blocks(GRIDDIM_X,GRIDDIM_Y,GRIDDIM_Z);
    dim3 threads(BLOCKDIM_X,BLOCKDIM_Y,BLOCKDIM_Z);

    //int nsamples = (nsteps >= 2000)*2000 + (nsteps < 2000)*nsteps;
    //int j = nsteps/nsamples;
    hipDeviceSynchronize();
    hipEventRecord( dstart, 0 );
    int i=0;
    for (i = 0; i < nsteps; i++ ) 
    {
        if ( (i%2) == 0 ) //==> EVEN
        timeStep<<<blocks, threads>>>( dvoltA, dv, dvoltB );
        else              //==> ODD
        timeStep<<<blocks, threads>>>( dvoltB, dv, dvoltA );
        //
        /*if ( (i%j) == 0 ) {
        if ( (i%2) == 0 ) //==> EVEN
        HANDLE_ERROR( cudaMemcpy( hvolt, dvoltB, MODELSIZE3D*sizeof(float), cudaMemcpyDeviceToHost ) );
        else              //==> ODD
        HANDLE_ERROR( cudaMemcpy( hvolt, dvoltA, MODELSIZE3D*sizeof(float), cudaMemcpyDeviceToHost ) );
        //
        fprintf(prof,"%d,%6.4f,%6.4f\n", (i+1), ((i+1)*DT), hvolt[pointIdx]);
        }*/
        hipError_t err = hipSuccess;
        err = hipGetLastError();
        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to launch _3Dstencil_global kernel (error code %s)!\n", hipGetErrorString(err));
        }
    }
    hipDeviceSynchronize();
    hipEventRecord( dstop, 0 );
    hipEventSynchronize ( dstop );
    float elapsed;
    hipEventElapsedTime( &elapsed, dstart, dstop );
    //printf("GPU elapsed time: %f s (%f milliseconds)\n", (elapsed/1000.0), elapsed);

    //arq = fopen("TempoExecucaoOrig12000.txt", "a");
    //printf("X %d || Y %d \nBX %d || BY %d \n",X,Y,BX,BY);
        //fprintf (arq,"[%d,%.5f],\n",MODEL_WIDTH,elapsed);
        printf ("[%d,%.5f]",0,elapsed);
    //fclose(arq);



    // if ( (i%2) == 0 )
    // HANDLE_ERROR( cudaMemcpy( hvolt, dvoltA, MODELSIZE2D*sizeof(float), cudaMemcpyDeviceToHost ) );
    // else
    // HANDLE_ERROR( cudaMemcpy( hvolt, dvoltB, MODELSIZE2D*sizeof(float), cudaMemcpyDeviceToHost ) );


    // arq = fopen("resultado.txt", "wt");
    // for(int i=0;i<MODELSIZE_X;i++)
    // {
    //     for(int j=0;j<MODELSIZE_Y;j++)
    //     {
    //         fprintf(arq," %6.4f",hvolt[i+j*MODELSIZE_X]);
    //     }
    //     fprintf(arq,"\n");
    // }
    // fclose(arq);


    //fclose( prof );
    free( hvolt );
    hipFree( dvoltA );
    hipFree( dvoltB );
    hipFree( dv );
    //
    // cudaDeviceSynchronize();
    // gettimeofday(&timecheck, NULL);
    // end = (long)timecheck.tv_sec * 1000 + (long)timecheck.tv_usec / 1000;
    //printf("CPU elapsed time: %f s (%ld milliseconds)\n", ((end - start)/1000.0), (end - start));
    //
    hipEventDestroy( dstart );
    hipEventDestroy( dstop );
    hipDeviceReset();
    //
    return 0;
}
