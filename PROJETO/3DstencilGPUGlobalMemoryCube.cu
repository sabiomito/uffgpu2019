
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <unistd.h>
#include <stdlib.h>
#include <math.h>
using namespace std;

__device__ void _3Dstencil_(float *d_e,float *d_r,int X,int Y,int Z,int k, int x, int y,int z)
{     
    int h_r_i = x + ( y * (X) ) + ( z* (X*Y) );
    int h_e_i = h_r_i;
    d_r[h_r_i] = d_e[h_e_i];
    for(int lk =0;lk<(k/2);lk++)
    {
        h_e_i = (x+lk) + ( (y) * (X) ) + ( (z) * (X*Y) );
        d_r[h_r_i] += d_e[h_e_i];

        h_e_i = (x-lk) + ( (y) * (X) ) + ( (z) * (X*Y) );
        d_r[h_r_i] += d_e[h_e_i];

        h_e_i = (x) + ( (y+lk) * (X) ) + ( (z) * (X*Y) );
        d_r[h_r_i] += d_e[h_e_i];

        h_e_i = (x) + ( (y-lk) * (X) ) + ( (z) * (X*Y) );
        d_r[h_r_i] += d_e[h_e_i];


        h_e_i = (x) + ( (y) * (X) ) + ( (z+lk) * (X*Y) );
        d_r[h_r_i] += d_e[h_e_i];

        h_e_i = (x) + ( (y) * (X) ) + ( (z-lk) * (X*Y) );
        d_r[h_r_i] += d_e[h_e_i];
    }  
}
__global__ void _3Dstencil_SharedMemory_Cube(float *d_e,float *d_r,int X,int Y,int Z,int k,int t)
{
    int x,y,globalidx,localIdx;
    x = threadIdx.x + (blockIdx.x*blockDim.x);
    y = threadIdx.y + (blockIdx.y*blockDim.y);
    int k2=k/2;

    //conta para tamanho do cubo em cada dimensão DIM+t*k2

    extern __shared__ float cube[];
    for(int z=0;z<Z;z++)
    {
        globalidx = x + y*X + z*(X*Y);
        localIdx = threadIdx.x + threadIdx.y*blockDim.x + z*(blockDim.x*blockDim.y);
        cube[localIdx] = d_e[globalidx];
    }
    for(int z=k2;z<Z-k2;z++)
    {
        _3Dstencil_(d_e,d_r,X,Y,Z,k,x+k2,y+k2,z);
    }
}

/*
*argumentos
*1 - n_elementos
*2 - threads por bloco
*3 - n_blocos
*4 - print
*/
int main(int argc, char* argv[]) {

    float *h_e,*h_r,*h_r_test;
    float *d_e, *d_r;
    int size,tam,times,sharedSize;
    clock_t Ticks[2];

    

    

    times = 1;
    int X=8;
    int Y=8;
    int BX=8;
    int BY=8;
    int Z=4;
    int k=2;
    int GX=1;
    int GY=1;

    if(argc > 1)
    {
        X = atoi(argv[1]);
        BX=X;
    }
      
    if(argc > 2)
    {
        Y = atoi(argv[2]);
        BY = Y;
    }
      
    if(argc > 3)
      Z = atoi(argv[3]);
    if(argc > 4)
      k = atoi(argv[4]);

    if(X>32)
    {
        GX = ceil((float)X/(float)32);
        BX = 32;
    }
    if(Y>32)
    {
        GY = ceil((float)Y/(float)32);
        BY = 32;
    }
    
    
    dim3 block_dim(BX,BY,1);
    dim3 grid_dim(GX,GY,1);

    sharedSize = ((block_dim.x+k)*(block_dim.y+k)*(Z+k))*sizeof(float);
    size = (X+k) * (Y+k) * (Z+k) * sizeof(float);
    tam = X * Y * Z;

    X=X+k;
    Y=Y+k;
    Z=Z+k;


    h_e = (float*) malloc(size);
    h_r = (float*) malloc(size);
    h_r_test = (float*) malloc(size);
    hipMalloc(&d_e, size);
    hipMalloc(&d_r, size);


    for (int i = 0; i < size/sizeof(float); i++) {
        h_e[i] = (float)(rand()%9000)/100.0;
        h_r[i] = 0;
    }

    /* Copy vectors from host memory to device memory */
    hipMemcpy(d_e, h_e, size, hipMemcpyHostToDevice);
    hipMemcpy(d_r, h_r, size, hipMemcpyHostToDevice);
    int k2 = k/2;
    for(int t =0; t<times; t++)
    {

        for(int z=k2;  z<Z-k2;  z++)
        {
        
            for(int y=k2;  y<Y-k2;  y++)
            {
                for(int x=k2;  x<X-k2;  x++)
                {
                    
                    
                    int h_r_i = x + ( y * (X) ) + ( z* (X*Y) );
                        
                    int h_e_i = h_r_i;
                    h_r_test[h_r_i] = h_e[h_e_i];
                    for(int lk =0;lk<(k/2);lk++)
                        {
                            

                            
                            if(x+lk >= X)
                                h_e_i = (x-lk) + ( (y) * (X) ) + ( (z) * (X*Y) );
                            else
                                h_e_i = (x+lk) + ( (y) * (X) ) + ( (z) * (X*Y) );
                            h_r_test[h_r_i] += h_e[h_e_i];

                            if(x-lk < 0)
                                h_e_i = (x+lk) + ( (y) * (X) ) + ( (z) * (X*Y) );
                            else
                                h_e_i = (x-lk) + ( (y) * (X) ) + ( (z) * (X*Y) );
                            h_r_test[h_r_i] += h_e[h_e_i];


                            if(y+lk >= Y)
                                h_e_i = (x) + ( (y-lk) * (X) ) + ( (z) * (X*Y) );
                            else
                                h_e_i = (x) + ( (y+lk) * (X) ) + ( (z) * (X*Y) );
                            h_r_test[h_r_i] += h_e[h_e_i];

                            if(y-lk < 0)
                                h_e_i = (x) + ( (y+lk) * (X) ) + ( (z) * (X*Y) );
                            else
                                h_e_i = (x) + ( (y-lk) * (X) ) + ( (z) * (X*Y) );
                            h_r_test[h_r_i] += h_e[h_e_i];


                            if(z+lk >= Z)
                                h_e_i = (x) + ( (y) * (X) ) + ( (z-lk) * (X*Y) );
                            else
                                h_e_i = (x) + ( (y) * (X) ) + ( (z+lk) * (X*Y) );
                            h_r_test[h_r_i] += h_e[h_e_i];

                            if(z-lk < 0)
                                h_e_i = (x) + ( (y) * (X) ) + ( (z+lk) * (X*Y) );
                            else
                                h_e_i = (x) + ( (y) * (X) ) + ( (z-lk) * (X*Y) );
                            h_r_test[h_r_i] += h_e[h_e_i];

                        }  
                }
            }
            
        }

        for (int i = 0; i < tam; i++) 
        {
            h_e[i] = h_r_test[i];
        }

    }


    hipEvent_t start, stop;
    hipEventCreate (&start);
    hipEventCreate (&stop);
    hipEventRecord (start, 0); 

    /******************
    *** Kernel Call ***
    *******************/
    //_3Dstencil_global<<<blks,th_p_blk>>>(d_e,d_r,X,Y,Z);
    _3Dstencil_SharedMemory_Cube<<<grid_dim,block_dim,sharedSize>>>(d_e,d_r,X,Y,Z,k,times);

    hipError_t err = hipSuccess;
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch _3Dstencil_global kernel (error code %s)!\n", hipGetErrorString(err));
    }
    /******************
    *** Kernel Call ***
    *******************/



    hipDeviceSynchronize();
    hipEventRecord (stop, 0);
    hipEventSynchronize (stop);
    float elapsedTime;
    hipEventElapsedTime (&elapsedTime, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    
    Ticks[1] = clock();
    double Tempo = (Ticks[1] - Ticks[0]) * 1000.0 / CLOCKS_PER_SEC;
    printf("X %d || Y %d \nBX %d || BY %d\nZ %d \n",X,Y,BX,BY,Z);
    printf ("[%d,%.5f,%lf],\n", tam,elapsedTime,Tempo/1000.0);
 
    hipMemcpy(h_r, d_r, size, hipMemcpyDeviceToHost);

    bool certo=true;
    //printf("threads/blk %d -- blocks %d\n",th_p_blk,blks);
    for (int i = 0; i < tam; i++){
        //printf("%d - %d\n",h_z_res[i],h_z[i]);
        if(h_r_test[i] != h_r[i])
          certo=false;
    }
    if(!certo)
    printf("\n*****\n certo = %s\n*****\n", certo ? "true" : "false");

    hipFree(d_e);
    hipFree(d_r);
    std::free(h_e);
    std::free(h_r);
    std::free(h_r_test);

    return 0;
} /* main */

