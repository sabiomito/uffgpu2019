
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <unistd.h>
#include <stdlib.h>
#include <math.h>
using namespace std;

__global__ void _copy_dr_to_de(int *d_e,int *d_r,int X,int Y,int k2){
    int x,y;
    x = threadIdx.x + (blockIdx.x*blockDim.x)+k2;
    y = threadIdx.y + (blockIdx.y*blockDim.y)+k2;
    int h_r_i = x + ( y * (X) );
    if(x<X && y<Y)
    d_e[h_r_i] = d_r[h_r_i];
}
__global__ void _2Dstencil_global(int *d_e,int *d_r,float *c_coeff,int X,int Y,int k){

    int x,y;
    int k2=k/2;
    x = threadIdx.x + (blockIdx.x*blockDim.x);
    y = threadIdx.y + (blockIdx.y*blockDim.y);

    x+=k2;
    y+=k2;
    int h_r_i = x + ( y * (X) );
    int h_e_i = h_r_i;
    d_r[h_r_i] = d_e[h_e_i]*c_coeff[0];
    for(int lk =1;lk<(k/2)+1;lk++)
    {
        h_e_i = (x+lk) + ( (y) * (X) );
        d_r[h_r_i] += d_e[h_e_i]*c_coeff[lk];

        h_e_i = (x-lk) + ( (y) * (X) );
        d_r[h_r_i] += d_e[h_e_i]*c_coeff[lk];

        h_e_i = (x) + ( (y+lk) * (X) );
        d_r[h_r_i] += d_e[h_e_i]*c_coeff[lk];

        h_e_i = (x) + ( (y-lk) * (X) );
        d_r[h_r_i] += d_e[h_e_i]*c_coeff[lk];
    }
}


int main(int argc, char* argv[]) {

int *h_e,*h_r;
int *d_e, *d_r;
int size,tam;
int X=32;
int Y=32;
int k=4;
int times = 1;
int BX=32;
int BY=32;
int GX=1;
int GY=1;
float *c_coeff,*d_c_coeff;
if(argc > 1)
{
    X = atoi(argv[1]);
    Y = X;
}
if(argc > 2)
{
    k = atoi(argv[2]);
}

if(argc > 3)
{
    times = atoi(argv[3]);
}


if(X>32)
{
    GX = ceil((float)X/(float)32);
    BX = 32;
}
if(Y>32)
{
    GY = ceil((float)Y/(float)32);
    BY = 32;
}
    
int k2=k/2;
dim3 block_dim(BX,BY,1);
dim3 grid_dim(GX,GY,1);

size = (X+k) * (Y+k) * sizeof(int);
tam = X * Y;



h_e = (int*) malloc(size);
h_r = (int*) malloc(size);
c_coeff = (float*)malloc((k/2+1)*sizeof(float));
hipMalloc(&d_e, size);
hipMalloc(&d_r, size);
hipMalloc(&d_c_coeff,(k/2+1)*sizeof(float));

printf("\n coefs \n");
for(int i=0;i<(k/2+1);i++)
{
    c_coeff[i]=(float)((k/2+1)-i)/(float)(k/2+1);
   
}
for(int i=0;i<(k/2+1);i++)
{
    printf(" %f",c_coeff[i]);
}
printf("\n coefs \n");


FILE *arq;
arq = fopen("entrada.txt", "rt");
for(int i=k2;i<(X+k2);i++)
    for(int j=k2;j<(Y+k2);j++)
        fscanf(arq," %d",&h_e[i+j*(X+k)]);
fclose(arq);

for(int i=k2;i<(X+k2);i++)
    for(int j=1;j<k2+1;j++)
    {
        h_e[i+(k2-j)*(X+k)] = h_e[i+(k2+j-1)*(X+k)];
        h_e[i+(Y+k2+j-1)*(X+k)] =  h_e[i+(Y+k2-j)*(X+k)] ;
    }

for(int i=1;i<k2+1;i++)
    for(int j=k2;j<(X+k2);j++)
    {
        h_e[(k2-i)+(j)*(X+k)] = h_e[(k2+i-1)+(j)*(X+k)];
        h_e[(X+k2+i-1)+(j)*(X+k)] =  h_e[(X+k2-i)+(j)*(X+k)];
    }

       
        

/* Copy vectors from host memory to device memory */
hipMemcpy(d_e, h_e, size, hipMemcpyHostToDevice);
hipMemcpy(d_c_coeff, c_coeff, (k/2+1)*sizeof(float), hipMemcpyHostToDevice);


hipEvent_t start, stop;
hipEventCreate (&start);
hipEventCreate (&stop);
hipEventRecord (start, 0); 

/******************
*** Kernel Call ***
*******************/
//_3Dstencil_global<<<blks,th_p_blk>>>(d_e,d_r,X,Y,Z);
for(int t=0;t<times;t++)
{
_2Dstencil_global<<<grid_dim,block_dim>>>(d_e,d_r,d_c_coeff,X+k,Y+k,k);
_copy_dr_to_de<<<grid_dim,block_dim>>>(d_e,d_r,X+k,Y+k,k/2);
}
hipError_t err = hipSuccess;
err = hipGetLastError();
if (err != hipSuccess)
{
    fprintf(stderr, "Failed to launch _3Dstencil_global kernel (error code %s)!\n", hipGetErrorString(err));
}
/******************
*** Kernel Call ***
*******************/



    hipDeviceSynchronize();
    hipEventRecord (stop, 0);
    hipEventSynchronize (stop);
    float elapsedTime;
    hipEventElapsedTime (&elapsedTime, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);


    printf("X %d || Y %d \nBX %d || BY %d \n",X,Y,BX,BY);
    printf ("[%d,%.5f],\n", tam,elapsedTime);
 
    hipMemcpy(h_r, d_r, size, hipMemcpyDeviceToHost);

    
arq = fopen("resultado.txt", "wt");
for(int i=k2;i<X+k2;i++)
{
    for(int j=k2;j<Y+k2;j++)
    {
      fprintf(arq," %d",h_r[i+j*(X+k)]);
    }
    fprintf(arq,"\n");
}
fclose(arq);

// arq = fopen("resultado.txt", "wt");
// for(int i=0;i<X+k;i++)
// {
//     for(int j=0;j<Y+k;j++)
//     {
//       fprintf(arq," %d",h_e[i+j*(X+k)]);
//     }
//     fprintf(arq,"\n");
// }
// fclose(arq);


    hipFree(d_e);
    hipFree(d_r);
    hipFree(d_c_coeff);
    std::free(h_e);
    std::free(h_r);
    std::free(c_coeff);

    return 0;
} /* main */

