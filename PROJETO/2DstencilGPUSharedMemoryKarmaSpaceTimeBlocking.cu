#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <unistd.h>
#include <stdlib.h>
#include <math.h>

/*
Instruções
COMPILAR -->  nvcc 2DstencilGPUSharedMemoryBlankBorderTimeSpaceSharingOpencvKarma.cu -o go `pkg-config --cflags --libs opencv` -w
EXECUTAR --> ./go DOMAIN_DIMS STENCIL_ORDER SPACE_TIME_BLOCK_TIMES BLOCK_DIM_X BLOCK_DIM_Y
*/

#include <iostream>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <stdio.h>
#include <opencv2/imgcodecs.hpp>
#include <math.h>
#include <string>

using namespace std;

//===> CONSTANTES karma model <===//
#define Eh 3.0f
#define En 1.0f
#define Re 0.6f
#define tauE 5.0f
#define tauN 250.0f
#define gam 0.001f
#define East 1.5415f
#define DT 0.05f
#define DX (12.0f / 32)

/*
Função somente da GPU que recebe os parametros para o calculo de um stencil
d_e - dado de entrada
d_r - dado de saida
d_v - campo que deve ser atualizado
c_coeff - variável utilizada para armazenar o valores dos coeficcientes do stencil (utilizada apenas na versão com stencil simples usado anteriormente)
X - Y - Dimensões das estruturas de entrada
k - ordem do stencil
x -y - posição do centro do stencil na estrutura de entrada
GX - Dimensão horizontal da estrutura do dado de saída
Gx - Gy posição do centro do stencil na estrutura de saida
*/
__device__ void destinyTest(float *d_r,int GX, int Gx, int Gy,float val)
{
    d_r[Gx + ((Gy) * (GX))] = val;
}
__device__ void _2Dstencil_(float *d_e, float *d_r, float *d_v, float *c_coeff, int X, int k, int x, int y, int GX, int Gx, int Gy)
{
    int h_e_i;
    int h_r_i = x + (y * (X));
    h_e_i = h_r_i;
    float temp = d_e[h_r_i];

    float rv = d_v[h_e_i];
    float Rn = (1.0f / (1.0f - expf(-Re))) - rv;
    float p = (temp > En) * 1.0f;
    float dv = (Rn * p - (1.0f - p) * rv) / tauN;
    float Dn = rv * rv;
    float hE = (1.0f - tanh(temp - Eh)) * temp * temp / 2.0f;
    float du = (((East - Dn) * hE) - temp) / tauE;

    float xlapr = d_e[(x + 1) + ((y) * (X))] - temp;
    float xlapl = temp - d_e[(x - 1) + ((y) * (X))];
    float xlapf = d_e[(x) + ((y + 1) * (X))] - temp;
    float xlapb = temp - d_e[(x) + ((y - 1) * (X))];

    float lap = xlapr - xlapl + xlapf - xlapb;

   
    temp = (temp + (du * DT) + (lap * DT * gam / (DX * DX)));

    //testes para saber se o problema estava na visualização pois o opencv foi configurado pra aceitar valores entre 0.0f e 1.0f
    //if(temp >= 0.0f && temp =< 1.0f)
    h_r_i = Gx + ((Gy) * (GX));
    d_r[h_r_i] = temp;//d_v[h_e_i];// d_e[h_e_i]+1;// = temp;
    //else
    //   d_r[h_r_i] = 1.0f;
    d_v[h_e_i] = rv + dv * DT;

    //*** código utilizado apenas na versão com stencil simples usado anteriormente
    //temp *= c_coeff[0];
    // for(int lk =1;lk<(k/2)+1;lk++)
    // {
    //     h_e_i = (x+lk) + ( (y) * (X) );
    //     temp += d_e[h_e_i]*c_coeff[lk];

    //     h_e_i = (x-lk) + ( (y) * (X) );
    //     temp += d_e[h_e_i]*c_coeff[lk];

    //     h_e_i = (x) + ( (y+lk) * (X) );
    //     temp += d_e[h_e_i]*c_coeff[lk];

    //     h_e_i = (x) + ( (y-lk) * (X) );
    //     temp += d_e[h_e_i]*c_coeff[lk];
    // }
    //  h_r_i = Gx + ( (Gy) * (GX) );
    // if(temp < 1.0f)
    //     d_r[h_r_i] = temp;
    // else
    //     d_r[h_r_i] = 1.0f;
}
/*
função chamada pelo host que controla as cópias e a ordem do calculo dos stencils bem como a carga para cada thread
*/
__global__ void _2Dstencil_global(float *d_e, float *d_r, float *d_v, float *c_coeff, int X, int Y, int k, int times)
{

    int x, y; //,h_e_i,h_r_i,Xs,Ys,Dx,Dy;
    x = threadIdx.x + (blockIdx.x * blockDim.x);
    y = threadIdx.y + (blockIdx.y * blockDim.y);
    int k2 = k / 2 * times;
    extern __shared__ float shared[];

    int blockThreadIndex = threadIdx.x + threadIdx.y * blockDim.x;
    // Xs = threadIdx.x;
    // Ys = threadIdx.y;
    int Dx = blockDim.x + (k * times);
    int Dy = blockDim.y + (k * times);
    int sharedTam = Dx * Dy;
    float *sharedRes = &shared[sharedTam];
    float *sharedV = &sharedRes[sharedTam];

    /*
    Copia o Tile de memória compartilhada necessária para a configuração de tempo desejada
    Stride é utilizado pois a quantidade de elementos a serem copiados é sempre maior que a quantidade de threads
    As bordas
    */
    for (int stride = blockThreadIndex; stride < sharedTam; stride += (blockDim.x * blockDim.y))
    {
         
         int globalIdxX = (blockIdx.x * blockDim.x) - k2 + stride % Dx;
         int globalIdxY = ((blockIdx.y * blockDim.y) - k2 + int(stride / Dx));
         int globalIdx = globalIdxX + (globalIdxX==-1) - (globalIdxX==X)      +      (globalIdxY + (globalIdxY==-1) - (globalIdxY==Y)) * X;
      
        shared[stride] = d_e[globalIdx];
        sharedV[stride] = d_v[globalIdx];
    }

    __syncthreads();

    /*
    Envia pra ser calculado todos os elementos além do ultimo instante de tempo
    */
    for (int t = times - 1; t > 0; t--)
    {
        //_2Dstencil_(shared,sharedRes,c_coeff,Dx,Dy,k,threadIdx.x+k2,threadIdx.y+k2,Dx,threadIdx.x+k2,threadIdx.y+k2);
        int tDx = blockDim.x + (t * k);
        int tDy = blockDim.y + (t * k);
        int tk2 = (times - t) * k / 2;
        // int tDx = blockDim.x+(1*k);
        // int tDy = blockDim.y+(1*k);
        // int tk2 = (1)*k/2;
        int tSharedTam = tDx * tDy;
        for (int stride = blockThreadIndex; stride < tSharedTam; stride += (blockDim.x * blockDim.y))
        {
            //int globalIdx = (stride % tDx) + tk2 + Dx*(int(stride / Dx)) + tk2;
            //destinyTest(shared, Dx, (stride % tDx) + tk2, int(stride / Dx) + tk2,t+1);
            _2Dstencil_(shared, sharedRes, sharedV, c_coeff, Dx, k, (stride % tDx) + tk2, (int(stride / Dx)) + tk2, Dx, (stride % tDx) + tk2, (int(stride / Dx)) + tk2);
        }
        __syncthreads();
        for (int stride = blockThreadIndex; stride < sharedTam; stride += (blockDim.x * blockDim.y))
        {
            shared[stride] = sharedRes[stride];
        }
        __syncthreads();
    }
    /*
    Envia pra ser calculado todos os elementos do ultimo instante de tempo
   */
    //_2Dstencil_(shared, d_r, sharedV, c_coeff, Dx, k, threadIdx.x + k2, threadIdx.y + k2, X, x, y);
    
    //  for(int stride=blockThreadIndex;stride<sharedTam;stride+=(blockDim.x*blockDim.y))
    // {
    //      int globalIdx = (blockIdx.x*blockDim.x)-k2+stride%Dx + ((blockIdx.y*blockDim.y)-k2+stride/Dx)*X;
    //     if(globalIdx > 0 && (blockIdx.x*blockDim.x)-k2+stride%Dx < X && ((blockIdx.y*blockDim.y)-k2+stride/Dx)<Y)
    //      d_r[globalIdx] = sharedRes[stride];
    //  }
    
    //destinyTest(d_r,X, x, y,1.0f);
    __syncthreads();
    for (int stride = blockThreadIndex; stride < sharedTam; stride += (blockDim.x * blockDim.y))
    {
         int globalIdxX = (blockIdx.x * blockDim.x) - k2 + stride % Dx;
         int globalIdxY = ((blockIdx.y * blockDim.y) - k2 + int(stride / Dx));
         int globalIdx = globalIdxX + (globalIdxX==-1) - (globalIdxX==X)      +      (globalIdxY + (globalIdxY==-1) - (globalIdxY==Y)) * X;
         if(blockIdx.x == 1 && blockIdx.y == 1)
            d_r[globalIdx] = shared[stride];
    }
   
}

int main(int argc, char *argv[])
{
    /*
    Declarações e valores padroes
    */
    float *h_e, *h_r, *h_v;
    float *d_e, *d_r, *d_v;
    int size,tam, sharedSize;
    int X = 32;
    int Y = 32;
    int k = 2;
    int times = 1;
    int BX = 32;
    int BY = 32;
    int GX = 1;
    int GY = 1;
    float *c_coeff, *d_c_coeff;

    /*
    Obtenção dos parâmetros de entrada
    */
    if (argc > 1)
    {
        X = atoi(argv[1]);
        Y = X;
    }
    if (argc > 2)
    {
        k = atoi(argv[2]);
    }

    if (argc > 3)
    {
        times = atoi(argv[3]);
    }

    if (X > 32)
    {
        GX = ceil((float)X / (float)32);
        BX = 32;
    }
    if (Y > 32)
    {
        GY = ceil((float)Y / (float)32);
        BY = 32;
    }

    /*
    Allocações de memória e configuração dos blocos e grid
    */
    dim3 block_dim(BX, BY, 1);
    dim3 grid_dim(GX, GY, 1);
    //sharedSize = ((block_dim.x+k)*(block_dim.y+k))*sizeof(int);
    sharedSize = ((block_dim.x + (k * times)) * (block_dim.y + (k * times))) * sizeof(float) * 3;
    //sharedTam = ((block_dim.x+(k*2))*(block_dim.y+(k*2)));
    size = X * Y * sizeof(float);
    tam = X * Y;

    h_e = (float *)malloc(size);
    h_r = (float *)malloc(size);
    h_v = (float *)malloc(size);
    c_coeff = (float *)malloc((k / 2 + 1) * sizeof(float));
    hipMalloc(&d_e, size);
    hipMalloc(&d_r, size);
    hipMalloc(&d_v, size);
    hipMalloc(&d_c_coeff, (k / 2 + 1) * sizeof(float));

    printf("\n coefs \n");
    for (int i = 0; i < (k / 2 + 1); i++)
    {
        c_coeff[i] = (float)((k / 2 + 1) - i) / (float)(k / 2 + 1);
    }
    //c_coeff[0] = 0.0;
    for (int i = 0; i < (k / 2 + 1); i++)
    {
        printf(" %f", c_coeff[i]);
    }
    printf("\n coefs \n");
    hipMemcpy(d_c_coeff, c_coeff, (k / 2 + 1) * sizeof(float), hipMemcpyHostToDevice);


//Copia os dados do campo e envia para a GPU e inicializa o dominio de entrada

        


    FILE *arq;
    arq = fopen("entrada.txt", "rt");
    for (int i = 0; i < X; i++)
        for (int j = 0; j < Y; j++)
        {
            h_v[i + j * X] = 0.5f;
            int temp;
            fscanf(arq," %d",&temp);
            h_e[i + j * X] = temp;
        }

    fclose(arq);
    hipMemcpy(d_v, h_v, size, hipMemcpyHostToDevice);
   
    /* 
    Copy vectors from host memory to device memory
    Copia os dados da entrada de volta a GPU
        */
    hipMemcpy(d_e, h_e, size, hipMemcpyHostToDevice);
    
    /*
    Começa o Timer
    */
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    /******************
    *** Kernel Call ***
    *******************/
    //_3Dstencil_global<<<blks,th_p_blk>>>(d_e,d_r,X,Y,Z);
    /*
    Executa o kernel
    */
    _2Dstencil_global<<<grid_dim, block_dim, sharedSize>>>(d_e, d_r, d_v, d_c_coeff, X, Y, k, times);

    /*
    Identifica possíveis erros
    */
    hipError_t err = hipSuccess;
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch _3Dstencil_global kernel (error code %s)!\n", hipGetErrorString(err));
    }
    /******************
    *** Kernel Call ***
    *******************/

    hipDeviceSynchronize();
    /*
    Para o Timer
    */
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    //printf("X %d || Y %d \nBX %d || BY %d \n",X,Y,BX,BY);
    //printf ("[%d,%.5f],\n", tam,elapsedTime);
    /*
    Copia o resultado de volta para o CPU
    */
    hipMemcpy(h_r, d_r, size, hipMemcpyDeviceToHost);
    /*
    Copia o resultado para a imagem de visualização
    A estrutura de 
    */
    arq = fopen("resultado.txt", "wt");
    for (int i = 0; i < X; i++)
    {
        for (int j = 0; j < Y; j++)
        {
            fprintf(arq," %6.4f",h_r[i+j*X]);
        }
        fprintf(arq,"\n");
    }
    fclose(arq);
        

    hipFree(d_e);
    hipFree(d_r);
    hipFree(d_c_coeff);
    std::free(h_e);
    std::free(h_r);
    std::free(c_coeff);

    return 0;
} /* main */
