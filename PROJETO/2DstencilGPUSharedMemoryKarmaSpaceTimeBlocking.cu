
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <unistd.h>
#include <stdlib.h>
#include <math.h>


/*
Instruções
COMPILAR -->  nvcc 2DstencilGPUSharedMemoryBlankBorderTimeSpaceSharingOpencvKarma.cu -o go `pkg-config --cflags --libs opencv` -w
EXECUTAR --> ./go DOMAIN_DIMS STENCIL_ORDER SPACE_TIME_BLOCK_TIMES BLOCK_DIM_X BLOCK_DIM_Y
*/

#include <iostream>
#include <fstream>
#include <stdio.h>
#include <math.h>
#include <string>

using namespace std;

//===> CONSTANTES karma model <===//
#ifndef MODEL_WIDTH
#define MODEL_WIDTH 96
#endif

#ifndef BLOCK_TIMES
#define BLOCK_TIMES 1
#endif

#define Eh 3.0f
#define En 1.0f
#define Re 0.6f
#define tauE 5.0f
#define tauN 250.0f
#define gam 0.001f
#define East 1.5415f
#define DT 0.05f
#define DX (12.0f / MODEL_WIDTH)

#define MODELSIZE_X (MODEL_WIDTH)
#define MODELSIZE_Y (MODEL_WIDTH)
#define MODELSIZE_Z 1
#define MODELSIZE2D ( MODELSIZE_X*MODELSIZE_Y )

#ifndef BLOCKDIM_X
#define BLOCKDIM_X 32
#endif

#ifndef BLOCKDIM_Y
#define BLOCKDIM_Y 32
#endif

#define BLOCKDIM_Z 1
#define BLOCKDIM2D ( BLOCKDIM_X*BLOCKDIM_Y )

//==> CUDA GRID <==//
#define GRIDDIM_X ( ( MODELSIZE_X / BLOCKDIM_X ) + ( ( MODELSIZE_X % BLOCKDIM_X ) > 0 ) )
#define GRIDDIM_Y ( ( MODELSIZE_Y / BLOCKDIM_Y ) + ( ( MODELSIZE_Y % BLOCKDIM_Y ) > 0 ) )
#define GRIDDIM_Z 1

#define SHARED_TAM ((BLOCKDIM_X + (2 * BLOCK_TIMES)) * (BLOCKDIM_Y + (2 * BLOCK_TIMES)))
#define SHARED_DX (BLOCKDIM_X + (2 * BLOCK_TIMES))
#define SHARED_DY (BLOCKDIM_Y + (2 * BLOCK_TIMES))
/*
Função somente da GPU que recebe os parametros para o calculo de um stencil
d_e - dado de entrada
d_r - dado de saida
d_v - campo que deve ser atualizado
c_coeff - variável utilizada para armazenar o valores dos coeficcientes do stencil (utilizada apenas na versão com stencil simples usado anteriormente)
X - Y - Dimensões das estruturas de entrada
k - ordem do stencil
x -y - posição do centro do stencil na estrutura de entrada
GX - Dimensão horizontal da estrutura do dado de saída
Gx - Gy posição do centro do stencil na estrutura de saida
*/
__forceinline__ __device__ void _2Dstencil_(float *d_e, float *d_r, float *d_v, int X, int x, int y, int GX, int Gx, int Gy)
{
    int h_e_i = x + (y * (X));
    float temp = d_e[h_e_i];
    
    float rv = d_v[h_e_i];


    float Rn = (1.0f / (1.0f - expf(-Re))) - rv;
    float p = (temp > En) * 1.0f;
    float dv = (Rn * p - (1.0f - p) * rv) / tauN;
    float Dn = rv * rv;
    float hE = (1.0f - tanh(temp - Eh)) * temp * temp / 2.0f;
    float du = (((East - Dn) * hE) - temp) / tauE;

    float xlapr = d_e[(x + 1) + ((y) * (X))] - temp;
    float xlapl = temp - d_e[(x - 1) + ((y) * (X))];
    float xlapf = d_e[(x) + ((y + 1) * (X))] - temp;
    float xlapb = temp - d_e[(x) + ((y - 1) * (X))];

    float lap = xlapr - xlapl + xlapf - xlapb;
   
    temp = (temp + (du * DT) + (lap * DT * gam / (DX * DX)));

    d_v[h_e_i] = rv + dv * DT;
    h_e_i = Gx + ((Gy) * (GX));
    d_r[h_e_i] = temp;
}
/*
função chamada pelo host que controla as cópias e a ordem do calculo dos stencils bem como a carga para cada thread
, MODELSIZE_X, MODELSIZE_Y, BLOCK_TIMES
 int X, int Y, int times
*/
__global__ void _2Dstencil_global(float *d_e, float *d_r, float *d_v)
{

    int x, y; //,h_e_i,h_r_i,Xs,Ys,Dx,Dy;
    x = threadIdx.x + (blockIdx.x * BLOCKDIM_X);
    y = threadIdx.y + (blockIdx.y * BLOCKDIM_Y);
    extern __shared__ float sharedOrig[];

    int blockThreadIndex = threadIdx.x + threadIdx.y * BLOCKDIM_X;

    float * shared = sharedOrig;
    float * sharedRes = shared + SHARED_TAM;
    float * sharedV = sharedRes + SHARED_TAM; 
    /*
    Copia o Tile de memória compartilhada necessária para a configuração de tempo desejada
    Stride é utilizado pois a quantidade de elementos a serem copiados é sempre maior que a quantidade de threads
    As bordas
    */
    for (int stride = blockThreadIndex; stride < SHARED_TAM; stride += (BLOCKDIM_X * BLOCKDIM_Y))
    {
        int sharedIdxX = stride % SHARED_DX;
        int sharedIdxY = int(stride / SHARED_DX);
        int globalIdxX = (blockIdx.x * BLOCKDIM_X) + sharedIdxX - BLOCK_TIMES;
        int globalIdxY = (blockIdx.y * BLOCKDIM_Y) + sharedIdxY - BLOCK_TIMES;
        int globalIdx = globalIdxX + (-1*globalIdxX)*(globalIdxX < 0) - (globalIdxX-MODELSIZE_X+1)*(globalIdxX >= MODELSIZE_X)  +  (globalIdxY + (-1*globalIdxY)*(globalIdxY < 0) - (globalIdxY-MODELSIZE_Y+1)*(globalIdxY >= MODELSIZE_Y)) * MODELSIZE_X;
       
        shared[stride] = d_e[globalIdx];
        sharedV[stride] = d_v[globalIdx];
    }

    __syncthreads();

    /*
    Envia pra ser calculado todos os elementos além do ultimo instante de tempo
    */
    for (int t = 1; t < BLOCK_TIMES; t++)
    {
        int tDx = BLOCKDIM_X + ((BLOCK_TIMES - t) * 2);
        int tDy = BLOCKDIM_Y + ((BLOCK_TIMES - t) * 2);
        int tk2 = (t);
        int tSharedTam = tDx * tDy;
        for (int stride = blockThreadIndex; stride < tSharedTam; stride += (BLOCKDIM_X * BLOCKDIM_Y))
        {
            int tempX = (stride % tDx) + tk2;
            int tempY = (int(stride / tDx)) + tk2;
            _2Dstencil_(shared, sharedRes, sharedV, SHARED_DX, tempX, tempY, SHARED_DX, tempX, tempY);
        }

        float * temp = shared;
        shared = sharedRes;
        sharedRes = temp;
        __syncthreads();
    }
    /*
    Envia pra ser calculado todos os elementos do ultimo instante de tempo
   */
    _2Dstencil_(shared, d_r, sharedV, SHARED_DX, ((x%(BLOCKDIM_X))+BLOCK_TIMES), ((y%(BLOCKDIM_Y))+BLOCK_TIMES), MODELSIZE_X, x, y);
    
     int globalIdx = x + y * MODELSIZE_X;
     int sharedIdx = ((x%(BLOCKDIM_X))+BLOCK_TIMES) + ((y%(BLOCKDIM_Y))+BLOCK_TIMES)*SHARED_DX;
     d_v[globalIdx] = sharedV[sharedIdx];
}

int main(int argc, char *argv[])
{
    /*
    Declarações e valores padroes
    */
    //float *h_e, *h_r, *h_v;
    bool resultado = false;
    float *h_e, *h_v;
    float *d_e, *d_r, *d_v;
    int sharedSize;
    int globalTimes = 1;

    /*
    Obtenção dos parâmetros de entrada
    */
    if (argc > 1)
    {
        globalTimes = atoi(argv[1]);
    }

    if(argc > 2)
    {
        resultado = atoi(argv[2])==1;
    }


    /*
    Allocações de memória e configuração dos blocos e grid
    */
    dim3 grid_dim(GRIDDIM_X,GRIDDIM_Y,GRIDDIM_Z);
    dim3 block_dim(BLOCKDIM_X,BLOCKDIM_Y,BLOCKDIM_Z);
    sharedSize = SHARED_TAM * sizeof(float) * 3;
    h_e = (float *)malloc(MODELSIZE2D*sizeof(float));
    h_v = (float *)malloc(MODELSIZE2D*sizeof(float));
    hipMalloc(&d_e, MODELSIZE2D*sizeof(float));
    hipMalloc(&d_r, MODELSIZE2D*sizeof(float));
    hipMalloc(&d_v, MODELSIZE2D*sizeof(float));

//Copia os dados do campo e envia para a GPU e inicializa o dominio de entrada

        


    FILE *arq;
    arq = fopen("entrada.txt", "rt");
    for (int i = 0; i < MODELSIZE_X; i++)
        for (int j = 0; j < MODELSIZE_Y; j++)
        {
            h_v[i + j * MODELSIZE_X] =0.5f;
            int temp;
            fscanf(arq," %d",&temp);
            h_e[i + j * MODELSIZE_X] = temp;
        }

    fclose(arq);
    hipMemcpy(d_v, h_v, MODELSIZE2D*sizeof(float), hipMemcpyHostToDevice);
   
    /* 
    Copy vectors from host memory to device memory
    Copia os dados da entrada de volta a GPU
        */
    hipMemcpy(d_e, h_e, MODELSIZE2D*sizeof(float), hipMemcpyHostToDevice);
    
    /*
    Começa o Timer
    */
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    /******************
    *** Kernel Call ***
    *******************/
    //_3Dstencil_global<<<blks,th_p_blk>>>(d_e,d_r,X,Y,Z);
    /*
    Executa o kernel
    */
    for(int i=0; i<globalTimes/BLOCK_TIMES; i ++)
    {
        _2Dstencil_global<<<grid_dim, block_dim, sharedSize>>>(d_e, d_r, d_v);
        float * temp = d_e;
        d_e = d_r;
        d_r = temp;
    }
    

    /*
    Identifica possíveis erros
    */
    hipError_t err = hipSuccess;
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf ("-1");
        hipFree(d_e);
        hipFree(d_r);
        hipFree(d_v);
        std::free(h_e);
        std::free(h_v);
        fprintf(stderr, "Failed to launch _2Dstencil_global kernel (error code %s)!\n", hipGetErrorString(err));
        return 0;
    }
    /******************
    *** Kernel Call ***
    *******************/

    hipDeviceSynchronize();
    /*
    Para o Timer
    */
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    //printf("X %d || Y %d \nBX %d || BY %d \n",X,Y,BX,BY);
    //printf ("[%d,%.5f], sharedSize %d  CPUMem = %d bytes  GPUMen = %d bytes\n", times,elapsedTime,sharedSize,size*2,size*3);
    //printf("GPU elapsed time: %f s (%f milliseconds)\n", (elapsedTime/1000.0), elapsedTime);
    printf ("%.5f",elapsedTime);
    /*
    Copia o resultado de volta para o CPU
    */
    hipMemcpy(h_e, d_e, MODELSIZE2D*sizeof(float), hipMemcpyDeviceToHost);
    /*
    Copia o resultado para a imagem de visualização
    A estrutura de 
    */
    if(resultado)
    {
        arq = fopen("resultado.txt", "wt");
        for (int i = 0; i < MODELSIZE_X; i++)
        {
            for (int j = 0; j < MODELSIZE_Y; j++)
            {
                fprintf(arq," %6.4f",h_e[i+j*MODELSIZE_X]);
            }
            fprintf(arq,"\n");
        }
        fclose(arq);
    }
    
        

    hipFree(d_e);
    hipFree(d_r);
    hipFree(d_v);
    std::free(h_e);
    std::free(h_v);

    return 0;
} /* main */
