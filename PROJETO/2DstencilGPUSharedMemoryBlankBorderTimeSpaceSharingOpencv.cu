#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <unistd.h>
#include <stdlib.h>
#include <math.h>

/*
COMPILE -->  nvcc 2DstencilGPUSharedMemoryBlankBorderTimeSpaceSharingOpencv.cu -o go `pkg-config --cflags --libs opencv` -w
EXECUTE --> ./main.exe
*/


//**********
//**OPENCV**
//**********
#include <iostream>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <stdio.h>
#include <opencv2/imgcodecs.hpp>
#include <math.h>
#include <string>

#define JAN_OFFSET 0
using namespace cv;
using namespace std;

void CallBackFunc(int event, int x, int y, int flags, void* userdata)
{
    Mat *img = (Mat*)userdata;
     if  ( event == EVENT_LBUTTONDOWN )
     {
          cout << "Left button of the mouse is clicked - position (" << x << ", " << y << ")" << endl;
          img->at<uchar>(Point(x,y)) = 16;
     }
     else if  ( event == EVENT_RBUTTONDOWN )
     {
         // cout << "Right button of the mouse is clicked - position (" << x << ", " << y << ")" << endl;
     }
     else if  ( event == EVENT_MBUTTONDOWN )
     {
          //cout << "Middle button of the mouse is clicked - position (" << x << ", " << y << ")" << endl;
     }
     else if ( event == EVENT_MOUSEMOVE )
     {
         // cout << "Mouse move over the window - position (" << x << ", " << y << ")" << endl;

     }
}
class Window
{
	char *m_name;

    public:
	Window(char *name, int tam_ja, int x, int y,Mat *img = NULL)
	{
		m_name = name;
		namedWindow(m_name, WINDOW_NORMAL & CV_GUI_NORMAL);
		moveWindow(m_name, tam_ja * x + JAN_OFFSET, tam_ja * y + JAN_OFFSET);
        resizeWindow(m_name, tam_ja, tam_ja);
        setMouseCallback(m_name, CallBackFunc, img);
	}
	void imshow(Mat img)
	{
		cv::imshow(m_name, img);
	}
	void createTrackbar(char *trackName, int *var, int max_val)
	{
		cv::createTrackbar(trackName, m_name, var, max_val);
	}
};

//**********
//**OPENCV**
//**********



__device__ void _2Dstencil_(int *d_e,int *d_r,float* c_coeff,int X,int Y,int k, int x, int y,int GX,int Gx,int Gy)
{     
    int h_e_i;
    int h_r_i = x + ( y * (X) );
    h_e_i = h_r_i;
    int temp = d_e[h_r_i];
    temp *= c_coeff[0];
    for(int lk =1;lk<(k/2)+1;lk++)
    {
        h_e_i = (x+lk) + ( (y) * (X) );
        temp += d_e[h_e_i]*c_coeff[lk];

        h_e_i = (x-lk) + ( (y) * (X) );
        temp += d_e[h_e_i]*c_coeff[lk];

        h_e_i = (x) + ( (y+lk) * (X) );
        temp += d_e[h_e_i]*c_coeff[lk];

        h_e_i = (x) + ( (y-lk) * (X) );
        temp += d_e[h_e_i]*c_coeff[lk];
    }
     h_r_i = Gx + ( (Gy) * (GX) );
    if(temp < 255)
        d_r[h_r_i] = temp;
    else
        d_r[h_r_i] = 255;    
}
__global__ void _2Dstencil_global(int *d_e,int *d_r,float *c_coeff,int X,int Y,int k,int times){

    int x,y;//,h_e_i,h_r_i,Xs,Ys,Dx,Dy;
    x = threadIdx.x + (blockIdx.x*blockDim.x);
    y = threadIdx.y + (blockIdx.y*blockDim.y);
    int k2 = k/2*times;
    extern __shared__ int shared[];
    
    int blockThreadIndex = threadIdx.x + threadIdx.y*blockDim.x;
    // Xs = threadIdx.x;
    // Ys = threadIdx.y;
    int Dx = blockDim.x+(k*times);
    int Dy = blockDim.y+(k*times);
    int sharedTam = Dx*Dy;
    int * sharedRes = &shared[sharedTam];
    
    for(int stride=blockThreadIndex;stride<sharedTam;stride+=(blockDim.x*blockDim.y))
    {
        int globalIdx = (blockIdx.x*blockDim.x)-k2+stride%Dx + ((blockIdx.y*blockDim.y)-k2+stride/Dx)*X;
        if(globalIdx > 0 && (blockIdx.x*blockDim.x)-k2+stride%Dx < X && ((blockIdx.y*blockDim.y)-k2+stride/Dx)<Y)
            shared[stride] = d_e[globalIdx];
        else
            shared[stride] = 0;
       
    }
    __syncthreads();
    for(int t=times-1;t>0;t--)
    {  
        //_2Dstencil_(shared,sharedRes,c_coeff,Dx,Dy,k,threadIdx.x+k2,threadIdx.y+k2,Dx,threadIdx.x+k2,threadIdx.y+k2);
        int tDx = blockDim.x+(t*k);
        int tDy = blockDim.y+(t*k);
        int tk2 = (times-t)*k/2;
        // int tDx = blockDim.x+(1*k);
        // int tDy = blockDim.y+(1*k);
        // int tk2 = (1)*k/2;
        int tSharedTam = tDx * tDy;
        for(int stride=blockThreadIndex;stride<tSharedTam;stride+=(blockDim.x*blockDim.y))
        {
            _2Dstencil_(shared,sharedRes,c_coeff,Dx,Dy,k,(stride%tDx)+tk2,(stride/tDx)+tk2,Dx,(stride%tDx)+tk2,(stride/tDx)+tk2);
        }
        __syncthreads();
        for(int stride=blockThreadIndex;stride<sharedTam;stride+=(blockDim.x*blockDim.y))
        {
            shared[stride]=sharedRes[stride];
        }
        __syncthreads();
   }
   
    _2Dstencil_(shared,d_r,c_coeff,Dx,Dy,k,threadIdx.x+k2,threadIdx.y+k2,X,x,y);
   
    // for(int stride=blockThreadIndex;stride<sharedTam;stride+=(blockDim.x*blockDim.y))
    // {
    //     int globalIdx = (blockIdx.x*blockDim.x)-k2+stride%Dx + ((blockIdx.y*blockDim.y)-k2+stride/Dx)*X;
    //     if(globalIdx > 0 && (blockIdx.x*blockDim.x)-k2+stride%Dx < X && ((blockIdx.y*blockDim.y)-k2+stride/Dx)<Y)
    //     d_r[globalIdx] = sharedRes[stride];
       
    // }
        
}


int main(int argc, char* argv[]) {

int *h_e,*h_r;
int *d_e, *d_r;
int size,tam,sharedSize,sharedTam;
int X=32;
int Y=32;
int k=4;
int times = 1;
int BX=32;
int BY=32;
int GX=1;
int GY=1;
float *c_coeff,*d_c_coeff;
if(argc > 1)
{
    X = atoi(argv[1]);
    Y = X;
}
if(argc > 2)
{
    k = atoi(argv[2]);
}

if(argc > 3)
{
    times = atoi(argv[3]);
}


if(X>32)
{
    GX = ceil((float)X/(float)32);
    BX = 32;
}
if(Y>32)
{
    GY = ceil((float)Y/(float)32);
    BY = 32;
}
    

dim3 block_dim(BX,BY,1);
dim3 grid_dim(GX,GY,1);
//sharedSize = ((block_dim.x+k)*(block_dim.y+k))*sizeof(int);
sharedSize = ((block_dim.x+(k*times))*(block_dim.y+(k*times)))*sizeof(int)*2;
//sharedTam = ((block_dim.x+(k*2))*(block_dim.y+(k*2)));
size = X * Y * sizeof(int);
tam = X * Y;


h_e = (int*) malloc(size);
h_r = (int*) malloc(size);
c_coeff = (float*)malloc((k/2+1)*sizeof(float));
hipMalloc(&d_e, size);
hipMalloc(&d_r, size);
hipMalloc(&d_c_coeff,(k/2+1)*sizeof(float));

printf("\n coefs \n");
for(int i=0;i<(k/2+1);i++)
{
    c_coeff[i]=(float)((k/2+1)-i)/(float)(k/2+1);
   
}
//c_coeff[0] = 0.0;
for(int i=0;i<(k/2+1);i++)
{
    printf(" %f",c_coeff[i]);
}
printf("\n coefs \n");

//**********
//**OPENCV**
//**********
if (argc < 2)
	{
		printf("\nespecifique a imagem\n");
		return -1;
    }
    
    //Mat orig = Mat::zeros(1024,1024,)//imread("doidera2.PNG"); //imread(argv[1]);
    Mat orig = Mat::zeros(X,Y, CV_8U);
    Mat result = Mat::zeros(X,Y, CV_8U);
    //Window original("orig", 600, 0, 0,&orig);
    Window resultado("result", 600, 2, 0,&result);
    
//**********
//**OPENCV**
//**********

// FILE *arq;
// arq = fopen("entrada.txt", "rt");
// for(int i=0;i<X;i++)
//     for(int j=0;j<Y;j++)
//     {
//         //fscanf(arq," %d",&h_e[i+j*X]);
//         h_e[i+j*X] = 0;
//         if(i > 100 && i < 150 && j > 100 && j < 150 )
//             h_e[i+j*X] = 2;
//         orig.at<uchar>(Point(i,j)) = h_e[i+j*X];

//     }
//     //Mat orig = Mat(1024,1024, CV_8U, h_e); 
    
// fclose(arq);



while (true)
{
    for(int i=0;i<X;i++)
    for(int j=0;j<Y;j++)
    {
        h_e[i+j*X] = (int)result.at<uchar>(Point(i,j));
    }
      /* Copy vectors from host memory to device memory */
  hipMemcpy(d_e, h_e, size, hipMemcpyHostToDevice);
  hipMemcpy(d_c_coeff, c_coeff, (k/2+1)*sizeof(float), hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate (&start);
    hipEventCreate (&stop);
    hipEventRecord (start, 0); 

    /******************
    *** Kernel Call ***
    *******************/
    //_3Dstencil_global<<<blks,th_p_blk>>>(d_e,d_r,X,Y,Z);
    _2Dstencil_global<<<grid_dim,block_dim,sharedSize>>>(d_e,d_r,d_c_coeff,X,Y,k,times);

    hipError_t err = hipSuccess;
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch _3Dstencil_global kernel (error code %s)!\n", hipGetErrorString(err));
    }
    /******************
    *** Kernel Call ***
    *******************/



    hipDeviceSynchronize();
    hipEventRecord (stop, 0);
    hipEventSynchronize (stop);
    float elapsedTime;
    hipEventElapsedTime (&elapsedTime, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);


    //printf("X %d || Y %d \nBX %d || BY %d \n",X,Y,BX,BY);
    printf ("[%d,%.5f],\n", tam,elapsedTime);

    hipMemcpy(h_r, d_r, size, hipMemcpyDeviceToHost);

    for(int i=0;i<X;i++)
    for(int j=0;j<Y;j++)
        result.at<uchar>(Point(i,j)) = (uchar)h_r[i+j*X];

    //original.imshow(orig);
    resultado.imshow(result);
    

    int *temp = h_e;
    h_e = h_r;
    h_r = temp;
    // Wait for key is pressed then break loop
    if (waitKey(0) == 27) //ESC == 27
    {
        break;
    }
	}


    hipFree(d_e);
    hipFree(d_r);
    hipFree(d_c_coeff);
    std::free(h_e);
    std::free(h_r);
    std::free(c_coeff);


    

    return 0;
} /* main */
