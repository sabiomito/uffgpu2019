#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <unistd.h>
#include <stdlib.h>
#include <math.h>

/*
Instruções
COMPILAR -->  nvcc 2DstencilGPUSharedMemoryBlankBorderTimeSpaceSharingOpencvKarma.cu -o go `pkg-config --cflags --libs opencv` -w
EXECUTAR --> ./go DOMAIN_DIMS STENCIL_ORDER SPACE_TIME_BLOCK_TIMES BLOCK_DIM_X BLOCK_DIM_Y
*/

#include <iostream>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <stdio.h>
#include <opencv2/imgcodecs.hpp>
#include <math.h>
#include <string>

using namespace std;

//===> CONSTANTES karma model <===//
#define Eh 3.0f
#define En 1.0f
#define Re 0.6f
#define tauE 5.0f
#define tauN 250.0f
#define gam 0.001f
#define East 1.5415f
#define DT 0.05f
#define DX (12.0f / 32)

//**********
//**OPENCV**
//**********
#define JAN_OFFSET 0
using namespace cv;
/*
Função que detecta um clique na imagem do opencv e atribui na estrutura Mat da imagem mostrada o pixel clicado com valor de 1.0f
*/
void CallBackFunc(int event, int x, int y, int flags, void *userdata)
{
    Mat *img = (Mat *)userdata;
    if (event == EVENT_LBUTTONDOWN)
    {
        cout << "Left button of the mouse is clicked - position (" << x << ", " << y << ")" << endl;
        img->at<float>(Point(x, y)) = 1.0f;
    }
    else if (event == EVENT_RBUTTONDOWN)
    {
        // cout << "Right button of the mouse is clicked - position (" << x << ", " << y << ")" << endl;
    }
    else if (event == EVENT_MBUTTONDOWN)
    {
        //cout << "Middle button of the mouse is clicked - position (" << x << ", " << y << ")" << endl;
    }
    else if (event == EVENT_MOUSEMOVE)
    {
        // cout << "Mouse move over the window - position (" << x << ", " << y << ")" << endl;
    }
}
/*
Função que ajuda a gerenciar a posição e nome das janelas do opencv de forma prática
*/
class Window
{
    char *m_name;

public:
    Window(char *name, int tam_ja, int x, int y, Mat *img = NULL)
    {
        m_name = name;
        namedWindow(m_name, WINDOW_NORMAL & CV_GUI_NORMAL);
        moveWindow(m_name, tam_ja * x + JAN_OFFSET, tam_ja * y + JAN_OFFSET);
        resizeWindow(m_name, tam_ja, tam_ja);
        setMouseCallback(m_name, CallBackFunc, img);
    }
    void imshow(Mat img)
    {
        cv::imshow(m_name, img);
    }
    void createTrackbar(char *trackName, int *var, int max_val)
    {
        cv::createTrackbar(trackName, m_name, var, max_val);
    }
};

//**********
//**OPENCV**
//**********

/*
Função somente da GPU que recebe os parametros para o calculo de um stencil
d_e - dado de entrada
d_r - dado de saida
d_v - campo que deve ser atualizado
c_coeff - variável utilizada para armazenar o valores dos coeficcientes do stencil (utilizada apenas na versão com stencil simples usado anteriormente)
X - Y - Dimensões das estruturas de entrada
k - ordem do stencil
x -y - posição do centro do stencil na estrutura de entrada
GX - Dimensão horizontal da estrutura do dado de saída
Gx - Gy posição do centro do stencil na estrutura de saida
*/

__device__ void _2Dstencil_(float *d_e, float *d_r, float *d_v, float *c_coeff, int X, int Y, int k, int x, int y, int GX, int Gx, int Gy)
{
    int h_e_i;
    int h_r_i = x + (y * (X));
    h_e_i = h_r_i;
    float temp = d_e[h_r_i];

    float rv = d_v[h_r_i];
    float Rn = (1.0f / (1.0f - expf(-Re))) - rv;
    float p = (temp > En) * 1.0f;
    float dv = (Rn * p - (1.0f - p) * rv) / tauN;
    float Dn = rv * rv;
    float hE = (1.0f - tanh(temp - Eh)) * temp * temp / 2.0f;
    float du = (((East - Dn) * hE) - temp) / tauE;

    float xlapr = d_e[(x + 1) + ((y) * (X))] - temp;
    float xlapl = temp - d_e[(x - 1) + ((y) * (X))];
    float xlapf = d_e[(x) + ((y + 1) * (X))] - temp;
    float xlapb = temp - d_e[(x) + ((y - 1) * (X))];

    float lap = xlapr - xlapl + xlapf - xlapb;

    h_r_i = Gx + ((Gy) * (GX));
    temp = (temp + (du * DT) + (lap * DT * gam / (DX * DX)));

    //testes para saber se o problema estava na visualização pois o opencv foi configurado pra aceitar valores entre 0.0f e 1.0f
    //if(temp >= 0.0f && temp =< 1.0f)
    d_r[h_r_i] = temp;
    //else
    //   d_r[h_r_i] = 1.0f;
    d_v[h_e_i] = rv + dv * DT;

    //*** código utilizado apenas na versão com stencil simples usado anteriormente
    //temp *= c_coeff[0];
    // for(int lk =1;lk<(k/2)+1;lk++)
    // {
    //     h_e_i = (x+lk) + ( (y) * (X) );
    //     temp += d_e[h_e_i]*c_coeff[lk];

    //     h_e_i = (x-lk) + ( (y) * (X) );
    //     temp += d_e[h_e_i]*c_coeff[lk];

    //     h_e_i = (x) + ( (y+lk) * (X) );
    //     temp += d_e[h_e_i]*c_coeff[lk];

    //     h_e_i = (x) + ( (y-lk) * (X) );
    //     temp += d_e[h_e_i]*c_coeff[lk];
    // }
    //  h_r_i = Gx + ( (Gy) * (GX) );
    // if(temp < 1.0f)
    //     d_r[h_r_i] = temp;
    // else
    //     d_r[h_r_i] = 1.0f;
}
/*
função chamada pelo host que controla as cópias e a ordem do calculo dos stencils bem como a carga para cada thread
*/
__global__ void _2Dstencil_global(float *d_e, float *d_r, float *d_v, float *c_coeff, int X, int Y, int k, int times)
{

    int x, y; //,h_e_i,h_r_i,Xs,Ys,Dx,Dy;
    x = threadIdx.x + (blockIdx.x * blockDim.x);
    y = threadIdx.y + (blockIdx.y * blockDim.y);
    int k2 = k / 2 * times;
    extern __shared__ float shared[];

    int blockThreadIndex = threadIdx.x + threadIdx.y * blockDim.x;
    // Xs = threadIdx.x;
    // Ys = threadIdx.y;
    int Dx = blockDim.x + (k * times);
    int Dy = blockDim.y + (k * times);
    int sharedTam = Dx * Dy;
    float *sharedRes = &shared[sharedTam];

    /*
    Copia o Tile de memória compartilhada necessária para a configuração de tempo desejada
    Stride é utilizado pois a quantidade de elementos a serem copiados é sempre maior que a quantidade de threads
    As bordas
    */
    for (int stride = blockThreadIndex; stride < sharedTam; stride += (blockDim.x * blockDim.y))
    {
        int globalIdx = (blockIdx.x * blockDim.x) - k2 + stride % Dx + ((blockIdx.y * blockDim.y) - k2 + stride / Dx) * X;
        if (globalIdx > 0 && (blockIdx.x * blockDim.x) - k2 + stride % Dx < X && ((blockIdx.y * blockDim.y) - k2 + stride / Dx) < Y)
            shared[stride] = d_e[globalIdx];
        else
            shared[stride] = 0.0f;
    }

    __syncthreads();

    /*
    Envia pra ser calculado todos os elementos além do ultimo instante de tempo
    */
    for (int t = times - 1; t > 0; t--)
    {
        //_2Dstencil_(shared,sharedRes,c_coeff,Dx,Dy,k,threadIdx.x+k2,threadIdx.y+k2,Dx,threadIdx.x+k2,threadIdx.y+k2);
        int tDx = blockDim.x + (t * k);
        int tDy = blockDim.y + (t * k);
        int tk2 = (times - t) * k / 2;
        // int tDx = blockDim.x+(1*k);
        // int tDy = blockDim.y+(1*k);
        // int tk2 = (1)*k/2;
        int tSharedTam = tDx * tDy;
        for (int stride = blockThreadIndex; stride < tSharedTam; stride += (blockDim.x * blockDim.y))
        {
            _2Dstencil_(shared, sharedRes, d_v, c_coeff, Dx, Dy, k, (stride % tDx) + tk2, (stride / tDx) + tk2, Dx, (stride % tDx) + tk2, (stride / tDx) + tk2);
        }
        __syncthreads();
        for (int stride = blockThreadIndex; stride < sharedTam; stride += (blockDim.x * blockDim.y))
        {
            shared[stride] = sharedRes[stride];
        }
        __syncthreads();
    }
    /*
    Envia pra ser calculado todos os elementos do ultimo instante de tempo
   */
    _2Dstencil_(shared, d_r, d_v, c_coeff, Dx, Dy, k, threadIdx.x + k2, threadIdx.y + k2, X, x, y);

    // for(int stride=blockThreadIndex;stride<sharedTam;stride+=(blockDim.x*blockDim.y))
    // {
    //     int globalIdx = (blockIdx.x*blockDim.x)-k2+stride%Dx + ((blockIdx.y*blockDim.y)-k2+stride/Dx)*X;
    //     if(globalIdx > 0 && (blockIdx.x*blockDim.x)-k2+stride%Dx < X && ((blockIdx.y*blockDim.y)-k2+stride/Dx)<Y)
    //     d_r[globalIdx] = sharedRes[stride];

    // }
}

int main(int argc, char *argv[])
{
    /*
    Declarações e valores padroes
    */
    float *h_e, *h_r, *h_v;
    float *d_e, *d_r, *d_v;
    int size, tam, sharedSize, sharedTam;
    int X = 32;
    int Y = 32;
    int k = 2;
    int times = 1;
    int BX = 32;
    int BY = 32;
    int GX = 1;
    int GY = 1;
    float *c_coeff, *d_c_coeff;

    /*
    Obtenção dos parâmetros de entrada
    */
    if (argc > 1)
    {
        X = atoi(argv[1]);
        Y = X;
    }
    if (argc > 2)
    {
        k = atoi(argv[2]);
    }

    if (argc > 3)
    {
        times = atoi(argv[3]);
    }

    if (X > 32)
    {
        GX = ceil((float)X / (float)32);
        BX = 32;
    }
    if (Y > 32)
    {
        GY = ceil((float)Y / (float)32);
        BY = 32;
    }

    /*
    Allocações de memória e configuração dos blocos e grid
    */
    dim3 block_dim(BX, BY, 1);
    dim3 grid_dim(GX, GY, 1);
    //sharedSize = ((block_dim.x+k)*(block_dim.y+k))*sizeof(int);
    sharedSize = ((block_dim.x + (k * times)) * (block_dim.y + (k * times))) * sizeof(float) * 2;
    //sharedTam = ((block_dim.x+(k*2))*(block_dim.y+(k*2)));
    size = X * Y * sizeof(float);
    tam = X * Y;

    h_e = (float *)malloc(size);
    h_r = (float *)malloc(size);
    h_v = (float *)malloc(size);
    c_coeff = (float *)malloc((k / 2 + 1) * sizeof(float));
    hipMalloc(&d_e, size);
    hipMalloc(&d_r, size);
    hipMalloc(&d_v, size);
    hipMalloc(&d_c_coeff, (k / 2 + 1) * sizeof(float));

    printf("\n coefs \n");
    for (int i = 0; i < (k / 2 + 1); i++)
    {
        c_coeff[i] = (float)((k / 2 + 1) - i) / (float)(k / 2 + 1);
    }
    //c_coeff[0] = 0.0;
    for (int i = 0; i < (k / 2 + 1); i++)
    {
        printf(" %f", c_coeff[i]);
    }
    printf("\n coefs \n");
    hipMemcpy(d_c_coeff, c_coeff, (k / 2 + 1) * sizeof(float), hipMemcpyHostToDevice);

    //**********
    //**OPENCV**
    //**********

    //Mat orig = Mat::zeros(1024,1024,)//imread("doidera2.PNG"); //imread(argv[1]);//Pro caso de querer carregar um dominio específico, caso contrario inicia com zeros
    //Cria as imagens e as janelas do opencv
    Mat orig = Mat::zeros(X, Y, CV_32F);
    Mat result = Mat::zeros(X, Y, CV_32F);
    Window original("orig", 600, 0, 0, &orig);
    Window resultado("result", 600, 2, 0, &result);

    //**********
    //**OPENCV**
    //**********

//Copia os dados do campo e envia para a GPU e inicializa o dominio de entrada
    for (int i = 0; i < X; i++)
        for (int j = 0; j < Y; j++)
        {
            h_v[i + j * X] = 0.5f;
            orig.at<float>(Point(i, j)) = 0.5f;
            h_e[i + j * X] = (float)result.at<float>(Point(i, j));
        }
    hipMemcpy(d_v, h_v, size, hipMemcpyHostToDevice);
    while (true)
    {
        /* 
        Copy vectors from host memory to device memory
        Copia os dados da entrada de volta a GPU
         */
        hipMemcpy(d_e, h_e, size, hipMemcpyHostToDevice);
        
        /*
        Começa o Timer
        */
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);

        /******************
        *** Kernel Call ***
        *******************/
        //_3Dstencil_global<<<blks,th_p_blk>>>(d_e,d_r,X,Y,Z);
        /*
        Executa o kernel
        */
        _2Dstencil_global<<<grid_dim, block_dim, sharedSize>>>(d_e, d_r, d_v, d_c_coeff, X, Y, k, times);

        /*
        Identifica possíveis erros
        */
        hipError_t err = hipSuccess;
        err = hipGetLastError();
        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to launch _3Dstencil_global kernel (error code %s)!\n", hipGetErrorString(err));
        }
        /******************
        *** Kernel Call ***
        *******************/

        hipDeviceSynchronize();
        /*
        Para o Timer
        */
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        float elapsedTime;
        hipEventElapsedTime(&elapsedTime, start, stop);
        hipEventDestroy(start);
        hipEventDestroy(stop);

        //printf("X %d || Y %d \nBX %d || BY %d \n",X,Y,BX,BY);
        //printf ("[%d,%.5f],\n", tam,elapsedTime);
        /*
        Copia o resultado de volta para o CPU
        */
        hipMemcpy(h_r, d_r, size, hipMemcpyDeviceToHost);
        hipMemcpy(h_v, d_v, size, hipMemcpyDeviceToHost);
        /*
        Copia o resultado para a imagem de visualização
        A estrutura de 
        */
        for (int i = 0; i < X; i++)
            for (int j = 0; j < Y; j++)
            {
                result.at<float>(Point(i, j)) = (float)h_r[i + j * X];
                orig.at<float>(Point(i, j)) = h_v[i + j * X];
            }
        /*
        Original se refere ao campo v
        */
        original.imshow(orig);
        resultado.imshow(result);

        /*
        Realiza a troca das referencias para a entrada ser a saida anterior e continuar o loop
        */
        float *temp = h_e;
        h_e = h_r;
        h_r = temp;
        // waitKey(##) ## define o tempo de espera de cada frame 1 == velocidade máxima 0 == para sempre
        if (waitKey(30) == 27) //ESC == 27
        {
            break;
        }

        /*
        Atualiza a entrada caso o usuário tenha clicado na imagem e adicionado algum disturbio no domínio
        */
        for (int i = 0; i < X; i++)
            for (int j = 0; j < Y; j++)
            {
                h_e[i + j * X] = (float)result.at<float>(Point(i, j));
            }
        
    }

    hipFree(d_e);
    hipFree(d_r);
    hipFree(d_c_coeff);
    std::free(h_e);
    std::free(h_r);
    std::free(c_coeff);

    return 0;
} /* main */
