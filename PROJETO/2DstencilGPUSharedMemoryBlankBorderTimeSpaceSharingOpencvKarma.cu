#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <unistd.h>
#include <stdlib.h>
#include <math.h>

/*
COMPILE -->  nvcc 2DstencilGPUSharedMemoryBlankBorderTimeSpaceSharingOpencv.cu -o go `pkg-config --cflags --libs opencv` -w
EXECUTE --> ./main.exe
*/


//**********
//**OPENCV**
//**********
#include <iostream>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <stdio.h>
#include <opencv2/imgcodecs.hpp>
#include <math.h>
#include <string>

#define JAN_OFFSET 0
using namespace cv;
using namespace std;

//===> CONSTANTES <===//
#define Eh 3.0f
#define En 1.0f
#define Re 0.6f
#define tauE 5.0f
#define tauN 250.0f
#define gam 0.001f
#define East 1.5415f
#define DT 0.05f
#define DX ( 12.0f / 32 ) 

void CallBackFunc(int event, int x, int y, int flags, void* userdata)
{
    Mat *img = (Mat*)userdata;
     if  ( event == EVENT_LBUTTONDOWN )
     {
          cout << "Left button of the mouse is clicked - position (" << x << ", " << y << ")" << endl;
          img->at<float>(Point(x,y)) = 1.0f;
     }
     else if  ( event == EVENT_RBUTTONDOWN )
     {
         // cout << "Right button of the mouse is clicked - position (" << x << ", " << y << ")" << endl;
     }
     else if  ( event == EVENT_MBUTTONDOWN )
     {
          //cout << "Middle button of the mouse is clicked - position (" << x << ", " << y << ")" << endl;
     }
     else if ( event == EVENT_MOUSEMOVE )
     {
         // cout << "Mouse move over the window - position (" << x << ", " << y << ")" << endl;

     }
}
class Window
{
	char *m_name;

    public:
	Window(char *name, int tam_ja, int x, int y,Mat *img = NULL)
	{
		m_name = name;
		namedWindow(m_name, WINDOW_NORMAL & CV_GUI_NORMAL);
		moveWindow(m_name, tam_ja * x + JAN_OFFSET, tam_ja * y + JAN_OFFSET);
        resizeWindow(m_name, tam_ja, tam_ja);
        setMouseCallback(m_name, CallBackFunc, img);
	}
	void imshow(Mat img)
	{
		cv::imshow(m_name, img);
	}
	void createTrackbar(char *trackName, int *var, int max_val)
	{
		cv::createTrackbar(trackName, m_name, var, max_val);
	}
};

//**********
//**OPENCV**
//**********



__device__ void _2Dstencil_(float *d_e,float *d_r,float * d_v,float* c_coeff,int X,int Y,int k, int x, int y,int GX,int Gx,int Gy)
{
    int h_e_i;
    int h_r_i = x + ( y * (X) );
    h_e_i = h_r_i;
    float temp = d_e[h_r_i];

    float rv = d_v[h_r_i];
    float Rn = ( 1.0f / ( 1.0f - expf(-Re) ) ) - rv;
    float p = ( temp > En ) * 1.0f;
    float dv = ( Rn * p - ( 1.0f - p ) * rv ) / tauN;
    float Dn = rv * rv;
    float hE = ( 1.0f - tanh(temp - Eh) ) * temp * temp / 2.0f;
    float du = ( ( ( East - Dn ) * hE ) - temp ) / tauE;


    float xlapr = d_e[(x+1) + ( (y) * (X) )] - temp;
    float xlapl = temp   - d_e[(x-1) + ( (y) * (X) )];
    float xlapf = d_e[(x) + ( (y+1) * (X) )] - temp;
    float xlapb = temp   - d_e[(x) + ( (y-1) * (X) )];

    float lap = xlapr - xlapl + xlapf - xlapb;


    
    h_r_i = Gx + ( (Gy) * (GX) );
    temp = ( temp + ( du * DT ) + ( lap * DT * gam / ( DX * DX ) ) );
    //if(temp < 1.0f)
        d_r[h_r_i] = temp;
    //else
     //   d_r[h_r_i] = 1.0f;
    d_v[h_e_i] = rv + dv*DT;

    
    //temp *= c_coeff[0];
    // for(int lk =1;lk<(k/2)+1;lk++)
    // {
    //     h_e_i = (x+lk) + ( (y) * (X) );
    //     temp += d_e[h_e_i]*c_coeff[lk];

    //     h_e_i = (x-lk) + ( (y) * (X) );
    //     temp += d_e[h_e_i]*c_coeff[lk];

    //     h_e_i = (x) + ( (y+lk) * (X) );
    //     temp += d_e[h_e_i]*c_coeff[lk];

    //     h_e_i = (x) + ( (y-lk) * (X) );
    //     temp += d_e[h_e_i]*c_coeff[lk];
    // }
    //  h_r_i = Gx + ( (Gy) * (GX) );
    // if(temp < 1.0f)
    //     d_r[h_r_i] = temp;
    // else
    //     d_r[h_r_i] = 1.0f;    
}
__global__ void _2Dstencil_global(float *d_e,float *d_r,float *d_v,float *c_coeff,int X,int Y,int k,int times){

    int x,y;//,h_e_i,h_r_i,Xs,Ys,Dx,Dy;
    x = threadIdx.x + (blockIdx.x*blockDim.x);
    y = threadIdx.y + (blockIdx.y*blockDim.y);
    int k2 = k/2*times;
    extern __shared__ float shared[];
    
    int blockThreadIndex = threadIdx.x + threadIdx.y*blockDim.x;
    // Xs = threadIdx.x;
    // Ys = threadIdx.y;
    int Dx = blockDim.x+(k*times);
    int Dy = blockDim.y+(k*times);
    int sharedTam = Dx*Dy;
    float * sharedRes = &shared[sharedTam];
    
    for(int stride=blockThreadIndex;stride<sharedTam;stride+=(blockDim.x*blockDim.y))
    {
        int globalIdx = (blockIdx.x*blockDim.x)-k2+stride%Dx + ((blockIdx.y*blockDim.y)-k2+stride/Dx)*X;
        if(globalIdx > 0 && (blockIdx.x*blockDim.x)-k2+stride%Dx < X && ((blockIdx.y*blockDim.y)-k2+stride/Dx)<Y)
            shared[stride] = d_e[globalIdx];
        else
            shared[stride] = 0;
       
    }
    __syncthreads();
    for(int t=times-1;t>0;t--)
    {  
        //_2Dstencil_(shared,sharedRes,c_coeff,Dx,Dy,k,threadIdx.x+k2,threadIdx.y+k2,Dx,threadIdx.x+k2,threadIdx.y+k2);
        int tDx = blockDim.x+(t*k);
        int tDy = blockDim.y+(t*k);
        int tk2 = (times-t)*k/2;
        // int tDx = blockDim.x+(1*k);
        // int tDy = blockDim.y+(1*k);
        // int tk2 = (1)*k/2;
        int tSharedTam = tDx * tDy;
        for(int stride=blockThreadIndex;stride<tSharedTam;stride+=(blockDim.x*blockDim.y))
        {
            _2Dstencil_(shared,sharedRes,d_v,c_coeff,Dx,Dy,k,(stride%tDx)+tk2,(stride/tDx)+tk2,Dx,(stride%tDx)+tk2,(stride/tDx)+tk2);
        }
        __syncthreads();
        for(int stride=blockThreadIndex;stride<sharedTam;stride+=(blockDim.x*blockDim.y))
        {
            shared[stride]=sharedRes[stride];
        }
        __syncthreads();
   }
   
    _2Dstencil_(shared,d_r,d_v,c_coeff,Dx,Dy,k,threadIdx.x+k2,threadIdx.y+k2,X,x,y);
   
    // for(int stride=blockThreadIndex;stride<sharedTam;stride+=(blockDim.x*blockDim.y))
    // {
    //     int globalIdx = (blockIdx.x*blockDim.x)-k2+stride%Dx + ((blockIdx.y*blockDim.y)-k2+stride/Dx)*X;
    //     if(globalIdx > 0 && (blockIdx.x*blockDim.x)-k2+stride%Dx < X && ((blockIdx.y*blockDim.y)-k2+stride/Dx)<Y)
    //     d_r[globalIdx] = sharedRes[stride];
       
    // }
        
}


int main(int argc, char* argv[]) 
{

    float *h_e,*h_r,*h_v;
    float *d_e, *d_r,*d_v;
    int size,tam,sharedSize,sharedTam;
    int X=32;
    int Y=32;
    int k=4;
    int times = 1;
    int BX=32;
    int BY=32;
    int GX=1;
    int GY=1;
    float *c_coeff,*d_c_coeff;
    if(argc > 1)
    {
        X = atoi(argv[1]);
        Y = X;
    }
    if(argc > 2)
    {
        k = atoi(argv[2]);
    }

    if(argc > 3)
    {
        times = atoi(argv[3]);
    }


    if(X>32)
    {
        GX = ceil((float)X/(float)32);
        BX = 32;
    }
    if(Y>32)
    {
        GY = ceil((float)Y/(float)32);
        BY = 32;
    }
        

    dim3 block_dim(BX,BY,1);
    dim3 grid_dim(GX,GY,1);
    //sharedSize = ((block_dim.x+k)*(block_dim.y+k))*sizeof(int);
    sharedSize = ((block_dim.x+(k*times))*(block_dim.y+(k*times)))*sizeof(float)*2;
    //sharedTam = ((block_dim.x+(k*2))*(block_dim.y+(k*2)));
    size = X * Y * sizeof(float);
    tam = X * Y;


    h_e = (float*) malloc(size);
    h_r = (float*) malloc(size);
    h_v = (float*) malloc(size);
    c_coeff = (float*)malloc((k/2+1)*sizeof(float));
    hipMalloc(&d_e, size);
    hipMalloc(&d_r, size);
    hipMalloc(&d_v, size);
    hipMalloc(&d_c_coeff,(k/2+1)*sizeof(float));

    printf("\n coefs \n");
    for(int i=0;i<(k/2+1);i++)
    {
        c_coeff[i]=(float)((k/2+1)-i)/(float)(k/2+1);
    
    }
    //c_coeff[0] = 0.0;
    for(int i=0;i<(k/2+1);i++)
    {
        printf(" %f",c_coeff[i]);
    }
    printf("\n coefs \n");

    //**********
    //**OPENCV**
    //**********
    if (argc < 2)
        {
            printf("\nespecifique a imagem\n");
            return -1;
        }
        
        //Mat orig = Mat::zeros(1024,1024,)//imread("doidera2.PNG"); //imread(argv[1]);
        Mat orig = Mat::zeros(X,Y, CV_32F);
        Mat result = Mat::zeros(X,Y, CV_32F);
        Window original("orig", 600, 0, 0,&orig);
        Window resultado("result", 600, 2, 0,&result);
        
    //**********
    //**OPENCV**
    //**********
    for(int i=0;i<X;i++)
        for(int j=0;j<Y;j++)
        {
            h_v[i+j*X] =0.5f;
            orig.at<float>(Point(i,j)) = 0.5f;
        }
        hipMemcpy(d_v, h_v, size, hipMemcpyHostToDevice);
    while (true)
    {
        for(int i=0;i<X;i++)
        for(int j=0;j<Y;j++)
        {
            h_e[i+j*X] = (float)result.at<float>(Point(i,j));
            orig.at<float>(Point(i,j)) = h_v[i+j*X];
        }
        /* Copy vectors from host memory to device memory */
    hipMemcpy(d_e, h_e, size, hipMemcpyHostToDevice);
    hipMemcpy(d_c_coeff, c_coeff, (k/2+1)*sizeof(float), hipMemcpyHostToDevice);

        hipEvent_t start, stop;
        hipEventCreate (&start);
        hipEventCreate (&stop);
        hipEventRecord (start, 0); 

        /******************
        *** Kernel Call ***
        *******************/
        //_3Dstencil_global<<<blks,th_p_blk>>>(d_e,d_r,X,Y,Z);
        _2Dstencil_global<<<grid_dim,block_dim,sharedSize>>>(d_e,d_r,d_v,d_c_coeff,X,Y,k,times);

        hipError_t err = hipSuccess;
        err = hipGetLastError();
        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to launch _3Dstencil_global kernel (error code %s)!\n", hipGetErrorString(err));
        }
        /******************
        *** Kernel Call ***
        *******************/



        hipDeviceSynchronize();
        hipEventRecord (stop, 0);
        hipEventSynchronize (stop);
        float elapsedTime;
        hipEventElapsedTime (&elapsedTime, start, stop);
        hipEventDestroy(start);
        hipEventDestroy(stop);


        //printf("X %d || Y %d \nBX %d || BY %d \n",X,Y,BX,BY);
        //printf ("[%d,%.5f],\n", tam,elapsedTime);

        hipMemcpy(h_r, d_r, size, hipMemcpyDeviceToHost);
        hipMemcpy(h_v, d_v, size, hipMemcpyDeviceToHost);

        for(int i=0;i<X;i++)
            for(int j=0;j<Y;j++)
                result.at<float>(Point(i,j)) = (float)h_r[i+j*X];

        original.imshow(orig);
        resultado.imshow(result);
        

        float *temp = h_e;
        h_e = h_r;
        h_r = temp;
        // Wait for key is pressed then break loop
        if (waitKey(1) == 27) //ESC == 27
        {
            break;
        }
        }


        hipFree(d_e);
        hipFree(d_r);
        hipFree(d_c_coeff);
        std::free(h_e);
        std::free(h_r);
        std::free(c_coeff);


        

        return 0;
} /* main */
