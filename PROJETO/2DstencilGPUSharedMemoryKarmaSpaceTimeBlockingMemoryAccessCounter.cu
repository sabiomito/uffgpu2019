
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <unistd.h>
#include <stdlib.h>
#include <math.h>


/*
Instruções
COMPILAR -->  nvcc 2DstencilGPUSharedMemoryBlankBorderTimeSpaceSharingOpencvKarma.cu -o go `pkg-config --cflags --libs opencv` -w
EXECUTAR --> ./go DOMAIN_DIMS STENCIL_ORDER SPACE_TIME_BLOCK_TIMES BLOCK_DIM_X BLOCK_DIM_Y
*/

#include <iostream>
#include <fstream>
#include <stdio.h>
#include <math.h>
#include <string>

using namespace std;

//===> CONSTANTES karma model <===//
#ifndef MODEL_WIDTH
#define MODEL_WIDTH 96
#endif

#ifndef BLOCK_TIMES
#define BLOCK_TIMES 1
#endif

#define Eh 3.0f
#define En 1.0f
#define Re 0.6f
#define tauE 5.0f
#define tauN 250.0f
#define gam 0.001f
#define East 1.5415f
#define DT 0.05f
#define DX (12.0f / MODEL_WIDTH)

#define MODELSIZE_X (MODEL_WIDTH)
#define MODELSIZE_Y (MODEL_WIDTH)
#define MODELSIZE_Z 1
#define MODELSIZE2D ( MODELSIZE_X*MODELSIZE_Y )

#ifndef BLOCKDIM_X
#define BLOCKDIM_X 32
#endif

#ifndef MEM_TYPE
#define MEM_TYPE 0
#endif


#ifndef BLOCKDIM_Y
#define BLOCKDIM_Y 32
#endif

#define BLOCKDIM_Z 1
#define BLOCKDIM2D ( BLOCKDIM_X*BLOCKDIM_Y )

//==> CUDA GRID <==//
#define GRIDDIM_X ( ( MODELSIZE_X / BLOCKDIM_X ) + ( ( MODELSIZE_X % BLOCKDIM_X ) > 0 ) )
#define GRIDDIM_Y ( ( MODELSIZE_Y / BLOCKDIM_Y ) + ( ( MODELSIZE_Y % BLOCKDIM_Y ) > 0 ) )
#define GRIDDIM_Z 1

#define SHARED_TAM ((BLOCKDIM_X + (2 * BLOCK_TIMES)) * (BLOCKDIM_Y + (2 * BLOCK_TIMES)))
#define SHARED_DX (BLOCKDIM_X + (2 * BLOCK_TIMES))
#define SHARED_DY (BLOCKDIM_Y + (2 * BLOCK_TIMES))
/*
Função somente da GPU que recebe os parametros para o calculo de um stencil
d_e - dado de entrada
d_r - dado de saida
d_v - campo que deve ser atualizado
c_coeff - variável utilizada para armazenar o valores dos coeficcientes do stencil (utilizada apenas na versão com stencil simples usado anteriormente)
X - Y - Dimensões das estruturas de entrada
k - ordem do stencil
x -y - posição do centro do stencil na estrutura de entrada
GX - Dimensão horizontal da estrutura do dado de saída
Gx - Gy posição do centro do stencil na estrutura de saida
*/
__forceinline__ __device__ void _2Dstencil_(float *d_e, float *d_r, float *d_v, int X, int x, int y, int GX, int Gx, int Gy)
{
    int h_e_i = x + (y * (X));

    
    float temp = d_e[h_e_i];
    float rv = d_v[h_e_i];


    float Rn = (1.0f / (1.0f - expf(-Re))) - rv;
    float p = (temp > En) * 1.0f;
    float dv = (Rn * p - (1.0f - p) * rv) / tauN;
    float Dn = rv * rv;
    float hE = (1.0f - tanh(temp - Eh)) * temp * temp / 2.0f;
    float du = (((East - Dn) * hE) - temp) / tauE;

    float xlapr = d_e[(x + 1) + ((y) * (X))] - temp;
    float xlapl = temp - d_e[(x - 1) + ((y) * (X))];
    float xlapf = d_e[(x) + ((y + 1) * (X))] - temp;
    float xlapb = temp - d_e[(x) + ((y - 1) * (X))];

    float lap = xlapr - xlapl + xlapf - xlapb;
   
    temp = (temp + (du * DT) + (lap * DT * gam / (DX * DX)));

    d_v[h_e_i] = rv + dv * DT;
    h_e_i = Gx + ((Gy) * (GX));
    d_r[h_e_i] = temp;
}
/*
função chamada pelo host que controla as cópias e a ordem do calculo dos stencils bem como a carga para cada thread
, MODELSIZE_X, MODELSIZE_Y, BLOCK_TIMES
 int X, int Y, int times
*/
__global__ void _2Dstencil_global(float *d_e, float *d_r, float *d_v,int * sharedAcesses,int * globalAcesses)
{
    
    int x, y; //,h_e_i,h_r_i,Xs,Ys,Dx,Dy;
    x = threadIdx.x + (blockIdx.x * BLOCKDIM_X);
    y = threadIdx.y + (blockIdx.y * BLOCKDIM_Y);
    extern __shared__ float sharedOrig[];

    int blockThreadIndex = threadIdx.x + threadIdx.y * BLOCKDIM_X;

    float * shared = sharedOrig;
    float * sharedRes = shared + SHARED_TAM;
    float * sharedV = sharedRes + SHARED_TAM; 
    /*
    Copia o Tile de memória compartilhada necessária para a configuração de tempo desejada
    Stride é utilizado pois a quantidade de elementos a serem copiados é sempre maior que a quantidade de threads
    As bordas
    */
    for (int stride = blockThreadIndex; stride < SHARED_TAM; stride += (BLOCKDIM_X * BLOCKDIM_Y))
    {
        int sharedIdxX = stride % SHARED_DX;
        int sharedIdxY = int(stride / SHARED_DX);
        int globalIdxX = (blockIdx.x * BLOCKDIM_X) + sharedIdxX - BLOCK_TIMES;
        int globalIdxY = (blockIdx.y * BLOCKDIM_Y) + sharedIdxY - BLOCK_TIMES;
        int globalIdx = globalIdxX + (-1*globalIdxX)*(globalIdxX < 0) - (globalIdxX-MODELSIZE_X+1)*(globalIdxX >= MODELSIZE_X)  +  (globalIdxY + (-1*globalIdxY)*(globalIdxY < 0) - (globalIdxY-MODELSIZE_Y+1)*(globalIdxY >= MODELSIZE_Y)) * MODELSIZE_X;
       
        atomicAdd(globalAcesses,2);
        atomicAdd(sharedAcesses,2);
        shared[stride] = d_e[globalIdx];
        sharedV[stride] = d_v[globalIdx];
    }

    __syncthreads();

    /*
    Envia pra ser calculado todos os elementos além do ultimo instante de tempo
    */
    for (int t = 1; t < BLOCK_TIMES; t++)
    {
        int tDx = BLOCKDIM_X + ((BLOCK_TIMES - t) * 2);
        int tDy = BLOCKDIM_Y + ((BLOCK_TIMES - t) * 2);
        int tk2 = (t);
        int tSharedTam = tDx * tDy;
        for (int stride = blockThreadIndex; stride < tSharedTam; stride += (BLOCKDIM_X * BLOCKDIM_Y))
        {
            int tempX = (stride % tDx) + tk2;
            int tempY = (int(stride / tDx)) + tk2;
            _2Dstencil_(shared, sharedRes, sharedV, SHARED_DX, tempX, tempY, SHARED_DX, tempX, tempY);
            atomicAdd(sharedAcesses,8);
            
        }

        float * temp = shared;
        shared = sharedRes;
        sharedRes = temp;
        __syncthreads();
    }
    /*
    Envia pra ser calculado todos os elementos do ultimo instante de tempo
   */
    _2Dstencil_(shared, d_r, sharedV, SHARED_DX, ((x%(BLOCKDIM_X))+BLOCK_TIMES), ((y%(BLOCKDIM_Y))+BLOCK_TIMES), MODELSIZE_X, x, y);
    atomicAdd(globalAcesses,1);
    atomicAdd(sharedAcesses,7);

     int globalIdx = x + y * MODELSIZE_X;
     int sharedIdx = ((x%(BLOCKDIM_X))+BLOCK_TIMES) + ((y%(BLOCKDIM_Y))+BLOCK_TIMES)*SHARED_DX;
     atomicAdd(globalAcesses,1);
     atomicAdd(sharedAcesses,1);
     d_v[globalIdx] = sharedV[sharedIdx];
}

int main(int argc, char *argv[])
{
    /*
    Declarações e valores padroes
    */
    float *h_e, *h_v;
    float *d_e, *d_r, *d_v;
    int sharedSize;
    int globalTimes = 1;
    int * h_sharedAcesses;
    int * h_globalAcesses;
    int * d_sharedAcesses;
    int * d_globalAcesses;
    /*
    Obtenção dos parâmetros de entrada
    */
    if (argc > 1)
    {
        globalTimes = atoi(argv[1]);
    }


    /*
    Allocações de memória e configuração dos blocos e grid
    */
    dim3 grid_dim(GRIDDIM_X,GRIDDIM_Y,GRIDDIM_Z);
    dim3 block_dim(BLOCKDIM_X,BLOCKDIM_Y,BLOCKDIM_Z);
    sharedSize = SHARED_TAM * sizeof(float) * 3;
    h_e = (float *)malloc(MODELSIZE2D*sizeof(float));
    h_v = (float *)malloc(MODELSIZE2D*sizeof(float));


    hipMalloc(&d_sharedAcesses, sizeof(int));
    hipMalloc(&d_globalAcesses, sizeof(int));

    h_sharedAcesses = (int *)malloc(sizeof(int));
    h_globalAcesses = (int *)malloc(sizeof(int));
    *h_sharedAcesses = 0.0f;
    *h_globalAcesses = 0.0f;
    hipMemcpy(d_sharedAcesses, h_sharedAcesses,sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_globalAcesses, h_globalAcesses,sizeof(int), hipMemcpyHostToDevice);


    hipMalloc(&d_e, MODELSIZE2D*sizeof(float));
    hipMalloc(&d_r, MODELSIZE2D*sizeof(float));
    hipMalloc(&d_v, MODELSIZE2D*sizeof(float));


    /******************
    *** Kernel Call ***
    *******************/
    //_3Dstencil_global<<<blks,th_p_blk>>>(d_e,d_r,X,Y,Z);
    /*
    Executa o kernel
    */
    for(int i=0; i<globalTimes/BLOCK_TIMES; i ++)
    {
        _2Dstencil_global<<<grid_dim, block_dim, sharedSize>>>(d_e, d_r, d_v,d_sharedAcesses,d_globalAcesses);
        float * temp = d_e;
        d_e = d_r;
        d_r = temp;
    }
    

    /*
    Identifica possíveis erros
    */
    hipError_t err = hipSuccess;
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf ("-1");
        hipFree(d_e);
        hipFree(d_r);
        hipFree(d_v);
        std::free(h_e);
        std::free(h_v);
        hipFree(d_sharedAcesses);
        hipFree(d_globalAcesses);
        std::free(h_sharedAcesses);
        std::free(h_globalAcesses);
        fprintf(stderr, "Failed to launch _2Dstencil_global kernel (error code %s)!\n", hipGetErrorString(err));
        return 0;
    }
    /******************
    *** Kernel Call ***
    *******************/




    //cudaMemcpy(h_e, d_e, MODELSIZE2D*sizeof(float), cudaMemcpyDeviceToHost);
    hipMemcpy(h_sharedAcesses, d_sharedAcesses,sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_globalAcesses, d_globalAcesses,sizeof(int), hipMemcpyDeviceToHost);
    if(MEM_TYPE == 0)
    printf ("%d",*h_globalAcesses);
    else if(MEM_TYPE == 1)
    printf ("%d",*h_sharedAcesses);
    hipFree(d_e);
    hipFree(d_r);
    hipFree(d_v);
    std::free(h_e);
    std::free(h_v);
    hipFree(d_sharedAcesses);
    hipFree(d_globalAcesses);
    std::free(h_sharedAcesses);
    std::free(h_globalAcesses);

    return 0;
} /* main */