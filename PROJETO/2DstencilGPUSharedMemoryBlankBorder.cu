
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <unistd.h>
#include <stdlib.h>
#include <math.h>
using namespace std;

__global__ void _copy_dr_to_de(int *d_e,int *d_r,int X,int Y){
    int x,y;
    x = threadIdx.x + (blockIdx.x*blockDim.x);
    y = threadIdx.y + (blockIdx.y*blockDim.y);
    int h_r_i = x + ( y * (X) );
    if(x<X && y<Y)
        d_e[h_r_i] = d_r[h_r_i];
}
__device__ void _2Dstencil_(int *d_e,int *d_r,float* c_coeff,int X,int Y,int k, int x, int y,int GX,int Gx,int Gy)
{     
    int h_e_i;
    int h_r_i = x + ( y * (X) );
    h_e_i = h_r_i;
    int temp = d_e[h_r_i];
    temp *= c_coeff[0];
    for(int lk =1;lk<(k/2)+1;lk++)
    {
        h_e_i = (x+lk) + ( (y) * (X) );
        temp += d_e[h_e_i]*c_coeff[lk];

        h_e_i = (x-lk) + ( (y) * (X) );
        temp += d_e[h_e_i]*c_coeff[lk];

        h_e_i = (x) + ( (y+lk) * (X) );
        temp += d_e[h_e_i]*c_coeff[lk];

        h_e_i = (x) + ( (y-lk) * (X) );
        temp += d_e[h_e_i]*c_coeff[lk];
    }
     h_r_i = Gx + ( (Gy) * (GX) );
    d_r[h_r_i] = temp;    
}
__global__ void _2Dstencil_global(int *d_e,int *d_r,float *c_coeff,int X,int Y,int k,int sharedTam){

    int x,y;//,h_e_i,h_r_i,Xs,Ys,Dx,Dy;
    x = threadIdx.x + (blockIdx.x*blockDim.x);
    y = threadIdx.y + (blockIdx.y*blockDim.y);
    int k2 = k/2;
    extern __shared__ int shared[];
    int blockThreadIndex = threadIdx.x + threadIdx.y*blockDim.x;
    // Xs = threadIdx.x;
    // Ys = threadIdx.y;
    int Dx = blockDim.x+k;
    int Dy = blockDim.y+k;

    
    for(int stride=blockThreadIndex;stride<sharedTam;stride+=(blockDim.x*blockDim.y))
    {
        int globalIdx = (blockIdx.x*blockDim.x)-k2+stride%Dx + ((blockIdx.y*blockDim.y)-k2+stride/Dx)*X;
        if(globalIdx > 0 && (blockIdx.x*blockDim.x)-k2+stride%Dx < X && ((blockIdx.y*blockDim.y)-k2+stride/Dx)<Y)
            shared[stride] = d_e[globalIdx];
        else
            shared[stride] = 0;
       
    }
    __syncthreads();
    _2Dstencil_(shared,d_r,c_coeff,Dx,Dy,k,threadIdx.x+k2,threadIdx.y+k2,X,x,y);
    
}


int main(int argc, char* argv[]) {

int *h_e,*h_r;
int *d_e, *d_r;
int size,tam,sharedSize,sharedTam;
int X=32;
int Y=32;
int k=4;
int times = 1;
int BX=32;
int BY=32;
int GX=1;
int GY=1;
float *c_coeff,*d_c_coeff;
if(argc > 1)
{
    X = atoi(argv[1]);
    Y = X;
}
if(argc > 2)
{
    k = atoi(argv[2]);
}

if(argc > 3)
{
    times = atoi(argv[3]);
}


if(X>32)
{
    GX = ceil((float)X/(float)32);
    BX = 32;
}
if(Y>32)
{
    GY = ceil((float)Y/(float)32);
    BY = 32;
}
    

dim3 block_dim(BX,BY,1);
dim3 grid_dim(GX,GY,1);
//sharedSize = ((block_dim.x+k)*(block_dim.y+k))*sizeof(int);
sharedSize = ((block_dim.x+k)*(block_dim.y+k))*sizeof(int);
sharedTam = ((block_dim.x+k)*(block_dim.y+k));
size = X * Y * sizeof(int);
tam = X * Y;


h_e = (int*) malloc(size);
h_r = (int*) malloc(size);
c_coeff = (float*)malloc((k/2+1)*sizeof(float));
hipMalloc(&d_e, size);
hipMalloc(&d_r, size);
hipMalloc(&d_c_coeff,(k/2+1)*sizeof(float));

printf("\n coefs \n");
for(int i=0;i<(k/2+1);i++)
{
    c_coeff[(k/2+1)-i-1]=(float)i/(float)(k/2+1);
    printf(" %f",c_coeff[(k/2+1)-i-1]);
}
printf("\n coefs \n");


FILE *arq;
arq = fopen("entrada.txt", "rt");
for(int i=0;i<X;i++)
    for(int j=0;j<Y;j++)
        fscanf(arq," %d",&h_e[i+j*X]);
fclose(arq);


/* Copy vectors from host memory to device memory */
hipMemcpy(d_e, h_e, size, hipMemcpyHostToDevice);
hipMemcpy(d_c_coeff, c_coeff, (k/2+1)*sizeof(float), hipMemcpyHostToDevice);


hipEvent_t start, stop;
hipEventCreate (&start);
hipEventCreate (&stop);
hipEventRecord (start, 0); 

/******************
*** Kernel Call ***
*******************/
//_3Dstencil_global<<<blks,th_p_blk>>>(d_e,d_r,X,Y,Z);
for(int t=0;t<times;t++)
{
_2Dstencil_global<<<grid_dim,block_dim,sharedSize>>>(d_e,d_r,d_c_coeff,X,Y,k,sharedTam);
_copy_dr_to_de<<<grid_dim,block_dim>>>(d_e,d_r,X,Y);
}
hipError_t err = hipSuccess;
err = hipGetLastError();
if (err != hipSuccess)
{
    fprintf(stderr, "Failed to launch _3Dstencil_global kernel (error code %s)!\n", hipGetErrorString(err));
}
/******************
*** Kernel Call ***
*******************/



    hipDeviceSynchronize();
    hipEventRecord (stop, 0);
    hipEventSynchronize (stop);
    float elapsedTime;
    hipEventElapsedTime (&elapsedTime, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);


    printf("X %d || Y %d \nBX %d || BY %d \n",X,Y,BX,BY);
    printf ("[%d,%.5f],\n", tam,elapsedTime);
 
    hipMemcpy(h_r, d_r, size, hipMemcpyDeviceToHost);

    
arq = fopen("resultado.txt", "wt");
for(int i=0;i<X;i++)
{
    for(int j=0;j<Y;j++)
    {
      fprintf(arq," %d",h_r[i+j*X]);
    }
    fprintf(arq,"\n");
}
fclose(arq);


    hipFree(d_e);
    hipFree(d_r);
    hipFree(d_c_coeff);
    std::free(h_e);
    std::free(h_r);
    std::free(c_coeff);

    return 0;
} /* main */



/*
for(int lk = 1;lk<(k/2)+1;lk++)
    {
        if(x+lk < X)
        {
            if((x+lk)/Dx == blockIdx.x)
            {
                h_e_i = ((x+lk)%Dx) + ( (Ys) * (Dx) );
                temp += shared[h_e_i]*c_coeff[lk];
            }else
            {
                h_e_i = (x+lk) + ( (y) * (X) );
                temp += d_e[h_e_i]*c_coeff[lk];
            }
            
        }
        if(x-lk >= 0)
        {
            if((x-lk)/Dx == blockIdx.x)
            {
                h_e_i = ((x-lk)%Dx) + ( (Ys) * (Dx) );
                temp += shared[h_e_i]*c_coeff[lk];
            }
            else
            {
                h_e_i = (x-lk) + ( (y) * (X) );
                temp += d_e[h_e_i]*c_coeff[lk];
            }
               
        }
        if(y+lk < Y)
        {
            if((y+lk)/Dy == blockIdx.y)
            {
                h_e_i = ((Xs) + ( ((y+lk)%Dy) * (Dx) ));
                temp += shared[h_e_i]*c_coeff[lk];
            }
            else
            {
                h_e_i = (x) + ( (y+lk) * (X) );
                temp += d_e[h_e_i]*c_coeff[lk];
            }
        }
        if(y-lk >= 0)
        {
            if((y-lk)/Dy == blockIdx.y)
            {
                h_e_i = ((Xs) + ( ((y-lk)%Dy) * (Dx) ));
                temp += shared[h_e_i]*c_coeff[lk];
            }
            else
            {
                h_e_i = (x) + ( (y-lk) * (X) );
                temp += d_e[h_e_i]*c_coeff[lk];
            }
        }
    }
    d_r[h_r_i] = temp;  
*/