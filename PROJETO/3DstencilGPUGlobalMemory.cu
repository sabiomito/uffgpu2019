
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <unistd.h>
#include <stdlib.h>
#include <math.h>
using namespace std;

__global__ void _3Dstencil_global(float *d_e,float *d_r,int X,int Y,int Z,int k){

    //int thread_id = threadIdx.x + blockIdx.x * blockDim.x;
    //printf("sou id %d || threadIdx.x %d || blockIdx.x %d || blockDim.x %d \n",thread_id,threadIdx.x ,blockIdx.x,blockDim.x);
    //int thread_id = threadIdx.x + threadIdx.y*blockDim.x + (blockIdx.x + blockIdx.y*gridDim.x)*blockDim.x*blockDim.y;
    //printf("sou id %d || threadIdx.x %d || blockIdx.x %d|| blockIdx.y %d || blockDim.x %d|| blockDim.y %d \n",thread_id,threadIdx.x ,blockIdx.x,blockIdx.y,blockDim.x,blockDim.y);
    int x,y;
    x = threadIdx.x + (blockIdx.x*blockDim.x);
    y = threadIdx.y + (blockIdx.y*blockDim.y);
    //printf("X = %d || Y = %d\n",x,y);
    for(int z=0;  z<Z;  z++)
    {           
        int h_r_i = x + ( y * (X) ) + ( z* (X*Y) );
        int h_e_i = h_r_i;
        d_r[h_r_i] = d_e[h_e_i];
        for(int lk =0;lk<(k/2);lk++)
            {
                if(x+lk >= X)
                    h_e_i = (x-lk) + ( (y) * (X) ) + ( (z) * (X*Y) );
                else
                    h_e_i = (x+lk) + ( (y) * (X) ) + ( (z) * (X*Y) );
                d_r[h_r_i] += d_e[h_e_i];

                if(x-lk < 0)
                    h_e_i = (x+lk) + ( (y) * (X) ) + ( (z) * (X*Y) );
                else
                    h_e_i = (x-lk) + ( (y) * (X) ) + ( (z) * (X*Y) );
                d_r[h_r_i] += d_e[h_e_i];


                if(y+lk >= Y)
                    h_e_i = (x) + ( (y-lk) * (X) ) + ( (z) * (X*Y) );
                else
                    h_e_i = (x) + ( (y+lk) * (X) ) + ( (z) * (X*Y) );
                d_r[h_r_i] += d_e[h_e_i];

                if(y-lk < 0)
                    h_e_i = (x) + ( (y+lk) * (X) ) + ( (z) * (X*Y) );
                else
                    h_e_i = (x) + ( (y-lk) * (X) ) + ( (z) * (X*Y) );
                d_r[h_r_i] += d_e[h_e_i];


                if(z+lk >= Z)
                    h_e_i = (x) + ( (y) * (X) ) + ( (z-lk) * (X*Y) );
                else
                    h_e_i = (x) + ( (y) * (X) ) + ( (z+lk) * (X*Y) );
                d_r[h_r_i] += d_e[h_e_i];

                if(z-lk < 0)
                    h_e_i = (x) + ( (y) * (X) ) + ( (z+lk) * (X*Y) );
                else
                    h_e_i = (x) + ( (y) * (X) ) + ( (z-lk) * (X*Y) );
                d_r[h_r_i] += d_e[h_e_i];

            }  
    }


}

/*
*argumentos
*1 - n_elementos
*2 - threads por bloco
*3 - n_blocos
*4 - print
*/
int main(int argc, char* argv[]) {

    float *h_e,*h_r,*h_r_test;
    float *d_e, *d_r;
    int size,tam,times;
    clock_t Ticks[2];

    

    

    times = 1;
    int X=8;
    int Y=8;
    int BX=8;
    int BY=8;
    int Z=4;
    int k=2;
    int GX=1;
    int GY=1;

    if(argc > 1)
    {
        X = atoi(argv[1]);
        BX=X;
    }
      
    if(argc > 2)
    {
        Y = atoi(argv[2]);
        BY = Y;
    }
      
    if(argc > 3)
      Z = atoi(argv[3]);
    if(argc > 4)
      k = atoi(argv[4]);

    if(X>32)
    {
        GX = ceil((float)X/(float)32);
        BX = 32;
    }
    if(Y>32)
    {
        GY = ceil((float)Y/(float)32);
        BY = 32;
    }
    
    
    dim3 block_dim(BX,BY,1);
    dim3 grid_dim(GX,GY,1);

    size = X * Y * Z * sizeof(float);
    tam = X * Y * Z;


    h_e = (float*) malloc(size);
    h_r = (float*) malloc(size);
    h_r_test = (float*) malloc(size);
    hipMalloc(&d_e, size);
    hipMalloc(&d_r, size);


    for (int i = 0; i < tam; i++) {
        h_e[i] = (float)(rand()%9000)/100.0;
        h_r[i] = 0;
    }

    /* Copy vectors from host memory to device memory */
    hipMemcpy(d_e, h_e, size, hipMemcpyHostToDevice);
    hipMemcpy(d_r, h_r, size, hipMemcpyHostToDevice);

    for(int t =0; t<times; t++)
    {

        for(int z=0;  z<Z;  z++)
        {
        
            for(int y=0;  y<Y;  y++)
            {
                for(int x=0;  x<X;  x++)
                {
                    
                    
                    int h_r_i = x + ( y * (X) ) + ( z* (X*Y) );
                        
                    int h_e_i = h_r_i;
                    h_r_test[h_r_i] = h_e[h_e_i];
                    for(int lk =0;lk<(k/2);lk++)
                        {
                            

                            
                            if(x+lk >= X)
                                h_e_i = (x-lk) + ( (y) * (X) ) + ( (z) * (X*Y) );
                            else
                                h_e_i = (x+lk) + ( (y) * (X) ) + ( (z) * (X*Y) );
                            h_r_test[h_r_i] += h_e[h_e_i];

                            if(x-lk < 0)
                                h_e_i = (x+lk) + ( (y) * (X) ) + ( (z) * (X*Y) );
                            else
                                h_e_i = (x-lk) + ( (y) * (X) ) + ( (z) * (X*Y) );
                            h_r_test[h_r_i] += h_e[h_e_i];


                            if(y+lk >= Y)
                                h_e_i = (x) + ( (y-lk) * (X) ) + ( (z) * (X*Y) );
                            else
                                h_e_i = (x) + ( (y+lk) * (X) ) + ( (z) * (X*Y) );
                            h_r_test[h_r_i] += h_e[h_e_i];

                            if(y-lk < 0)
                                h_e_i = (x) + ( (y+lk) * (X) ) + ( (z) * (X*Y) );
                            else
                                h_e_i = (x) + ( (y-lk) * (X) ) + ( (z) * (X*Y) );
                            h_r_test[h_r_i] += h_e[h_e_i];


                            if(z+lk >= Z)
                                h_e_i = (x) + ( (y) * (X) ) + ( (z-lk) * (X*Y) );
                            else
                                h_e_i = (x) + ( (y) * (X) ) + ( (z+lk) * (X*Y) );
                            h_r_test[h_r_i] += h_e[h_e_i];

                            if(z-lk < 0)
                                h_e_i = (x) + ( (y) * (X) ) + ( (z+lk) * (X*Y) );
                            else
                                h_e_i = (x) + ( (y) * (X) ) + ( (z-lk) * (X*Y) );
                            h_r_test[h_r_i] += h_e[h_e_i];

                        }  
                }
            }
            
        }

        for (int i = 0; i < tam; i++) 
        {
            h_e[i] = h_r_test[i];
        }

    }


    hipEvent_t start, stop;
    hipEventCreate (&start);
    hipEventCreate (&stop);
    hipEventRecord (start, 0); 

    /******************
    *** Kernel Call ***
    *******************/
    //_3Dstencil_global<<<blks,th_p_blk>>>(d_e,d_r,X,Y,Z);
    _3Dstencil_global<<<grid_dim,block_dim>>>(d_e,d_r,X,Y,Z,k);

    hipError_t err = hipSuccess;
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch _3Dstencil_global kernel (error code %s)!\n", hipGetErrorString(err));
    }
    /******************
    *** Kernel Call ***
    *******************/



    hipDeviceSynchronize();
    hipEventRecord (stop, 0);
    hipEventSynchronize (stop);
    float elapsedTime;
    hipEventElapsedTime (&elapsedTime, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    
    Ticks[1] = clock();
    double Tempo = (Ticks[1] - Ticks[0]) * 1000.0 / CLOCKS_PER_SEC;
    printf("X %d || Y %d \nBX %d || BY %d\nZ %d \n",X,Y,BX,BY,Z);
    printf ("[%d,%.5f,%lf],\n", tam,elapsedTime,Tempo/1000.0);
 
    hipMemcpy(h_r, d_r, size, hipMemcpyDeviceToHost);

    bool certo=true;
    //printf("threads/blk %d -- blocks %d\n",th_p_blk,blks);
    for (int i = 0; i < 256; i++){
        //printf("%d - %d\n",h_z_res[i],h_z[i]);
        if(h_r_test[i] != h_r[i])
          certo=false;
    }
    if(!certo)
    printf("\n*****\n certo = %s\n*****\n", certo ? "true" : "false");

    hipFree(d_e);
    hipFree(d_r);
    std::free(h_e);
    std::free(h_r);
    std::free(h_r_test);

    return 0;
} /* main */
