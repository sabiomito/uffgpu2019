

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <unistd.h>
#include <stdlib.h>
#include <math.h>
#include <hiprand/hiprand.h>
#include <time.h> 
/**
1- Gerar uma matriz aleatoria
2- Aplicar um blur ou filtro (gerar uma nova matriz de saida, com a media aritimetica da vizinhanca aplicada a cada elemento da matriz)
3- Testar e mandar resultados de tempo para os segintes casos:
- memoria unificada
- copia manual de memoria
- usando stream para copia CPU->GPU
- usando streams para os dois sentidos de copias (ida e volta)
4- Testar para matrizes de 100x100 , 1000x1000, 10000x10000
**/
__global__ void blur(unsigned int origData[],unsigned result[],int L) {


    int thread_idx = threadIdx.x + blockIdx.x * blockDim.x;
    int thread_idy = threadIdx.y + blockIdx.y * blockDim.y;
    
    
    if(thread_idx-1 >= 0 && thread_idx+1 <= L && thread_idy-1 >= 0 && thread_idy+1 <= L)
    {
        int temp = origData[(thread_idx) + (thread_idy)*L];
        temp += origData[(thread_idx-1) + (thread_idy-1)*L];
        temp += origData[(thread_idx) + (thread_idy-1)*L];
        temp += origData[(thread_idx+1) + (thread_idy-1)*L];

        temp += origData[(thread_idx-1) + (thread_idy)*L];
        //temp += origData[(thread_idx) + (thread_idy)*L];
        temp += origData[(thread_idx+1) + (thread_idy)*L];

        temp += origData[(thread_idx-1) + (thread_idy+1)*L];
        temp += origData[(thread_idx) + (thread_idy+1)*L];
        temp += origData[(thread_idx+1) + (thread_idy+1)*L];
        result[(thread_idx) + (thread_idy)*L] = temp/9;
    }else
    {
        result[(thread_idx) + (thread_idy)*L] = origData[(thread_idx) + (thread_idy)*L];
    }
        
}
/*
*argumentos
*1 - n_elementos
*2 - threads por bloco
*/
int main(int argc, char* argv[]) {
    unsigned int L,G, tam, *h_data,*d_data,*d_res;
    size_t size;
    hiprandGenerator_t gen;

    unsigned int th_p_blk = 1024;
    L = 40;
    G = 1;
    if(argc > 1)
      L = atoi(argv[1]);
    if(argc > 2)
      G = atoi(argv[2]);

    tam = L*L;
    size = tam*sizeof(unsigned int);

    dim3 block_dim(ceil(L/G),ceil(L/G),1);
    dim3 grid_dim(G,G,1);

    bool print = false;
    if(L<33)
        print = true;
    // Allocate memory for the vectors on host memory.
    h_data = (unsigned int*) malloc(size);

    for (int i = 0; i < tam; i++)
        h_data[i] = 0;
    
    

    /* Allocate vectors in device memory */
    hipMalloc(&d_data, size);
    hipMalloc(&d_res, size);
    hiprandCreateGenerator(&gen,HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen,time(NULL));
    hiprandGenerate(gen,d_data, size);
    

    hipMemcpy(h_data, d_data, size, hipMemcpyDeviceToHost);
    if(print)
    {
        printf("\n\n");
        for (int i = 0; i < tam; i++)
        {
            if(i%L==0)
                printf("\n");
            printf(" %u",h_data[i]);
        }
        printf("\n\n");
    }

    hipEvent_t start, stop;
    hipEventCreate (&start);
    hipEventCreate (&stop);
    hipEventRecord (start, 0); // 0 is the stream number
    // do Work…
    
    /* Kernel Call */
    blur<<<grid_dim,block_dim>>>(d_data, d_res, L);
    
    hipDeviceSynchronize();
    
    hipEventRecord (stop, 0);
    
    hipEventSynchronize (stop);
    
    float elapsedTime;
    hipEventElapsedTime (&elapsedTime, start, stop);
    printf ("[%d,%.5f],\n", tam,elapsedTime);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
 
    hipMemcpy(h_data, d_res, size, hipMemcpyDeviceToHost);
    if(print)
    {
        printf("\n\n");
        for (int i = 0; i < tam; i++)
        {
            if(i%L==0)
                printf("\n");
            printf(" %u",h_data[i]);
        }
        printf("\n\n");
    }


    /* Free device memory */
    hipFree(d_data);
    hipFree(d_res);
    /* Free host memory */
    free(h_data);

    return 0;
} /* main */

