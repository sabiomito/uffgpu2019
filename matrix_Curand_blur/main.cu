

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <unistd.h>
#include <stdlib.h>
#include <math.h>
#include <hiprand/hiprand.h>
#include <time.h> 
#define CUDA_CALL(x) do { if((x)!=hipSuccess) { \
    printf("Error at %s:%d : err => %s\n",__FILE__,__LINE__,hipGetErrorString(x));\
    return EXIT_FAILURE;}} while(0)
#define CURAND_CALL(x) do { if((x)!=HIPRAND_STATUS_SUCCESS) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    return EXIT_FAILURE;}} while(0)
/**
1- Gerar uma matriz aleatoria
2- Aplicar um blur ou filtro (gerar uma nova matriz de saida, com a media aritimetica da vizinhanca aplicada a cada elemento da matriz)
3- Testar e mandar resultados de tempo para os segintes casos:
- memoria unificada
- copia manual de memoria
- usando stream para copia CPU->GPU
- usando streams para os dois sentidos de copias (ida e volta)
4- Testar para matrizes de 100x100 , 1000x1000, 10000x10000
**/
__global__ void blur(unsigned int origData[],unsigned result[],int L) {


    int thread_idx = threadIdx.x + blockIdx.x * blockDim.x;
    int thread_idy = threadIdx.y + blockIdx.y * blockDim.y;
    
    
    if(thread_idx-1 >= 0 && thread_idx+1 < L && thread_idy-1 >= 0 && thread_idy+1 < L)
    {
        int temp = origData[(thread_idx) + (thread_idy)*L];
        temp += origData[(thread_idx-1) + (thread_idy-1)*L];
        temp += origData[(thread_idx) + (thread_idy-1)*L];
        temp += origData[(thread_idx+1) + (thread_idy-1)*L];

        temp += origData[(thread_idx-1) + (thread_idy)*L];
        //temp += origData[(thread_idx) + (thread_idy)*L];
        temp += origData[(thread_idx+1) + (thread_idy)*L];

        temp += origData[(thread_idx-1) + (thread_idy+1)*L];
        temp += origData[(thread_idx) + (thread_idy+1)*L];
        temp += origData[(thread_idx+1) + (thread_idy+1)*L];
        result[(thread_idx) + (thread_idy)*L] = temp/9;
    }else
    {
        result[(thread_idx) + (thread_idy)*L] = origData[(thread_idx) + (thread_idy)*L];
    }
        
}
/*
*argumentos
*1 - n_elementos
*2 - threads por bloco
*/
int main(int argc, char* argv[]) {
    unsigned int L, tam, *h_data,*d_data,*d_res;
    size_t size;
    hiprandGenerator_t gen;
    hipError_t err = hipSuccess;
    L = 40;
    if(argc > 1)
      L = atoi(argv[1]);

    tam = L*L;
    size = tam*sizeof(unsigned int);

    dim3 block_dim(L,L,1);
    dim3 grid_dim(1,1,1);
    if(L>32)
    {
        block_dim = dim3(32,32,1);
        grid_dim = dim3(ceil(L/32),ceil(L/32),1);
    }
    

    // Allocate memory for the vectors on host memory.
    h_data = (unsigned int*) malloc(size);

    for (int i = 0; i < tam; i++)
        h_data[i] = 0;
    
    

    /* Allocate vectors in device memory */
    CUDA_CALL(hipMalloc(&d_data, size));
    CUDA_CALL(hipMalloc(&d_res, size));

    CURAND_CALL(hiprandCreateGenerator(&gen,HIPRAND_RNG_PSEUDO_DEFAULT));
    CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen,time(NULL)));
    CURAND_CALL(hiprandGenerate(gen,d_data, tam));
    


    CUDA_CALL(hipDeviceSynchronize());



    CUDA_CALL(hipMemcpy(h_data, d_data, size, hipMemcpyDeviceToHost));

    hipEvent_t start, stop;
    CUDA_CALL(hipEventCreate (&start));
    CUDA_CALL(hipEventCreate (&stop));
    CUDA_CALL(hipEventRecord (start, 0)); // 0 is the stream number
    // do Work…
    
    /* Kernel Call */
    blur<<<grid_dim,block_dim>>>(d_data, d_res, L);
    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    CUDA_CALL(hipDeviceSynchronize());
    
    CUDA_CALL(hipEventRecord (stop, 0));
    
    CUDA_CALL(hipEventSynchronize (stop));
    
    float elapsedTime;
    CUDA_CALL(hipEventElapsedTime (&elapsedTime, start, stop));
    printf ("[%d,%.5f],\n", tam,elapsedTime);
    CUDA_CALL(hipEventDestroy(start));
    CUDA_CALL(hipEventDestroy(stop));
    
 
    CUDA_CALL(hipMemcpy(h_data, d_res, size, hipMemcpyDeviceToHost));
    
    /* Free device memory */
    CUDA_CALL( hipFree(d_data));
    CUDA_CALL( hipFree(d_res));
    /* Free host memory */
    free(h_data);

    return 0;
} /* main */

