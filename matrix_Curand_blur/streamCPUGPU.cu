

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <unistd.h>
#include <stdlib.h>
#include <math.h>
#include <hiprand/hiprand.h>
#include <time.h> 
#define CUDA_CALL(x) do { if((x)!=hipSuccess) { \
    printf("Error at %s:%d : err => %s\n",__FILE__,__LINE__,hipGetErrorString(x));\
    return EXIT_FAILURE;}} while(0)
#define CURAND_CALL(x) do { if((x)!=HIPRAND_STATUS_SUCCESS) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    return EXIT_FAILURE;}} while(0)
/**
1- Gerar uma matriz aleatoria
2- Aplicar um blur ou filtro (gerar uma nova matriz de saida, com a media aritimetica da vizinhanca aplicada a cada elemento da matriz)
3- Testar e mandar resultados de tempo para os segintes casos:
- memoria unificada
- copia manual de memoria
- usando stream para copia CPU->GPU
- usando streams para os dois sentidos de copias (ida e volta)
4- Testar para matrizes de 100x100 , 1000x1000, 10000x10000
**/
__global__ void blur(unsigned int origData[],unsigned result[],int L,int ox,int oy) {


    int thread_idx = ox + (threadIdx.x + blockIdx.x * blockDim.x);
    int thread_idy = oy + (threadIdx.y + blockIdx.y * blockDim.y);
    
 
    
    if(thread_idx-1 >= 0 && thread_idx+1 < L && thread_idy-1 >= 0 && thread_idy+1 < L)
    {
        int temp = origData[(thread_idx) + (thread_idy)*L];
        temp += origData[(thread_idx-1) + (thread_idy-1)*L];
        temp += origData[(thread_idx) + (thread_idy-1)*L];
        temp += origData[(thread_idx+1) + (thread_idy-1)*L];

        temp += origData[(thread_idx-1) + (thread_idy)*L];
        //temp += origData[(thread_idx) + (thread_idy)*L];
        temp += origData[(thread_idx+1) + (thread_idy)*L];

        temp += origData[(thread_idx-1) + (thread_idy+1)*L];
        temp += origData[(thread_idx) + (thread_idy+1)*L];
        temp += origData[(thread_idx+1) + (thread_idy+1)*L];
        result[(thread_idx) + (thread_idy)*L] = temp/9;
    }else
    {
        result[(thread_idx) + (thread_idy)*L] = origData[(thread_idx) + (thread_idy)*L];
    }
        
}
/*
*argumentos
*1 - n_elementos
*2 - threads por bloco
*/
int main(int argc, char* argv[]) {
    unsigned int L, tam, *h_data,*d_data,*d_res;
    size_t size;
    hiprandGenerator_t gen;
    hipError_t err = hipSuccess;
    L = 40;
    if(argc > 1)
      L = atoi(argv[1]);

    tam = L*L;
    size = tam*sizeof(unsigned int);

    dim3 block_dim(L,L,1);
    dim3 grid_dim(1,1,1);
    if(L>32)
    {
        block_dim = dim3(32,32,1);
        grid_dim = dim3(ceil(L/32),ceil(L/32),1);
    }
    

    // Allocate memory for the vectors on host memory.
    h_data = (unsigned int*) malloc(size);
      
    /* Allocate vectors in device memory */
    CUDA_CALL(hipMalloc(&d_data, size));
    CUDA_CALL(hipMalloc(&d_res, size));

    

    hipEvent_t start, stop;
    CUDA_CALL(hipEventCreate (&start));
    CUDA_CALL(hipEventCreate (&stop));
    CUDA_CALL(hipEventRecord (start, 0)); // 0 is the stream number
    // do Work…
  
    for(int i=0; i<tam;i++)
        h_data[i]=0;

    CURAND_CALL(hiprandCreateGenerator(&gen,HIPRAND_RNG_PSEUDO_DEFAULT));
    CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen,time(NULL)));
    CURAND_CALL(hiprandGenerate(gen,d_data, tam));
    


    CUDA_CALL(hipDeviceSynchronize());



    CUDA_CALL(hipMemcpy(h_data, d_data, size, hipMemcpyDeviceToHost));

    int rStreams = 2;
  
    /* Kernel Call */
    for(int ox=0; ox < L; ox+=tam/rStreams)
        {
            for(int oy=0; oy < L; oy+=tam/rStreams)
            {
                hipStream_t stream0;
                hipStreamCreate(&stream0);
                dim3 block_dim_temp = dim3(32,32,1);
                dim3 grid_dim_temp = dim3(ceil((L/rStreams)/32),ceil((L/rStreams)/32),1);
                if(ox == 0 && oy==0)
                    blur<<<grid_dim_temp,block_dim_temp>>>(d_data, d_res, L, ox, oy);
                else
                    blur<<<grid_dim_temp,block_dim_temp>>>(d_data, d_res, L+1, ox-1, oy-1);
                err = hipGetLastError();
                if (err != hipSuccess)
                {
                    fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
                    exit(EXIT_FAILURE);
                }
            }
        }
    
    
    

    CUDA_CALL(hipMemcpy(h_data, d_res, size, hipMemcpyDeviceToHost));

    CUDA_CALL(hipDeviceSynchronize());

    for(int i=0; i<tam;i++)
        h_data[i]=0;
    
    CUDA_CALL(hipEventRecord (stop, 0));
    
    CUDA_CALL(hipEventSynchronize (stop));

   

    float elapsedTime;
    CUDA_CALL(hipEventElapsedTime (&elapsedTime, start, stop));
    printf ("[%d,%.5f],\n", tam,elapsedTime);
    CUDA_CALL(hipEventDestroy(start));
    CUDA_CALL(hipEventDestroy(stop));
    
    /* Free device memory */
    CUDA_CALL( hipFree(d_data));
    CUDA_CALL( hipFree(d_res));
    /* Free host memory */
    free(h_data);

    return 0;
} /* main */

