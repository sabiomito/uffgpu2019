#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <unistd.h>
#include <stdlib.h>
#include <math.h>

__global__ void shared_mem(int x[], int n, int blks) {

  int thread_id = threadIdx.x + blockIdx.x * blockDim.x;
  __shared__ float temp_data;
  
  while(thread_id < n){
    temp_data = x[thread_id];
    for(int i=0;i<n;i++){
      temp_data+=1;
    }
    thread_id+=blks*blockDim.x;
  }

}

__global__ void global_mem(int x[], int n, int blks) {

  int thread_id = threadIdx.x + blockIdx.x * blockDim.x;
  while(thread_id < n){
    for(int i=0;i<n;i++){
        x[thread_id]+=1;
    }
    thread_id+=blks*blockDim.x;
  }
}
/*
*argumentos
*1 - n_elementos
*2 - threads por bloco
*3 - n_blocos
*/
int main(int argc, char* argv[]) {
    int n, th_p_blk;
    int *h_x;
    int *d_x;
    size_t size;

    th_p_blk = 1024;
    n = 1024;

  if(argc > 1)
      n = atoi(argv[1]);

    if(argc > 2)
      th_p_blk = atoi(argv[2]);


    int blks = ceil((float)n/(float)th_p_blk);

    if(argc > 3)
      blks = atoi(argv[3]);



    size = n*sizeof(int);

    // Allocate memory for the vectors on host memory.
    h_x = (int*) malloc(size);

    for (int i = 0; i < n; i++) {
        h_x[i] =0;
    }



    /* Allocate vectors in device memory */
    hipMalloc(&d_x, size);

    /* Copy vectors from host memory to device memory */
    hipMemcpy(d_x, h_x, size, hipMemcpyHostToDevice);


    float time_shared,time_global;
    hipEvent_t start, stop;

    hipEventCreate (&start);
    hipEventCreate (&stop);
    hipEventRecord (start, 0); // 0 is the stream number
    // do Work…

    /* Kernel Call */
    shared_mem<<<blks,th_p_blk>>>(d_x, n,blks);
    hipDeviceSynchronize();

    hipEventRecord (stop, 0);
    hipEventSynchronize (stop);
    float elapsedTime;
    hipEventElapsedTime (&elapsedTime, start, stop);
    time_shared = elapsedTime;


    hipEventCreate (&start);
    hipEventCreate (&stop);
    hipEventRecord (start, 0); // 0 is the stream number
    // do Work…

    /* Kernel Call */
    global_mem<<<blks,th_p_blk>>>(d_x, n,blks);
    hipDeviceSynchronize();

    hipEventRecord (stop, 0);
    hipEventSynchronize (stop);

    hipEventElapsedTime (&elapsedTime, start, stop);
    time_global = elapsedTime;
    //printf ("Total GPU Time: %.5f ms \n", elapsedTime);
    printf ("[%d,%.5f,%.5f],\n", n,time_shared,time_global);
    hipEventDestroy(start);



    /* Free device memory */
    hipFree(d_x);
    /* Free host memory */
    free(h_x);

    return 0;
} /* main */

