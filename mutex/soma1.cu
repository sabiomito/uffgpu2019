#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <unistd.h>
#include <stdlib.h>
#include <math.h>

__global__ void soma1(int x[],int hist[], int n, int blks) {

    int thread_id = threadIdx.x + blockIdx.x * blockDim.x;

    while(thread_id < n){

        int index = x[thread_id];
        //atomicAdd(&hist[x[thread_id]],1);
        hist[x[thread_id]]++;
        thread_id+=blks*blockDim.x;
    }

}
/*
*argumentos
*1 - n_elementos
*2 - threads por bloco
*3 - n_blocos
*4 - print
*/
int main(int argc, char* argv[]) {
    int n, th_p_blk;
    int *h_x;
    int *d_x;
    int * d_hist,* h_hist,* h_hist_res;
    size_t size,size_hist;
    int range = 10;

    int print = 0;
    th_p_blk = 1024;
    n = 1024;

  if(argc > 1)
      n = atoi(argv[1]);

    if(argc > 2)
      th_p_blk = atoi(argv[2]);


    int blks = ceil((float)n/(float)th_p_blk);

    if(argc > 3)
      blks = atoi(argv[3]);

    if(argc > 4)
    print=atoi(argv[4]);


    size = n*sizeof(int);
    size_hist = n*sizeof(int);

    // Allocate memory for the vectors on host memory.
    h_x = (int*) malloc(size);
    h_hist_res = (int*) malloc(size_hist);
    h_hist = (int*) malloc(size_hist);


    for (int i = 0; i < range; i++) {
        h_hist[i] = 0;
        h_hist_res[i] = 0;
    }
    for (int i = 0; i < n; i++) {
        h_x[i] = (int)rand()%range;
        h_hist_res[h_x[i]]+=1;
    }



    /* Allocate vectors in device memory */
    hipMalloc(&d_x, size);
    hipMalloc(&d_hist, size_hist);

    /* Copy vectors from host memory to device memory */
    hipMemcpy(d_x, h_x, size, hipMemcpyHostToDevice);
    hipMemcpy(d_hist, h_hist, size_hist, hipMemcpyHostToDevice);


    clock_t Ticks[2];
    Ticks[0] = clock();

    hipEvent_t start, stop;
    hipEventCreate (&start);
    hipEventCreate (&stop);
    hipEventRecord (start, 0); // 0 is the stream number
    // do Work…

    /* Kernel Call */
    soma1<<<blks,th_p_blk>>>(d_x,d_hist, n,blks);


    hipEventRecord (stop, 0);
    hipEventSynchronize (stop);
    float elapsedTime;
    hipEventElapsedTime (&elapsedTime, start, stop);
    //printf ("Total GPU Time: %.5f ms \n", elapsedTime);
    printf ("[%d,%.5f],\n", n,elapsedTime);
    hipEventDestroy(start);

    hipDeviceSynchronize();
    hipMemcpy(h_hist, d_hist, size, hipMemcpyDeviceToHost);


    bool certo=true;
for (int i = 0; i < range; i++){
        if(h_hist[i] != h_hist_res[i])
          certo=false;
    }

       // printf("\n*****\n certo = %s\n*****\n", certo ? "true" : "false");



    /* Free device memory */
    hipFree(d_x);
    hipFree(d_hist);
    /* Free host memory */
    free(h_x);
    free(h_hist);
    free(h_hist_res);

    return 0;
} /* main */

