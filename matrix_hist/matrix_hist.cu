#include "hip/hip_runtime.h"

#include <stdio.h>
#include <time.h>
#include <unistd.h>
#include <stdlib.h>
#include <math.h>

__global__ void Mat_hist(int x[], int z[], int n) {


    int thread_id = threadIdx.x + blockIdx.x * blockDim.x;
    
    __shared__ int hist[256];

    if(threadIdx.x < 256)
    hist[threadIdx.x]=0;
    __syncthreads();
    if(thread_id<n)
    {
        atomicAdd(&hist[x[thread_id]],1);
    }
    __syncthreads();
    if(threadIdx.x < 256)
        atomicAdd(&z[threadIdx.x],hist[threadIdx.x]);	
}
/*
*argumentos
*1 - n_elementos
*2 - threads por bloco
*/
int main(int argc, char* argv[]) {
    int n, th_p_blk;
    int *h_x, *h_z, *h_z_res;
    int *d_x, *d_z;
    size_t size,size_hist;


    th_p_blk = 1024;
    n = 32;
    
    if(argc > 1)
      n = atoi(argv[1]);
    if(argc > 2)
      th_p_blk = atoi(argv[2]);


    int blks = ceil((float)(n)/(float)th_p_blk);

    /* Define vector length */

    size = n*sizeof(int);
    size_hist = 256*sizeof(int);

    // Allocate memory for the vectors on host memory.
    h_x = (int*) malloc(size);
    h_z = (int*) malloc(size_hist);
    h_z_res = (int*) malloc(size_hist);

    for (int i = 0; i < 256; i++)
    {
        h_z_res[i] = 0;
        h_z[i] = 0;
    }
        

    for (int i = 0; i < n; i++) {
        h_x[i] = (int)rand()%256;
        h_z_res[h_x[i]]++;
    }

    
    
   

    /* Allocate vectors in device memory */
    hipMalloc(&d_x, size);
    hipMalloc(&d_z, size_hist);

    /* Copy vectors from host memory to device memory */
    hipMemcpy(d_x, h_x, size, hipMemcpyHostToDevice);
    hipMemcpy(d_z, h_z, size_hist, hipMemcpyHostToDevice);
    



    hipEvent_t start, stop;
    hipEventCreate (&start);
    hipEventCreate (&stop);
    hipEventRecord (start, 0); // 0 is the stream number
    // do Work…

    /* Kernel Call */
    Mat_hist<<<blks,th_p_blk>>>(d_x, d_z, n);

    hipDeviceSynchronize();
    hipEventRecord (stop, 0);
    hipEventSynchronize (stop);
    float elapsedTime;
    hipEventElapsedTime (&elapsedTime, start, stop);
    printf ("[%d,%.5f],\n", n,elapsedTime);
    hipEventDestroy(start);
    hipEventDestroy(stop);

 
    hipMemcpy(h_z, d_z, size_hist, hipMemcpyDeviceToHost);


    bool certo=true;
    //printf("threads/blk %d -- blocks %d\n",th_p_blk,blks);
    for (int i = 0; i < 256; i++){
        //printf("%d - %d\n",h_z_res[i],h_z[i]);
        if(h_z_res[i] != h_z[i])
          certo=false;
    }

    if(!certo)
    printf("\n*****\n certo = %s\n*****\n", certo ? "true" : "false");


    /* Free device memory */
    hipFree(d_x);
    hipFree(d_z);
    /* Free host memory */
    free(h_x);
    free(h_z);
    free(h_z_res);

    return 0;
} /* main */

