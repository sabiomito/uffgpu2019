
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <hiprand/hiprand.h>
#include <time.h> 
#include <iostream>
#include <string>
#include <fstream>
using namespace std;

#define CURAND_CALL(x) do { if((x)!=HIPRAND_STATUS_SUCCESS) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    return EXIT_FAILURE;}} while(0)


int main(int argc, char* argv[]) {
    int *h_data,*d_data,L,tam,print;
    size_t size;
    hiprandRngType_t type = HIPRAND_RNG_PSEUDO_DEFAULT;
    hiprandGenerator_t gen;
    L=10;
    if(argc > 1)
        L = atoi(argv[1]);
    if(argc > 2)
        print = atoi(argv[2]);
    if(argc > 3)
        if(strcmp(argv[3],"CURAND_RNG_PSEUDO_DEFAULT")==0)
            type = HIPRAND_RNG_PSEUDO_DEFAULT;
        else if(strcmp(argv[3],"CURAND_RNG_PSEUDO_MRG32K3A")==0)
            type = HIPRAND_RNG_PSEUDO_MRG32K3A;
        else if(strcmp(argv[3],"CURAND_RNG_PSEUDO_MT19937")==0)
            type = HIPRAND_RNG_PSEUDO_MT19937;
        else if(strcmp(argv[3],"CURAND_RNG_PSEUDO_XORWOW")==0)
            type = HIPRAND_RNG_PSEUDO_XORWOW;
        else if(strcmp(argv[3],"CURAND_RNG_PSEUDO_MTGP32")==0)
            type = HIPRAND_RNG_PSEUDO_MTGP32;
        else if(strcmp(argv[3],"CURAND_RNG_PSEUDO_PHILOX4_32_10")==0)
            type = HIPRAND_RNG_PSEUDO_PHILOX4_32_10;
        else if(strcmp(argv[3],"CURAND_RNG_QUASI_DEFAULT")==0)
            type = HIPRAND_RNG_QUASI_DEFAULT;
        else if(strcmp(argv[3],"CURAND_RNG_QUASI_SCRAMBLED_SOBOL32")==0)
            type = HIPRAND_RNG_QUASI_SCRAMBLED_SOBOL32;
        else if(strcmp(argv[3],"CURAND_RNG_QUASI_SCRAMBLED_SOBOL64")==0)
            type = HIPRAND_RNG_QUASI_SCRAMBLED_SOBOL64;
        else if(strcmp(argv[3],"CURAND_RNG_QUASI_SOBOL32")==0)
            type = HIPRAND_RNG_QUASI_SOBOL32;
        else if(strcmp(argv[3],"CURAND_RNG_QUASI_SOBOL64")==0)
            type = HIPRAND_RNG_QUASI_SOBOL64;
    tam = L*L;
    size = tam*sizeof(int);
    

    // Allocate memory for the vectors on host memory.
    h_data = (int*) malloc(size);
    for (int i = 0; i < tam; i++)
        h_data[i] = 0;

    hipMalloc((void **)&d_data, size);

    if( hiprandCreateGenerator(&gen,type) != HIPRAND_STATUS_SUCCESS)
    {
        printf("Error at %s:%d\n",__FILE__,__LINE__);
        return EXIT_FAILURE;
    }
    
    hiprandSetPseudoRandomGeneratorSeed(gen,0);
    hiprandGenerate(gen,(unsigned int *)d_data, size);

    hipMemcpy(h_data, d_data, size, hipMemcpyDeviceToHost);
    
    ofstream out("data.txt");
    if(print)
    printf("\n\n");
    for (int i = 0; i < tam; i++)
    {
        if(print)
        if(i%L==0)
            printf("\n");
        out << h_data[i] << " ";
        if(print)
        printf(" %u",h_data[i]);
    }
    if(print)
    printf("\n\n");

   
	
	out.close();
    hiprandDestroyGenerator(gen);
    /* Free host memory */
    free(h_data);
    hipFree(d_data);
    return 0;
} /* main */


