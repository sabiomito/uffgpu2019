
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <hiprand/hiprand.h>
#include <time.h> 
#include <iostream>
#include <string>
#include <fstream>
using namespace std;

#define CURAND_CALL(x) do { if((x)!=HIPRAND_STATUS_SUCCESS) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    return EXIT_FAILURE;}} while(0)
    
int main(int argc, char* argv[]) {
    int *h_data,L,tam,print;
    size_t size;
    hiprandGenerator_t gen;
    L=10;
    if(argc > 1)
        L = atoi(argv[1]);
    if(argc > 2)
        print = atoi(argv[2]);

    tam = L*L;
    size = tam*sizeof(int);
    

    // Allocate memory for the vectors on host memory.
    h_data = (int*) malloc(size);
    for (int i = 0; i < tam; i++)
        h_data[i] = 0;
    

    hiprandCreateGeneratorHost(&gen,HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen,time(NULL));
    hiprandGenerate(gen,(unsigned int *)h_data, size);
    
    ofstream out("data.txt");
    if(print)
    printf("\n\n");
    for (int i = 0; i < tam; i++)
    {
        if(print)
        if(i%L==0)
            printf("\n");
        out << h_data[i] << " ";
        if(print)
        printf(" %u",h_data[i]);
    }
    if(print)
    printf("\n\n");

   
	hiprandDestroyGenerator(gen);
	out.close();

    /* Free host memory */
    free(h_data);
    return 0;
} /* main */


