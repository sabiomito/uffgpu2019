
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <hiprand/hiprand.h>
#include <time.h> 
#include <iostream>
#include <string>
#include <fstream>
using namespace std;
int main(int argc, char* argv[]) {
    int *h_data,L,tam,print;
    size_t size;
    hiprandGenerator_t gen;
    L=10;
    if(argc > 1)
        L = atoi(argv[1]);
    if(argc > 2)
        print = atoi(argv[2]);

    tam = L*L;
    size = tam*sizeof(int);
    

    // Allocate memory for the vectors on host memory.
    h_data = (int*) malloc(size);
    for (int i = 0; i < tam; i++)
        h_data[i] = 0;
    

    hiprandCreateGeneratorHost(&gen,HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen,time(NULL));
    hiprandGenerate(gen,(unsigned int *)h_data, size);
    
    ofstream out("data.txt");
    if(print)
    printf("\n\n");
    for (int i = 0; i < tam; i++)
    {
        if(print)
        if(i%L==0)
            printf("\n");
        out << h_data[i] << " ";
        if(print)
        printf(" %u",h_data[i]);
    }
    if(print)
    printf("\n\n");

   
	
	out.close();

    /* Free host memory */
    free(h_data);
    return 0;
} /* main */


