#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h> 
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <string>
#include <fstream>
using namespace std;


 __global__ void setup_kernel(hiprandState *state,unsigned long long seed)
 {
     int id = threadIdx.x + blockIdx.x * blockDim.x;
     hiprand_init(/*seed*/seed,/*sequence*/id, /*offset*/0, &state[id]);
 }
 
 __global__ void generate_kernel(hiprandState *state, unsigned int *result)
 {
     int id = threadIdx.x + blockIdx.x * blockDim.x;
     /* Copy state to local memory for efficiency */
     hiprandState localState = state[id];
     /* Generate pseudo-random unsigned ints */
     result[id] = hiprand(&localState);
     /* Copy state back to global memory */
     state[id] = localState;
 }

 __global__ void generate_kernel2(unsigned int *result,unsigned long long seed)
 {
     int id = threadIdx.x + blockIdx.x * blockDim.x;
     hiprandState state;
     hiprand_init(/*seed*/seed,/*sequence*/id, /*offset*/0, &state);
     result[id] = hiprand(&state);
 }

 
 int main(int argc, char *argv[])
 {
    int *h_data,*d_data,L,tam,print;
    size_t size;
    hiprandState *devStates;

    L=10;
    if(argc > 1)
        L = atoi(argv[1]);
    if(argc > 2)
        print = atoi(argv[2]);

    tam = L*L;
    size = tam*sizeof(int);
    

    // Allocate memory for the vectors on host memory.
    h_data = (int*) malloc(size);
    for (int i = 0; i < tam; i++)
        h_data[i] = 0;

    hipMalloc((void **)&d_data, size);
    hipMalloc((void **)&devStates, tam * sizeof(hiprandState));


    //setup_kernel<<<L,L>>>(devStates,time(NULL));
    //generate_kernel<<<L,L>>>(devStates, (unsigned int *) d_data);
    generate_kernel2<<<L,L>>>((unsigned int *) d_data,time(NULL));

    hipMemcpy(h_data, d_data, size, hipMemcpyDeviceToHost);
    
    ofstream out("data.txt");
    if(print)
    printf("\n\n");
    for (int i = 0; i < tam; i++)
    {
        if(print)
        if(i%L==0)
            printf("\n");
        out << h_data[i] << " ";
        if(print)
        printf(" %u",h_data[i]);
    }
    if(print)
    printf("\n\n");  
	
	out.close();

    /* Free host memory */
    hipFree(devStates);
    hipFree(d_data);
    free(h_data);
    return 0;
 
 }