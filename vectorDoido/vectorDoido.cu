#include "hip/hip_runtime.h"
/* File:     vec_add.cu
 * Purpose:  Implement vector addition on a gpu using cuda
 *
 * Compile:  nvcc [-g] [-G] -o vec_add vec_add.cu
 * Run:      ./vec_add
 */

#include <stdio.h>
#include <time.h>
#include <unistd.h>
#include <stdlib.h>
#include <math.h>

__global__ void Vec_add(float x[], float y[], float z[], int n, int blks) {
    int thread_id = threadIdx.x + blockIdx.x * blockDim.x;
    while(thread_id < n){
        z[thread_id] = x[thread_id] + y[thread_id];
        thread_id+=blks*blockDim.x;
    }
}


int main(int argc, char* argv[]) {
    int n, th_p_blk;
    float *h_x, *h_y, *h_z, *h_z_res;
    float *d_x, *d_y, *d_z;
    size_t size;


    th_p_blk = 1024;
    n = 1024;
    if(argc > 1)
      n = atoi(argv[1]);
    if(argc > 2)
      th_p_blk = atoi(argv[2]);


    int blks = ceil((float)n/(float)th_p_blk);

    if(argc > 3)
      blks = atoi(argv[3]);

    /* Define vector length */

    size = n*sizeof(float);

    // Allocate memory for the vectors on host memory.
    h_x = (float*) malloc(size);
    h_y = (float*) malloc(size);
    h_z = (float*) malloc(size);
    h_z_res = (float*) malloc(size);

    for (int i = 0; i < n; i++) {
        h_x[i] = rand();
        h_y[i] = rand();
        h_z_res[i] = h_x[i]+h_y[i];
    }


    /* Allocate vectors in device memory */
    hipMalloc(&d_x, size);
    hipMalloc(&d_y, size);
    hipMalloc(&d_z, size);

    /* Copy vectors from host memory to device memory */
    hipMemcpy(d_x, h_x, size, hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, size, hipMemcpyHostToDevice);
    

    clock_t Ticks[2];
    Ticks[0] = clock();
    /* Kernel Call */
    Vec_add<<<blks,th_p_blk>>>(d_x, d_y, d_z, n,blks);


    Ticks[1] = clock();
    double Tempo = (Ticks[1] - Ticks[0]) * 1000.0 / CLOCKS_PER_SEC;
    printf("\n\n Tempo gasto: %g ms para:\n %d elementos \n %d blocks \n %d th_p_blk \n\n", Tempo,n,blks,th_p_blk);


    hipDeviceSynchronize();
    hipMemcpy(h_z, d_z, size, hipMemcpyDeviceToHost);


    bool certo=true;
    for (int i = 0; i < n; i++){
        if(h_z_res[i] != h_z[i])
          certo=false;
    }
    printf("\n*****\n certo = %s\n*****\n", certo ? "true" : "false");
   


    /* Free device memory */
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_z);
    /* Free host memory */
    free(h_x);
    free(h_y);
    free(h_z);
    free(h_z_res);

    return 0;
} /* main */
