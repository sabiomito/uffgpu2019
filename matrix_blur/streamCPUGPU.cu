

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <unistd.h>
#include <stdlib.h>
#include <math.h>
#include <hiprand/hiprand.h>
#include <time.h> 
#include <vector>
using namespace std;
#define CUDA_CALL(x) do { if((x)!=hipSuccess) { \
    printf("Error at %s:%d : err => %s\n",__FILE__,__LINE__,hipGetErrorString(x));\
    return EXIT_FAILURE;}} while(0)
#define CURAND_CALL(x) do { if((x)!=HIPRAND_STATUS_SUCCESS) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    return EXIT_FAILURE;}} while(0)
/**
1- Gerar uma matriz aleatoria
2- Aplicar um blur ou filtro (gerar uma nova matriz de saida, com a media aritimetica da vizinhanca aplicada a cada elemento da matriz)
3- Testar e mandar resultados de tempo para os segintes casos:
- memoria unificada
- copia manual de memoria
- usando stream para copia CPU->GPU
- usando streams para os dois sentidos de copias (ida e volta)
4- Testar para matrizes de 100x100 , 1000x1000, 10000x10000
**/
__global__ void blur(unsigned int origData[],unsigned result[],int L,int ox,int oy) {


    int thread_idx = ox + (threadIdx.x + blockIdx.x * blockDim.x);
    int thread_idy = oy + (threadIdx.y + blockIdx.y * blockDim.y);
    
 
    
    if(thread_idx-1 >= 0 && thread_idx+1 < L && thread_idy-1 >= 0 && thread_idy+1 < L)
    {
        int temp = origData[(thread_idx) + (thread_idy)*L];
        temp += origData[(thread_idx-1) + (thread_idy-1)*L];
        temp += origData[(thread_idx) + (thread_idy-1)*L];
        temp += origData[(thread_idx+1) + (thread_idy-1)*L];

        temp += origData[(thread_idx-1) + (thread_idy)*L];
        //temp += origData[(thread_idx) + (thread_idy)*L];
        temp += origData[(thread_idx+1) + (thread_idy)*L];

        temp += origData[(thread_idx-1) + (thread_idy+1)*L];
        temp += origData[(thread_idx) + (thread_idy+1)*L];
        temp += origData[(thread_idx+1) + (thread_idy+1)*L];
        result[(thread_idx) + (thread_idy)*L] = temp/9;
    }else
    {
        result[(thread_idx) + (thread_idy)*L] = origData[(thread_idx) + (thread_idy)*L];
    }
        
}
/*
*argumentos
*1 - n_elementos
*2 - threads por bloco
*/
int main(int argc, char* argv[]) {
    unsigned int L, tam, *h_data,*d_data,*d_res;
    size_t size;
    hipError_t err = hipSuccess;
    L = 40;
    if(argc > 1)
      L = atoi(argv[1]);

    tam = L*L;
    size = tam*sizeof(unsigned int);

    dim3 block_dim(L,L,1);
    dim3 grid_dim(1,1,1);
    if(L>32)
    {
        block_dim = dim3(32,32,1);
        grid_dim = dim3(ceil(L/32),ceil(L/32),1);
    }
    

    // Allocate memory for the vectors on host memory.
    h_data = (unsigned int*) malloc(size);
      
    /* Allocate vectors in device memory */
    CUDA_CALL(hipMalloc(&d_data, size));
    CUDA_CALL(hipMalloc(&d_res, size));

    

    hipEvent_t start, stop;
    CUDA_CALL(hipEventCreate (&start));
    CUDA_CALL(hipEventCreate (&stop));
    CUDA_CALL(hipEventRecord (start, 0)); // 0 is the stream number
    // do Work…
    srand(time(NULL));
    for(int i=0; i<tam;i++)
        h_data[i]=rand();



    //CUDA_CALL(cudaMemcpy(d_data,h_data, size, cudaMemcpyHostToDevice));

    int rStreams = 4;
    dim3 block_dim_temp = dim3(32,32,1);
    dim3 grid_dim_temp = dim3(ceil(L/32),ceil((L/rStreams)/32),1);
    if(grid_dim_temp.y<1)
        grid_dim_temp = dim3(grid_dim_temp.x,1,1);
    //printf("\n grid_dim_temp - %d , %d, %d \n",grid_dim_temp.x,grid_dim_temp.y,grid_dim_temp.z);
    vector<hipStream_t> streams;
    /* Kernel Call */
    for(int oy=0; oy < L; oy+=L/rStreams)
    {
        streams.push_back(hipStream_t());
        hipStreamCreate(&streams.back());
        hipMemcpyAsync (d_data, h_data+(oy*L),(L/rStreams*L)*sizeof(unsigned int), hipMemcpyHostToDevice,streams.back());
    }
    for(int oy=0,i=0; oy < L; oy+=L/rStreams,i++)
        if(i==0)
            blur<<<grid_dim_temp,block_dim_temp,0,streams[i]>>>(d_data, d_res, L, 0, oy);
        else
            blur<<<grid_dim_temp,block_dim_temp,0,streams[i]>>>(d_data, d_res, L+1, 0, oy-1);
            

    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    CUDA_CALL(hipMemcpy(h_data, d_res, size, hipMemcpyDeviceToHost));

    CUDA_CALL(hipDeviceSynchronize());

    for(int i=0; i<tam;i++)
        h_data[i]=0;
    
    CUDA_CALL(hipEventRecord (stop, 0));
    
    CUDA_CALL(hipEventSynchronize (stop));

   

    float elapsedTime;
    CUDA_CALL(hipEventElapsedTime (&elapsedTime, start, stop));
    printf ("[%d,%.5f],\n", tam,elapsedTime);
    CUDA_CALL(hipEventDestroy(start));
    CUDA_CALL(hipEventDestroy(stop));
    
    /* Free device memory */
    CUDA_CALL( hipFree(d_data));
    CUDA_CALL( hipFree(d_res));
    /* Free host memory */
    free(h_data);

    return 0;
} /* main */
