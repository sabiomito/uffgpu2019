

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <unistd.h>
#include <stdlib.h>
#include <math.h>
#include <hiprand/hiprand.h>
#include <time.h> 
#define CUDA_CALL(x) do { if((x)!=hipSuccess) { \
    printf("Error at %s:%d : err => %s\n",__FILE__,__LINE__,hipGetErrorString(x));\
    return EXIT_FAILURE;}} while(0)
#define CURAND_CALL(x) do { if((x)!=HIPRAND_STATUS_SUCCESS) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    return EXIT_FAILURE;}} while(0)
/**
- memoria unificada
[10000,0.02410],
[1000000,0.12742],
[100000000,10.52038],
1- Gerar uma matriz aleatoria
2- Aplicar um blur ou filtro (gerar uma nova matriz de saida, com a media aritimetica da vizinhanca aplicada a cada elemento da matriz)
3- Testar e mandar resultados de tempo para os segintes casos:
- memoria unificada
- copia manual de memoria
- usando stream para copia CPU->GPU
- usando streams para os dois sentidos de copias (ida e volta)
4- Testar para matrizes de 100x100 , 1000x1000, 10000x10000
**/
__global__ void blur(unsigned int origData[],unsigned result[],int L) {


    int thread_idx = threadIdx.x + blockIdx.x * blockDim.x;
    int thread_idy = threadIdx.y + blockIdx.y * blockDim.y;
    

    if(thread_idx-1 >= 0 && thread_idx+1 < L && thread_idy-1 >= 0 && thread_idy+1 < L)
    {
        int temp = origData[(thread_idx) + (thread_idy)*L];
        temp += origData[(thread_idx-1) + (thread_idy-1)*L];
        temp += origData[(thread_idx) + (thread_idy-1)*L];
        temp += origData[(thread_idx+1) + (thread_idy-1)*L];

        temp += origData[(thread_idx-1) + (thread_idy)*L];
        //temp += origData[(thread_idx) + (thread_idy)*L];
        temp += origData[(thread_idx+1) + (thread_idy)*L];

        temp += origData[(thread_idx-1) + (thread_idy+1)*L];
        temp += origData[(thread_idx) + (thread_idy+1)*L];
        temp += origData[(thread_idx+1) + (thread_idy+1)*L];
        result[(thread_idx) + (thread_idy)*L] = temp/9;
    }else
    {
        result[(thread_idx) + (thread_idy)*L] = origData[(thread_idx) + (thread_idy)*L];
    }
        
}
/*
*argumentos
*1 - n_elementos
*2 - threads por bloco
*/
int main(int argc, char* argv[]) {
    unsigned int L, tam, *data,*res;
    size_t size;
    hipError_t err = hipSuccess;
    L = 40;
    if(argc > 1)
      L = atoi(argv[1]);

    tam = L*L;
    size = tam*sizeof(unsigned int);
    hipMallocManaged(&data,size);
    hipMallocManaged(&res,size);

    
    hipEvent_t start, stop;
    CUDA_CALL(hipEventCreate (&start));
    CUDA_CALL(hipEventCreate (&stop));
    CUDA_CALL(hipEventRecord (start, 0)); // 0 is the stream number

    

    dim3 block_dim(L,L,1);
    dim3 grid_dim(1,1,1);
    if(L>32)
    {
        block_dim = dim3(32,32,1);
        grid_dim = dim3(ceil(L/32),ceil(L/32),1);
    }
    

    srand(time(NULL));
    for(int i=0; i<tam;i++)
        data[i]=rand();


    // do Work…
    
    /* Kernel Call */
    blur<<<grid_dim,block_dim>>>(data,res, L);
    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    CUDA_CALL(hipDeviceSynchronize());
    
   
    for(int i=0; i<tam;i++)
        data[i]=0;
    
    CUDA_CALL(hipEventRecord (stop, 0));
    CUDA_CALL(hipEventSynchronize (stop));
    
    float elapsedTime;
    CUDA_CALL(hipEventElapsedTime (&elapsedTime, start, stop));
    printf ("[%d,%.5f],\n", tam,elapsedTime);
    CUDA_CALL(hipEventDestroy(start));
    CUDA_CALL(hipEventDestroy(stop));
    
 
    
    /* Free device memory */
    CUDA_CALL( hipFree(data));
    CUDA_CALL( hipFree(res));


    return 0;
} /* main */

