#include "hip/hip_runtime.h"

#include <stdio.h>
#include <time.h>
#include <unistd.h>
#include <stdlib.h>
#include <math.h>

__global__ void Mat_mul(float x[], float y[], float z[], int n, int blks) {


    int thread_id = threadIdx.x + blockIdx.x * blockDim.x;

    while(thread_id < n*n){

    	z[thread_id] = 0;
    	int linha=thread_id/n;
    	int coluna = thread_id - (linha*n);

        for(int i=0;i<n;i++)
        	z[thread_id] += (x[(i)*n+coluna] * y[linha*n+(i)]);


        thread_id+=blks*blockDim.x;
    }


}
/*
*argumentos
*1 - n_elementos
*2 - threads por bloco
*3 - n_blocos
*4 - print
*/
int main(int argc, char* argv[]) {
    int n, th_p_blk;
    float *h_x, *h_y, *h_z, *h_z_res;
    float *d_x, *d_y, *d_z;
    size_t size;

    int print = 0;
    th_p_blk = 1024;
    n = 1024;
    
    if(argc > 1)
      n = atoi(argv[1]);
    if(argc > 2)
      th_p_blk = atoi(argv[2]);


    int blks = ceil((float)n/(float)th_p_blk);

    if(argc > 3)
      blks = atoi(argv[3]);

    if(argc > 4)
    print=atoi(argv[4]);

    if(argc<1){
        printf("/*\n*argumentos\n*1 - n_elementos\n*2 - threads por bloco\n*3 - n_blocos\n*4 - print\n*/\npadrao \nn = %d\nthreads por bloco = %d\nblocos = %d\n\n",n,th_p_blk,blks);
    }
    /* Define vector length */

    size = n*n*sizeof(float);

    // Allocate memory for the vectors on host memory.
    h_x = (float*) malloc(size);
    h_y = (float*) malloc(size);
    h_z = (float*) malloc(size);
    h_z_res = (float*) malloc(size);

    for (int i = 0; i < n*n; i++) {
        h_x[i] = (int)rand()%10;
        h_y[i] = (int)rand()%10;
        h_z_res[i] = h_x[i]+h_y[i];
    }

    for(int i=0;i<n;i++)
    {
    	for(int j=0;j<n;j++)
    	{
    		h_z_res[i*n+j]=0;
            for(int k=0;k<n;k++)
            {
                h_z_res[i*n+j] += (h_x[k*n+j]*h_y[i*n+k]);
            }
            
    	}
    }
    if(print)
    {
        for(int i=0;i<n;i++)
        {
            for(int j=0;j<n;j++)
            {
                printf("%f ",h_x[i*n+j]);
            }
            printf("\n");
        }
        printf("----\n");
        for(int i=0;i<n;i++)
        {
            for(int j=0;j<n;j++)
            {
                printf("%f ",h_y[i*n+j]);
            }
            printf("\n");
        }
        printf("----\n");
        for(int i=0;i<n;i++)
        {
            for(int j=0;j<n;j++)
            {
                printf("%f ",h_z_res[i*n+j]);
            }
            printf("\n");
        }
    }
   

    /* Allocate vectors in device memory */
    hipMalloc(&d_x, size);
    hipMalloc(&d_y, size);
    hipMalloc(&d_z, size);

    /* Copy vectors from host memory to device memory */
    hipMemcpy(d_x, h_x, size, hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, size, hipMemcpyHostToDevice);
    

    clock_t Ticks[2];
    Ticks[0] = clock();

    hipEvent_t start, stop;
    hipEventCreate (&start);
    hipEventCreate (&stop);
    hipEventRecord (start, 0); // 0 is the stream number
    // do Work…

    /* Kernel Call */
    Mat_mul<<<blks,th_p_blk>>>(d_x, d_y, d_z, n,blks);

    hipDeviceSynchronize();
    hipEventRecord (stop, 0);
    hipEventSynchronize (stop);
    float elapsedTime;
    hipEventElapsedTime (&elapsedTime, start, stop);
    printf ("[%d,%.5f],\n", n,elapsedTime);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    Ticks[1] = clock();
    double Tempo = (Ticks[1] - Ticks[0]) * 1000.0 / CLOCKS_PER_SEC;
    if(print)
    {
        printf("\n\n Tempo gasto: %g ms para:\n %d elementos \n %d blocks \n %d th_p_blk \n\n", Tempo,n,blks,th_p_blk);
    }
    //printf("%g",Tempo);
 
    hipMemcpy(h_z, d_z, size, hipMemcpyDeviceToHost);

    if(print)
    {
        printf("RESULTADO----\n");
        for(int i=0;i<n;i++)
        {
            for(int j=0;j<n;j++)
            {
                printf("%f ",h_z[i*n+j]);
            }
            printf("\n");
        }
    }

    bool certo=true;
    for (int i = 0; i < n*n; i++){
        if(h_z_res[i] != h_z[i])
          certo=false;
    }
    if(print)
    {
        printf("\n*****\n certo = %s\n*****\n", certo ? "true" : "false");
    }
    if(!certo)
    printf("\n*****\n certo = %s\n*****\n", certo ? "true" : "false");


    /* Free device memory */
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_z);
    /* Free host memory */
    free(h_x);
    free(h_y);
    free(h_z);
    free(h_z_res);

    return 0;
} /* main */

